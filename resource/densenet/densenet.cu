#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
extern "C" __global__ void fused_nn_conv2d_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[49];
  __shared__ float placeholder_shared[16];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 7) + ((int)threadIdx.z)) < 7) {
    if ((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 49) {
      if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 7) {
        if (((int)threadIdx.x) < 1) {
          pad_temp_shared[((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder[((((((((((int)threadIdx.z) / 7) * 196) + (((int)blockIdx.y) * 98)) + ((((int)threadIdx.z) % 7) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + ((int)threadIdx.y)))];
        }
      }
    }
  }
  if ((((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 16) {
    if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 2) {
      if (((int)threadIdx.x) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 2)) + ((int)threadIdx.x)) + ((int)threadIdx.y)))];
      }
    }
  }
  __syncthreads();
  compute_local[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[(((int)threadIdx.z))], compute_local[(0)]);
  compute_local[(1)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[((((int)threadIdx.z) + 8))], compute_local[(1)]);
  compute[(((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 1568))] = compute_local[(1)];
}

extern "C" __global__ void fused_nn_batch_flatten_kernel0(float* __restrict__ tensor, float* __restrict__ placeholder) {
  tensor[(0)] = placeholder[(0)];
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_8_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[4];
  __shared__ float pad_temp_shared[1792];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 112) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = placeholder[((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 112) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[(((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 4)) < 32) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) < 512) {
        if (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 8) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)))] = placeholder1[(((((((int)threadIdx.z) * 256) + ((((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 4) * 128)) + (rc_outer * 16)) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 560))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 560))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 561))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 561))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 672))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 672))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 673))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 673))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 785))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 785))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1008))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1008))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1009))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1009))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1120))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1120))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1121))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1121))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1232))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1232))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1233))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1233))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1344))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1344))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1345))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1345))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1456))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1456))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1457))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1457))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1568))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1568))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1569))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1569))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1680))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1680))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1681))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1681))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(3)]);
  }
  T_relu[(((((((int)threadIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12544))] = max((compute[(2)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12545))] = max((compute[(3)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_3_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[1];
  __shared__ float pad_temp_shared[128];
  __shared__ float placeholder_shared[72];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 32; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[((((((int)threadIdx.y) * 64) + (((int)threadIdx.x) * 32)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 2) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner & 15) >> 2))) && (((((int)blockIdx.y) * 2) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner & 15) >> 2)) < 15)) && (1 <= ((((int)blockIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner & 3)))) && (((((int)blockIdx.x) * 2) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner & 3)) < 15)) ? placeholder[((((((((((rc_outer * 1568) + (((int)threadIdx.y) * 784)) + (((int)threadIdx.x) * 392)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner >> 4) * 196)) + (((int)blockIdx.y) * 28)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner & 15) >> 2) * 14)) + (((int)blockIdx.x) * 2)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner & 3)) - 15))] : 0.000000e+00f);
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 18; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      placeholder_shared[((((((int)threadIdx.y) * 36) + (((int)threadIdx.x) * 18)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = placeholder1[(((((rc_outer * 72) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 18)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((rc_inner * 16) + (((int)threadIdx.y) * 4)) + (ry_inner * 4)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[((((rc_inner * 9) + (ry_inner * 3)) + rx_inner))], compute[(0)]);
        }
      }
    }
  }
  T_relu[(((((((int)blockIdx.y) * 28) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(0)]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_10_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[4];
  __shared__ float pad_temp_shared[32];
  __shared__ float placeholder_shared[32];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  if (((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)) < 32) {
    if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 1) {
      if (((int)threadIdx.x) < 1) {
        pad_temp_shared[(((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)))] = placeholder[(((((((int)blockIdx.y) * 224) + ((((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)) >> 3) * 56)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)) & 7)))];
      }
    }
  }
  if (((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)) < 32) {
    if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 1) {
      if (((int)threadIdx.x) < 1) {
        placeholder_shared[(((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)))] = placeholder1[(((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)))];
      }
    }
  }
  __syncthreads();
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((int)threadIdx.z))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 16))], placeholder_shared[(((int)threadIdx.z))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((int)threadIdx.z))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 17))], placeholder_shared[(((int)threadIdx.z))], compute[(3)]);
  T_relu[((((((((int)threadIdx.z) * 3136) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 3136) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 112))] = max((compute[(2)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 3136) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 3136) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 113))] = max((compute[(3)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_7_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[4];
  __shared__ float pad_temp_shared[112];
  __shared__ float placeholder_shared[32];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  if ((((((int)threadIdx.z) * 7) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)) < 112) {
    if (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) < 7) {
      if (((int)threadIdx.x) < 2) {
        pad_temp_shared[((((((int)threadIdx.z) * 7) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)))] = placeholder[(((((((int)blockIdx.y) * 112) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)))];
      }
    }
  }
  if ((((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 32) {
    if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 2) {
      if (((int)threadIdx.x) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[((((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) + ((int)threadIdx.y)))];
      }
    }
  }
  __syncthreads();
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((int)threadIdx.z))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) + 16))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((int)threadIdx.z))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) + 16))], compute[(3)]);
  T_relu[(((((((int)threadIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12544))] = max((compute[(2)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)threadIdx.z) * 784) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12545))] = max((compute[(3)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_1_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[4];
  __shared__ float pad_temp_shared[112];
  __shared__ float placeholder_shared[32];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  if ((((((int)threadIdx.z) * 7) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)) < 112) {
    if (((((int)threadIdx.y) * 2) + ((int)threadIdx.x)) < 7) {
      if (((int)threadIdx.x) < 2) {
        pad_temp_shared[((((((int)threadIdx.z) * 7) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)))] = placeholder[(((((((int)blockIdx.y) * 112) + (((int)threadIdx.z) * 7)) + (((int)threadIdx.y) * 2)) + ((int)threadIdx.x)))];
      }
    }
  }
  if ((((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 32) {
    if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 2) {
      if (((int)threadIdx.x) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((int)blockIdx.z) * 32) + (((int)threadIdx.z) * 2)) + ((int)threadIdx.x)) + ((int)threadIdx.y)))];
      }
    }
  }
  __syncthreads();
  compute_local[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((int)threadIdx.z))], compute_local[(0)]);
  compute_local[(2)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) + 16))], compute_local[(2)]);
  compute_local[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((int)threadIdx.z))], compute_local[(1)]);
  compute_local[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) + 16))], compute_local[(3)]);
  compute[((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12544))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12545))] = compute_local[(3)];
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_11_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[4];
  __shared__ float pad_temp_shared[512];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  pad_temp_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder[(((((((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))];
  pad_temp_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[((((((((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))];
  placeholder_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder1[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))];
  placeholder_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder1[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))];
  __syncthreads();
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 16))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 17))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 48))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 80))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 81))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 144))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 145))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 176))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 177))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 208))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 209))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 240))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 241))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 272))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 273))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 304))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 305))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 368))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 369))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 400))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 401))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 432))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 433))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 464))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 465))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 496))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 497))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(3)]);
  __syncthreads();
  pad_temp_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder[((((((((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 50176))];
  pad_temp_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[((((((((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 50177))];
  placeholder_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder1[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 16))];
  placeholder_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder1[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 17))];
  __syncthreads();
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 16))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 17))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 48))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 80))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 81))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 144))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 145))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 176))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 177))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 208))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 209))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 240))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 241))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 272))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 273))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 304))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 305))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 368))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 369))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 400))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 401))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 432))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 433))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 464))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 465))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 496))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 497))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(3)]);
  __syncthreads();
  pad_temp_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder[((((((((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100352))];
  pad_temp_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[((((((((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100353))];
  placeholder_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder1[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 32))];
  placeholder_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder1[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 33))];
  __syncthreads();
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 16))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 17))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 48))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 80))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 81))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 144))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 145))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 176))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 177))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 208))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 209))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 240))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 241))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 272))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 273))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 304))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 305))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 368))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 369))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 400))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 401))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 432))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 433))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 464))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 465))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 496))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 497))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(3)]);
  __syncthreads();
  pad_temp_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder[((((((((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 150528))];
  pad_temp_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[((((((((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 150529))];
  placeholder_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder1[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 48))];
  placeholder_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder1[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 49))];
  __syncthreads();
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 16))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 17))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 48))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 80))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 81))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 144))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 145))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 176))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 177))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 208))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 209))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 240))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 241))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 272))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 273))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 304))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 305))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 368))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 369))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 400))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 401))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 432))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 433))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 464))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 465))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(3)]);
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
  compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 496))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(2)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
  compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 497))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(3)]);
  T_relu[((((((((int)threadIdx.z) * 3136) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 3136) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 112))] = max((compute[(2)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 3136) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 3136) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 113))] = max((compute[(3)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_4_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[2];
  __shared__ float pad_temp_shared[49];
  __shared__ float placeholder_shared[16];
  compute[(0)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 7) + ((int)threadIdx.z)) < 7) {
    if ((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 49) {
      if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 7) {
        if (((int)threadIdx.x) < 1) {
          pad_temp_shared[((((((int)threadIdx.z) * 7) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder[((((((((((int)threadIdx.z) / 7) * 196) + (((int)blockIdx.y) * 98)) + ((((int)threadIdx.z) % 7) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + ((int)threadIdx.y)))];
        }
      }
    }
  }
  if ((((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 16) {
    if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 2) {
      if (((int)threadIdx.x) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((int)blockIdx.z) * 16) + (((int)threadIdx.z) * 2)) + ((int)threadIdx.x)) + ((int)threadIdx.y)))];
      }
    }
  }
  __syncthreads();
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[(((int)threadIdx.z))], compute[(0)]);
  compute[(1)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[((((int)threadIdx.z) + 8))], compute[(1)]);
  T_relu[(((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 1568))] = max((compute[(1)] + placeholder2[((((((int)blockIdx.z) * 16) + ((int)threadIdx.z)) + 8))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_dense_add_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense_rf[1];
  float red_buf0[1];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  if (((int)threadIdx.x) < 1) {
    T_dense_rf[(0)] = __ocml_fma_f32(placeholder[(((int)threadIdx.x))], placeholder1[((((int)blockIdx.x) + ((int)threadIdx.x)))], T_dense_rf[(0)]);
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = T_dense_rf[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = red_buf0[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_add[(((int)blockIdx.x))] = (T_dense[(0)] + placeholder2[(((int)blockIdx.x))]);
  }
}

extern "C" __global__ void fused_nn_avg_pool2d_add_nn_relu_kernel0(float* __restrict__ placeholder, float* __restrict__ T_relu, float* __restrict__ placeholder1) {
  float tensor[1];
  tensor[(0)] = 0.000000e+00f;
  for (int dh = 0; dh < 2; ++dh) {
    for (int dw = 0; dw < 2; ++dw) {
      if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 43904) {
        tensor[(0)] = (tensor[(0)] + placeholder[((((((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) / 7) * 28) + (dh * 14)) + ((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) % 7) * 2)) + dw))]);
      }
    }
  }
  if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 43904) {
    T_relu[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = max(__ocml_fma_f32(tensor[(0)], 2.500000e-01f, placeholder1[((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) / 49))]), 0.000000e+00f);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[1];
  __shared__ float pad_temp_shared[162];
  __shared__ float placeholder_shared[18];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if ((((((int)threadIdx.y) * 24) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 162) {
        if (((((int)threadIdx.x) * 4) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 24) {
          pad_temp_shared[((((((int)threadIdx.y) * 24) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((9 <= ((((((int)threadIdx.y) * 24) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 81)) && (((((((int)threadIdx.y) * 24) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 81) < 72)) && (1 <= ((((((int)threadIdx.y) * 24) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 9))) && (((((((int)threadIdx.y) * 24) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 9) < 8)) ? placeholder[((((((rc_outer * 98) + (((((((int)threadIdx.y) * 24) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 81) * 49)) + ((((((((int)threadIdx.y) * 24) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 81) / 9) * 7)) + ((((((int)threadIdx.y) * 24) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 9)) - 8))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.x) / 3) + ((int)threadIdx.y)) < 6) {
      if (((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) < 18) {
        if (((int)threadIdx.x) < 3) {
          placeholder_shared[(((((int)threadIdx.y) * 3) + ((int)threadIdx.x)))] = placeholder1[((((((((int)threadIdx.y) / 6) * 288) + (rc_outer * 18)) + ((((int)threadIdx.y) % 6) * 3)) + ((int)threadIdx.x)))];
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 2; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((rc_inner * 81) + (((int)threadIdx.y) * 9)) + (ry_inner * 9)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[((((rc_inner * 9) + (ry_inner * 3)) + rx_inner))], compute[(0)]);
        }
      }
    }
  }
  T_relu[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(0)]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_9_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[1];
  __shared__ float pad_temp_shared[360];
  __shared__ float placeholder_shared[36];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 7; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((int)threadIdx.y) * 5) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 9)) < 40) {
        if ((((((int)threadIdx.y) * 45) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 360) {
          if (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 45) {
            pad_temp_shared[((((((int)threadIdx.y) * 45) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 8) + (((((int)threadIdx.y) * 5) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 9)) % 10))) && (((((int)blockIdx.y) * 8) + (((((int)threadIdx.y) * 5) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 9)) % 10)) < 57)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 9)) < 57)) ? placeholder[((((((((rc_outer * 12544) + ((((((int)threadIdx.y) * 5) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 9)) / 10) * 3136)) + (((int)blockIdx.y) * 448)) + ((((((int)threadIdx.y) * 5) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 9)) % 10) * 56)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 9)) - 57))] : 0.000000e+00f);
          }
        }
      }
    }
    if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 36) {
      if (((int)threadIdx.x) < 5) {
        placeholder_shared[(((((int)threadIdx.y) * 5) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer * 36) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))];
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((rc_inner * 90) + (((int)threadIdx.y) * 9)) + (ry_inner * 9)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[((((rc_inner * 9) + (ry_inner * 3)) + rx_inner))], compute[(0)]);
        }
      }
    }
  }
  T_relu[(((((((int)blockIdx.y) * 448) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(0)]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_avg_pool2d_add_nn_relu_1_kernel0(float* __restrict__ placeholder, float* __restrict__ T_relu, float* __restrict__ placeholder1) {
  float tensor[1];
  tensor[(0)] = 0.000000e+00f;
  for (int dh = 0; dh < 2; ++dh) {
    for (int dw = 0; dw < 2; ++dw) {
      tensor[(0)] = (tensor[(0)] + placeholder[((((((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) / 14) * 56) + (dh * 28)) + ((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) % 14) * 2)) + dw))]);
    }
  }
  T_relu[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = max(__ocml_fma_f32(tensor[(0)], 2.500000e-01f, placeholder1[((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) / 196))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_5_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[2];
  __shared__ float pad_temp_shared[784];
  __shared__ float placeholder_shared[256];
  compute[(0)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))] = placeholder[((((((((rc_outer * 3136) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.y) * 2) + ((((int)threadIdx.x) * 2) / 7)) / 7) * 196)) + (((int)blockIdx.y) * 98)) + ((((((int)threadIdx.y) * 2) + ((((int)threadIdx.x) * 2) / 7)) % 7) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 2) % 7)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[((((((((rc_outer * 3136) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) / 7)) / 7) * 196)) + (((int)blockIdx.y) * 98)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) / 7)) % 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 2) + 1) % 7)))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 4)) < 16) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)) < 256) {
        if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 5) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 512)) + ((((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 4) * 256)) + (rc_outer * 16)) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 128))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 129))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))], placeholder_shared[(((((int)threadIdx.z) * 16) + 130))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))], placeholder_shared[(((((int)threadIdx.z) * 16) + 131))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 16) + 132))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 245))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 245))], placeholder_shared[(((((int)threadIdx.z) * 16) + 133))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 294))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 294))], placeholder_shared[(((((int)threadIdx.z) * 16) + 134))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 343))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 343))], placeholder_shared[(((((int)threadIdx.z) * 16) + 135))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 16) + 136))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))], placeholder_shared[(((((int)threadIdx.z) * 16) + 137))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 490))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 490))], placeholder_shared[(((((int)threadIdx.z) * 16) + 138))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 539))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 539))], placeholder_shared[(((((int)threadIdx.z) * 16) + 139))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 588))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 588))], placeholder_shared[(((((int)threadIdx.z) * 16) + 140))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))], placeholder_shared[(((((int)threadIdx.z) * 16) + 141))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 686))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 686))], placeholder_shared[(((((int)threadIdx.z) * 16) + 142))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 735))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 735))], placeholder_shared[(((((int)threadIdx.z) * 16) + 143))], compute[(1)]);
  }
  T_relu[(((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 1568))] = max((compute[(1)] + placeholder2[((((((int)blockIdx.z) * 16) + ((int)threadIdx.z)) + 8))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_1_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[1];
  __shared__ float pad_temp_shared[49];
  __shared__ float placeholder_shared[16];
  compute[(0)] = 0.000000e+00f;
  if ((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 49) {
    if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 4) {
      if (((int)threadIdx.x) < 1) {
        pad_temp_shared[((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder[((((((int)threadIdx.z) * 4) + ((int)threadIdx.x)) + ((int)threadIdx.y)))];
      }
    }
  }
  if (((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)) < 16) {
    if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 1) {
      if (((int)threadIdx.x) < 1) {
        placeholder_shared[(((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)))] = placeholder1[(((((((int)blockIdx.z) * 16) + ((int)threadIdx.x)) + ((int)threadIdx.y)) + ((int)threadIdx.z)))];
      }
    }
  }
  __syncthreads();
  compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[(((int)threadIdx.z))], compute[(0)]);
  T_relu[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_2_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[1];
  __shared__ float pad_temp_shared[196];
  __shared__ float placeholder_shared[112];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 7; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      pad_temp_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder[((((((rc_outer * 1372) + (((int)threadIdx.z) * 343)) + (((int)threadIdx.x) * 49)) + (((int)blockIdx.y) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))];
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 4; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      placeholder_shared[((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = placeholder1[((((((((int)blockIdx.z) * 3584) + (((int)threadIdx.z) * 896)) + (rc_outer * 28)) + (((int)threadIdx.x) * 4)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 28; ++rc_inner) {
      compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((rc_inner * 7) + ((int)threadIdx.x)))], placeholder_shared[(((((int)threadIdx.z) * 28) + rc_inner))], compute[(0)]);
    }
  }
  T_relu[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.z) * 49)) + (((int)blockIdx.y) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 4) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_2_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
  float compute_local[8];
  __shared__ float pad_temp_shared[32];
  __shared__ float placeholder_shared[64];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  if (((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)) < 32) {
    if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 1) {
      if (((int)threadIdx.x) < 1) {
        pad_temp_shared[(((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)))] = placeholder[(((((((int)blockIdx.y) * 224) + ((((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)) >> 3) * 56)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.x) + ((int)threadIdx.y)) + ((int)threadIdx.z)) & 7)))];
      }
    }
  }
  if ((((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 64) {
    if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 2) {
      if (((int)threadIdx.x) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 2) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((int)blockIdx.z) * 64) + (((int)threadIdx.z) * 2)) + ((int)threadIdx.x)) + ((int)threadIdx.y)))];
      }
    }
  }
  __syncthreads();
  compute_local[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((int)threadIdx.z))], compute_local[(0)]);
  compute_local[(4)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) + 32))], compute_local[(4)]);
  compute_local[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 16))], placeholder_shared[(((int)threadIdx.z))], compute_local[(2)]);
  compute_local[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 16))], placeholder_shared[((((int)threadIdx.z) + 32))], compute_local[(6)]);
  compute_local[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((int)threadIdx.z))], compute_local[(1)]);
  compute_local[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) + 32))], compute_local[(5)]);
  compute_local[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 17))], placeholder_shared[(((int)threadIdx.z))], compute_local[(3)]);
  compute_local[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 17))], placeholder_shared[((((int)threadIdx.z) + 32))], compute_local[(7)]);
  compute[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))] = compute_local[(0)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100352))] = compute_local[(4)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 112))] = compute_local[(2)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100464))] = compute_local[(6)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = compute_local[(1)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100353))] = compute_local[(5)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 113))] = compute_local[(3)];
  compute[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100465))] = compute_local[(7)];
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_12_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[1];
  __shared__ float pad_temp_shared[481];
  __shared__ float placeholder_shared[196];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 3; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 2; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((((int)threadIdx.z) * 121) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 481) {
        if ((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 121) {
          pad_temp_shared[(((((((int)threadIdx.z) * 121) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((3 <= ((((int)blockIdx.y) * 32) + (((((((int)threadIdx.z) * 121) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 13))) && (((((int)blockIdx.y) * 32) + (((((((int)threadIdx.z) * 121) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 13)) < 227)) && (3 <= ((((int)blockIdx.x) * 8) + (((((((int)threadIdx.z) * 121) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 13)))) && (((((int)blockIdx.x) * 8) + (((((((int)threadIdx.z) * 121) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 13)) < 227)) ? placeholder[(((((((rc_outer * 50176) + (((int)blockIdx.y) * 7168)) + ((((((((int)threadIdx.z) * 121) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 13) * 224)) + (((int)blockIdx.x) * 8)) + (((((((int)threadIdx.z) * 121) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 13)) - 675))] : 0.000000e+00f);
        }
      }
    }
    if (((((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) / 49) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 7) + (((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) / 7)) < 28) {
        if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 4)) + ((int)threadIdx.x)) < 196) {
          if (((((int)threadIdx.y) * 4) + ((int)threadIdx.x)) < 49) {
            placeholder_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 4)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 588) + (((int)threadIdx.z) * 147)) + (rc_outer * 49)) + (((int)threadIdx.y) * 4)) + ((int)threadIdx.x)))];
          }
        }
      }
    }
    __syncthreads();
    for (int ry_inner = 0; ry_inner < 7; ++ry_inner) {
      for (int rx_inner = 0; rx_inner < 7; ++rx_inner) {
        compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((((int)threadIdx.y) * 26) + (ry_inner * 13)) + (((int)threadIdx.x) * 2)) + rx_inner))], placeholder_shared[((((((int)threadIdx.z) * 49) + (ry_inner * 7)) + rx_inner))], compute[(0)]);
      }
    }
  }
  T_relu[(((((((((int)blockIdx.z) * 50176) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 1792)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 4) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_avg_pool2d_add_nn_relu_2_kernel0(float* __restrict__ placeholder, float* __restrict__ T_relu, float* __restrict__ placeholder1) {
  float tensor[1];
  tensor[(0)] = 0.000000e+00f;
  for (int dh = 0; dh < 2; ++dh) {
    for (int dw = 0; dw < 2; ++dw) {
      tensor[(0)] = (tensor[(0)] + placeholder[((((((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) / 28) * 112) + (dh * 56)) + ((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) % 28) * 2)) + dw))]);
    }
  }
  T_relu[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = max(__ocml_fma_f32(tensor[(0)], 2.500000e-01f, placeholder1[((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) / 784))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_avg_pool2d_kernel0(float* __restrict__ placeholder, float* __restrict__ tensor) {
  float tensor1[1];
  tensor1[(0)] = 0.000000e+00f;
  for (int dh = 0; dh < 7; ++dh) {
    for (int dw = 0; dw < 7; ++dw) {
      if (((int)threadIdx.x) < 1) {
        tensor1[(0)] = (tensor1[(0)] + placeholder[((((((int)threadIdx.x) * 49) + (dh * 7)) + dw))]);
      }
    }
  }
  if (((int)threadIdx.x) < 1) {
    tensor[(((int)threadIdx.x))] = (tensor1[(0)] * 2.040816e-02f);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_6_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[1];
  __shared__ float pad_temp_shared[384];
  __shared__ float placeholder_shared[36];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 7; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 96) + ((int)threadIdx.y)) < 4) {
        if (((((int)threadIdx.y) * 6) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 4)) < 24) {
          if ((((((int)threadIdx.y) * 96) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 384) {
            if (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 96) {
              pad_temp_shared[((((((int)threadIdx.y) * 96) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 4))) && (((((int)blockIdx.y) * 4) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 15)))) && (((((int)blockIdx.x) * 14) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 15)) < 29)) ? placeholder[((((((((rc_outer * 3136) + (((int)threadIdx.y) * 784)) + (((int)blockIdx.y) * 112)) + ((((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) & 15)) - 29))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((int)threadIdx.x) / 9) + ((int)threadIdx.y)) < 4) {
      if (((((int)threadIdx.y) * 3) + (((int)threadIdx.x) / 3)) < 12) {
        if (((((int)threadIdx.y) * 9) + ((int)threadIdx.x)) < 36) {
          if (((int)threadIdx.x) < 9) {
            placeholder_shared[(((((int)threadIdx.y) * 9) + ((int)threadIdx.x)))] = placeholder1[((((rc_outer * 36) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)))];
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 4; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((rc_inner * 96) + (((int)threadIdx.y) * 16)) + (ry_inner * 16)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[((((rc_inner * 9) + (ry_inner * 3)) + rx_inner))], compute[(0)]);
        }
      }
    }
  }
  T_relu[(((((((int)blockIdx.y) * 112) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(0)]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_max_pool2d_add_nn_relu_kernel0(float* __restrict__ placeholder, float* __restrict__ T_relu, float* __restrict__ placeholder1) {
  float tensor[1];
  tensor[(0)] = -3.402823e+38f;
  for (int dh = 0; dh < 3; ++dh) {
    for (int dw = 0; dw < 3; ++dw) {
      tensor[(0)] = max(tensor[(0)], (((1 <= ((((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) % 3136) / 56) * 2) + dh)) && (1 <= (((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) % 56) * 2) + dw))) ? placeholder[(((((((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) / 56) * 224) + (dh * 112)) + ((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) % 56) * 2)) + dw) - 113))] : -3.402823e+38f));
    }
  }
  T_relu[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = max((tensor[(0)] + placeholder1[((((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) / 3136))]), 0.000000e+00f);
}

