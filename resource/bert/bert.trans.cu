#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define CU_NUM 60

__device__ __forceinline__ bool is_first_thread() {
  return threadIdx.x == 0;
}

__device__ __forceinline__ unsigned int get_cu_id() {
  return blockIdx.x % CU_NUM;
}

__device__ __forceinline__ dim3 get_3d_idx(int idx, dim3 dim) {
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

__device__ void fused_nn_softmax_1_kernel0_device(float* __restrict__ T_softmax_maxelem, float* __restrict__ placeholder){
  if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 5760) {
    T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = -3.402823e+38f;
  }
  for (int k = 0; k < 480; ++k) {
    if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 5760) {
      T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = max(T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))], placeholder[((((((int)blockIdx.x) * 122880) + (((int)threadIdx.x) * 480)) + k))]);
    }
  }
}

__device__ void fused_reshape_add_add_kernel0_device(float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_add[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = ((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]) + placeholder2[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))]);
    }
  }
}

__device__ void fused_nn_batch_matmul_4_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute){
  float compute_local[8];
  __shared__ float placeholder_shared[720];
  __shared__ float placeholder_d_shared[300];
  float placeholder_shared_local[2];
  float placeholder_d_shared_local[4];
  for (int i_c_init = 0; i_c_init < 2; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 4; ++j_c_init) {
      compute_local[(((i_c_init * 4) + j_c_init))] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 32; ++k_outer) {
    __syncthreads();
    placeholder_shared[(((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)))] = placeholder[((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 1))] = placeholder[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 1))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 2))] = placeholder[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 2))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 15))] = placeholder[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 480))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 16))] = placeholder[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 481))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 17))] = placeholder[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 482))];
    if (((int)threadIdx.y) < 20) {
      placeholder_d_shared[(((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 3)))] = placeholder1[((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 9600)) + (((int)threadIdx.y) * 480)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)))];
    }
    if (((int)threadIdx.y) < 20) {
      placeholder_d_shared[((((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 3)) + 1))] = placeholder1[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 9600)) + (((int)threadIdx.y) * 480)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 1))];
    }
    if (((int)threadIdx.y) < 20) {
      placeholder_d_shared[((((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 3)) + 2))] = placeholder1[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 9600)) + (((int)threadIdx.y) * 480)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 2))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 15; ++k_inner) {
      placeholder_shared_local[(0)] = placeholder_shared[(((((int)threadIdx.y) * 30) + k_inner))];
      placeholder_shared_local[(1)] = placeholder_shared[((((((int)threadIdx.y) * 30) + k_inner) + 15))];
      placeholder_d_shared_local[(0)] = placeholder_d_shared[(((((int)threadIdx.x) * 60) + k_inner))];
      placeholder_d_shared_local[(1)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 15))];
      placeholder_d_shared_local[(2)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 30))];
      placeholder_d_shared_local[(3)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 45))];
      compute_local[(0)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(0)], compute_local[(0)]);
      compute_local[(1)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(1)], compute_local[(1)]);
      compute_local[(2)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(2)], compute_local[(2)]);
      compute_local[(3)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(3)], compute_local[(3)]);
      compute_local[(4)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(0)], compute_local[(4)]);
      compute_local[(5)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(1)], compute_local[(5)]);
      compute_local[(6)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(2)], compute_local[(6)]);
      compute_local[(7)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(3)], compute_local[(7)]);
    }
  }
  compute[((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 1))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 2))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 3))] = compute_local[(3)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 40))] = compute_local[(4)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 41))] = compute_local[(5)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 42))] = compute_local[(6)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 43))] = compute_local[(7)];
}

__device__ void fused_nn_batch_matmul_5_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute){
  float compute_local[15];
  __shared__ float placeholder_shared[800];
  __shared__ float placeholder_d_shared[480];
  float placeholder_shared_local[5];
  float placeholder_d_shared_local[3];
  for (int i_c_init = 0; i_c_init < 5; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 3; ++j_c_init) {
      compute_local[(((i_c_init * 3) + j_c_init))] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 4; ++k_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.y) * 50) + ((int)threadIdx.x)))] = placeholder[((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 10))] = placeholder[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 40))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 20))] = placeholder[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 80))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 30))] = placeholder[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 120))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 40))] = placeholder[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 160))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_d_shared[(((((int)threadIdx.y) * 30) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 1920)) + (((int)threadIdx.y) * 120)) + (k_outer * 10)) + ((int)threadIdx.x)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_d_shared[((((((int)threadIdx.y) * 30) + ((int)threadIdx.x)) + 10))] = placeholder1[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 1920)) + (((int)threadIdx.y) * 120)) + (k_outer * 10)) + ((int)threadIdx.x)) + 40))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_d_shared[((((((int)threadIdx.y) * 30) + ((int)threadIdx.x)) + 20))] = placeholder1[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 1920)) + (((int)threadIdx.y) * 120)) + (k_outer * 10)) + ((int)threadIdx.x)) + 80))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 10; ++k_inner) {
      placeholder_shared_local[(0)] = placeholder_shared[(((((int)threadIdx.y) * 50) + k_inner))];
      placeholder_shared_local[(1)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 10))];
      placeholder_shared_local[(2)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 20))];
      placeholder_shared_local[(3)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 30))];
      placeholder_shared_local[(4)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 40))];
      placeholder_d_shared_local[(0)] = placeholder_d_shared[(((((int)threadIdx.x) * 30) + k_inner))];
      placeholder_d_shared_local[(1)] = placeholder_d_shared[((((((int)threadIdx.x) * 30) + k_inner) + 10))];
      placeholder_d_shared_local[(2)] = placeholder_d_shared[((((((int)threadIdx.x) * 30) + k_inner) + 20))];
      compute_local[(0)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(0)], compute_local[(0)]);
      compute_local[(1)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(1)], compute_local[(1)]);
      compute_local[(2)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(2)], compute_local[(2)]);
      compute_local[(3)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(0)], compute_local[(3)]);
      compute_local[(4)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(1)], compute_local[(4)]);
      compute_local[(5)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(2)], compute_local[(5)]);
      compute_local[(6)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(0)], compute_local[(6)]);
      compute_local[(7)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(1)], compute_local[(7)]);
      compute_local[(8)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(2)], compute_local[(8)]);
      compute_local[(9)] = __ocml_fma_f32(placeholder_shared_local[(3)], placeholder_d_shared_local[(0)], compute_local[(9)]);
      compute_local[(10)] = __ocml_fma_f32(placeholder_shared_local[(3)], placeholder_d_shared_local[(1)], compute_local[(10)]);
      compute_local[(11)] = __ocml_fma_f32(placeholder_shared_local[(3)], placeholder_d_shared_local[(2)], compute_local[(11)]);
      compute_local[(12)] = __ocml_fma_f32(placeholder_shared_local[(4)], placeholder_d_shared_local[(0)], compute_local[(12)]);
      compute_local[(13)] = __ocml_fma_f32(placeholder_shared_local[(4)], placeholder_d_shared_local[(1)], compute_local[(13)]);
      compute_local[(14)] = __ocml_fma_f32(placeholder_shared_local[(4)], placeholder_d_shared_local[(2)], compute_local[(14)]);
    }
  }
  compute[((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 2))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 480))] = compute_local[(3)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 481))] = compute_local[(4)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 482))] = compute_local[(5)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 960))] = compute_local[(6)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 961))] = compute_local[(7)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 962))] = compute_local[(8)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1440))] = compute_local[(9)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1441))] = compute_local[(10)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1442))] = compute_local[(11)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1920))] = compute_local[(12)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1921))] = compute_local[(13)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1922))] = compute_local[(14)];
}

__device__ void fused_nn_softmax_1_kernel1_device(float* __restrict__ T_softmax_exp, float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem){
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 43; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    if ((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_softmax_exp[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = __ocml_exp_f32((placeholder[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] - T_softmax_maxelem[(((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]));
    }
  }
}

__device__ void fused_reshape_add_multiply_erf_multiply_add_multiply_reshape_kernel0_device(float* __restrict__ T_reshape, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = ((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]) * __ocml_fma_f32(__ocml_erf_f32(((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]) * 7.071068e-01f)), 5.000000e-01f, 5.000000e-01f));
    }
  }
}

__device__ void fused_mean_1_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder_red){
  float placeholder_red_rf[1];
  __shared__ float red_buf0[1024];
  placeholder_red_rf[(0)] = 0.000000e+00f;
  for (int k2_outer = 0; k2_outer < 15; ++k2_outer) {
    placeholder_red_rf[(0)] = (placeholder_red_rf[(0)] + placeholder[(((((((int)blockIdx.x) * 15360) + (((int)threadIdx.y) * 480)) + (k2_outer * 32)) + ((int)threadIdx.x)))]);
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = placeholder_red_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 16))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 8))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 4))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 2))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    placeholder_red[(((((int)blockIdx.x) * 32) + ((int)threadIdx.y)))] = ((volatile float*)red_buf0)[((((int)threadIdx.y) * 32))];
  }
}

__device__ void fused_subtract_add_sqrt_divide_multiply_add_1_kernel0_device(float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_add[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = __ocml_fma_f32(((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] - placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]) / __ocml_sqrt_f32((placeholder2[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))] + 1.000000e-12f))), placeholder3[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480))], placeholder4[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480))]);
    }
  }
}

__device__ void fused_nn_batch_matmul_3_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute){
  float compute_local[9];
  __shared__ float placeholder_shared[2400];
  __shared__ float placeholder_d_shared[300];
  float placeholder_shared_local[3];
  float placeholder_d_shared_local[3];
  for (int i_c_init = 0; i_c_init < 3; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 3; ++j_c_init) {
      compute_local[(((i_c_init * 3) + j_c_init))] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 24; ++k_outer) {
    __syncthreads();
    placeholder_shared[(((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 1))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 2))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 3))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 20))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 480))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 21))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 481))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 22))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 482))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 23))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 483))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 40))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 960))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 41))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 961))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 42))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 962))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 43))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 963))];
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))] = placeholder1[(((((((int)blockIdx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)))];
    }
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 1))] = placeholder1[((((((((int)blockIdx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 1))];
    }
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 2))] = placeholder1[((((((((int)blockIdx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 2))];
    }
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 3))] = placeholder1[((((((((int)blockIdx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 3))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 20; ++k_inner) {
      placeholder_shared_local[(0)] = placeholder_shared[(((((int)threadIdx.y) * 60) + k_inner))];
      placeholder_shared_local[(1)] = placeholder_shared[((((((int)threadIdx.y) * 60) + k_inner) + 20))];
      placeholder_shared_local[(2)] = placeholder_shared[((((((int)threadIdx.y) * 60) + k_inner) + 40))];
      placeholder_d_shared_local[(0)] = placeholder_d_shared[(((((int)threadIdx.x) * 60) + k_inner))];
      placeholder_d_shared_local[(1)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 20))];
      placeholder_d_shared_local[(2)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 40))];
      compute_local[(0)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(0)], compute_local[(0)]);
      compute_local[(1)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(1)], compute_local[(1)]);
      compute_local[(2)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(2)], compute_local[(2)]);
      compute_local[(3)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(0)], compute_local[(3)]);
      compute_local[(4)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(1)], compute_local[(4)]);
      compute_local[(5)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(2)], compute_local[(5)]);
      compute_local[(6)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(0)], compute_local[(6)]);
      compute_local[(7)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(1)], compute_local[(7)]);
      compute_local[(8)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(2)], compute_local[(8)]);
    }
  }
  compute[(((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 1))] = compute_local[(1)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 2))] = compute_local[(2)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 480))] = compute_local[(3)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 481))] = compute_local[(4)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 482))] = compute_local[(5)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 960))] = compute_local[(6)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 961))] = compute_local[(7)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 962))] = compute_local[(8)];
}

__device__ void fused_reshape_5_kernel0_device(float* __restrict__ T_reshape, float* __restrict__ placeholder){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 43; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))];
    }
  }
}

__device__ void fused_reshape_transpose_copy_reshape_1_kernel0_device(float* __restrict__ T_reshape, float* __restrict__ placeholder){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480) / 40) * 19200) + (((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480) * 40)) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 40)))];
    }
  }
}

__device__ void fused_reshape_add_reshape_transpose_reshape_transpose_kernel0_device(float* __restrict__ T_transpose, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_transpose[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = (placeholder[(((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480) * 480) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480))]);
    }
  }
}

__device__ void fused_reshape_add_reshape_transpose_transpose_reshape_transpose_kernel0_device(float* __restrict__ T_transpose, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_transpose[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = (placeholder[(((((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40) * 480) + (((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 19200) * 40)) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 40)))] + placeholder1[((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40))]);
    }
  }
}

__device__ void fused_reshape_add_reshape_transpose_divide_reshape_kernel0_device(float* __restrict__ T_reshape, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = ((placeholder[(((((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40) * 480) + (((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 19200) * 40)) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 40)))] + placeholder1[((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40))]) * 1.581139e-01f);
    }
  }
}

__device__ void fused_full_equal_reshape_kernel0_device(signed char* __restrict__ T_reshape){
  if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 480) {
    T_reshape[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = (signed char)0;
  }
}

__device__ void fused_cast_take_broadcast_to_like_cast_take_add_1_kernel0_device(float* __restrict__ T_add, float* __restrict__ placeholder, long* __restrict__ placeholder1, float* __restrict__ placeholder2, long* __restrict__ placeholder3){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_add[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = (placeholder[(((min(max(0, ((int)placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))])), 30521) * 480) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480)))] + placeholder2[(((min(max(0, ((int)placeholder3[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))])), 1023) * 480) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480)))]);
    }
  }
}

__device__ void fused_nn_softmax_1_kernel2_device(float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_exp){
  if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 5760) {
    T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = 0.000000e+00f;
  }
  for (int k = 0; k < 480; ++k) {
    if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 5760) {
      T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = (T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] + T_softmax_exp[((((((int)blockIdx.x) * 122880) + (((int)threadIdx.x) * 480)) + k))]);
    }
  }
}

__device__ void fused_mean_1_kernel1_device(float* __restrict__ T_divide, float* __restrict__ placeholder_red){
  if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 480) {
    T_divide[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = (placeholder_red[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] * 2.083333e-03f);
  }
}

__device__ void fused_reshape_4_kernel0_device(float* __restrict__ T_reshape, float* __restrict__ placeholder){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))];
    }
  }
}

__device__ void fused_variance_1_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_multiply_red){
  float T_multiply_red_rf[1];
  __shared__ float red_buf0[1024];
  T_multiply_red_rf[(0)] = 0.000000e+00f;
  for (int k2_outer = 0; k2_outer < 15; ++k2_outer) {
    T_multiply_red_rf[(0)] = __ocml_fma_f32((placeholder[(((((((int)blockIdx.x) * 15360) + (((int)threadIdx.y) * 480)) + (k2_outer * 32)) + ((int)threadIdx.x)))] - placeholder1[(((((int)blockIdx.x) * 32) + ((int)threadIdx.y)))]), (placeholder[(((((((int)blockIdx.x) * 15360) + (((int)threadIdx.y) * 480)) + (k2_outer * 32)) + ((int)threadIdx.x)))] - placeholder1[(((((int)blockIdx.x) * 32) + ((int)threadIdx.y)))]), T_multiply_red_rf[(0)]);
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = T_multiply_red_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 16))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 8))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 4))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 2))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_multiply_red[(((((int)blockIdx.x) * 32) + ((int)threadIdx.y)))] = ((volatile float*)red_buf0)[((((int)threadIdx.y) * 32))];
  }
}

__device__ void fused_variance_1_kernel1_device(float* __restrict__ T_divide, float* __restrict__ T_multiply_red){
  if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 480) {
    T_divide[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = (T_multiply_red[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] * 2.083333e-03f);
  }
}

__device__ void fused_reshape_cast_broadcast_to_like_where_kernel0_device(float* __restrict__ T_where, signed char* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 43; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_where[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = ((((int)((bool)placeholder[(((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480))])) != 0) ? -__int_as_float(0x7f800000) : placeholder1[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))]);
    }
  }
}

__device__ void fused_nn_softmax_1_kernel3_device(float* __restrict__ T_softmax_norm, float* __restrict__ T_softmax_exp, float* __restrict__ T_softmax_maxelem){
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 43; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    if ((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_softmax_norm[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = (T_softmax_exp[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] / T_softmax_maxelem[(((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]);
    }
  }
}


extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_softmax_1_kernel0_device_wrapper(float* __restrict__ T_softmax_maxelem, float* __restrict__ placeholder) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 23 + blockIdx.z * 1 * 23 >= 23 * 1 * 1) return;
    fused_nn_softmax_1_kernel0_device((float* __restrict__)T_softmax_maxelem,(float* __restrict__)placeholder);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_reshape_add_add_kernel0_device_wrapper(float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_reshape_add_add_kernel0_device((float* __restrict__)T_add,(float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(61))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_batch_matmul_4_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 5 + threadIdx.z * 24 * 5 >= 5 * 24 * 1) return;
    // if (blockIdx.x + blockIdx.y * 2 + blockIdx.z * 10 * 2 >= 2 * 10 * 12) return;
    fused_nn_batch_matmul_4_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)compute);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(61))) __attribute__((amdgpu_num_sgpr(34))) void fused_nn_batch_matmul_5_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 16 + threadIdx.z * 16 * 16 >= 16 * 16 * 1) return;
    // if (blockIdx.x + blockIdx.y * 10 + blockIdx.z * 6 * 10 >= 10 * 6 * 12) return;
    fused_nn_batch_matmul_5_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)compute);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_softmax_1_kernel1_device_wrapper(float* __restrict__ T_softmax_exp, float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_nn_softmax_1_kernel1_device((float* __restrict__)T_softmax_exp,(float* __restrict__)placeholder,(float* __restrict__)T_softmax_maxelem);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_reshape_add_multiply_erf_multiply_add_multiply_reshape_kernel0_device_wrapper(float* __restrict__ T_reshape, float* __restrict__ placeholder, float* __restrict__ placeholder1) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_reshape_add_multiply_erf_multiply_add_multiply_reshape_kernel0_device((float* __restrict__)T_reshape,(float* __restrict__)placeholder,(float* __restrict__)placeholder1);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_mean_1_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder_red) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 32 + threadIdx.z * 32 * 32 >= 32 * 32 * 1) return;
    // if (blockIdx.x + blockIdx.y * 15 + blockIdx.z * 1 * 15 >= 15 * 1 * 1) return;
    fused_mean_1_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder_red);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_subtract_add_sqrt_divide_multiply_add_1_kernel0_device_wrapper(float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_subtract_add_sqrt_divide_multiply_add_1_kernel0_device((float* __restrict__)T_add,(float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)placeholder2,(float* __restrict__)placeholder3,(float* __restrict__)placeholder4);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(61))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_batch_matmul_3_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 5 + threadIdx.z * 40 * 5 >= 5 * 40 * 1) return;
    // if (blockIdx.x + blockIdx.y * 32 + blockIdx.z * 4 * 32 >= 32 * 4 * 1) return;
    fused_nn_batch_matmul_3_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)compute);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_reshape_5_kernel0_device_wrapper(float* __restrict__ T_reshape, float* __restrict__ placeholder) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_reshape_5_kernel0_device((float* __restrict__)T_reshape,(float* __restrict__)placeholder);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_reshape_transpose_copy_reshape_1_kernel0_device_wrapper(float* __restrict__ T_reshape, float* __restrict__ placeholder) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_reshape_transpose_copy_reshape_1_kernel0_device((float* __restrict__)T_reshape,(float* __restrict__)placeholder);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_reshape_add_reshape_transpose_reshape_transpose_kernel0_device_wrapper(float* __restrict__ T_transpose, float* __restrict__ placeholder, float* __restrict__ placeholder1) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_reshape_add_reshape_transpose_reshape_transpose_kernel0_device((float* __restrict__)T_transpose,(float* __restrict__)placeholder,(float* __restrict__)placeholder1);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_reshape_add_reshape_transpose_transpose_reshape_transpose_kernel0_device_wrapper(float* __restrict__ T_transpose, float* __restrict__ placeholder, float* __restrict__ placeholder1) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_reshape_add_reshape_transpose_transpose_reshape_transpose_kernel0_device((float* __restrict__)T_transpose,(float* __restrict__)placeholder,(float* __restrict__)placeholder1);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_reshape_add_reshape_transpose_divide_reshape_kernel0_device_wrapper(float* __restrict__ T_reshape, float* __restrict__ placeholder, float* __restrict__ placeholder1) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_reshape_add_reshape_transpose_divide_reshape_kernel0_device((float* __restrict__)T_reshape,(float* __restrict__)placeholder,(float* __restrict__)placeholder1);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_full_equal_reshape_kernel0_device_wrapper(signed char* __restrict__ T_reshape) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 2 + blockIdx.z * 1 * 2 >= 2 * 1 * 1) return;
    fused_full_equal_reshape_kernel0_device((signed char* __restrict__)T_reshape);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_cast_take_broadcast_to_like_cast_take_add_1_kernel0_device_wrapper(float* __restrict__ T_add, float* __restrict__ placeholder, long* __restrict__ placeholder1, float* __restrict__ placeholder2, long* __restrict__ placeholder3) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_cast_take_broadcast_to_like_cast_take_add_1_kernel0_device((float* __restrict__)T_add,(float* __restrict__)placeholder,(long* __restrict__)placeholder1,(float* __restrict__)placeholder2,(long* __restrict__)placeholder3);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_softmax_1_kernel2_device_wrapper(float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_exp) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 23 + blockIdx.z * 1 * 23 >= 23 * 1 * 1) return;
    fused_nn_softmax_1_kernel2_device((float* __restrict__)T_softmax_maxelem,(float* __restrict__)T_softmax_exp);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_mean_1_kernel1_device_wrapper(float* __restrict__ T_divide, float* __restrict__ placeholder_red) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 2 + blockIdx.z * 1 * 2 >= 2 * 1 * 1) return;
    fused_mean_1_kernel1_device((float* __restrict__)T_divide,(float* __restrict__)placeholder_red);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_reshape_4_kernel0_device_wrapper(float* __restrict__ T_reshape, float* __restrict__ placeholder) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_reshape_4_kernel0_device((float* __restrict__)T_reshape,(float* __restrict__)placeholder);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_variance_1_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_multiply_red) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 32 + threadIdx.z * 32 * 32 >= 32 * 32 * 1) return;
    // if (blockIdx.x + blockIdx.y * 15 + blockIdx.z * 1 * 15 >= 15 * 1 * 1) return;
    fused_variance_1_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_multiply_red);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_variance_1_kernel1_device_wrapper(float* __restrict__ T_divide, float* __restrict__ T_multiply_red) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 2 + blockIdx.z * 1 * 2 >= 2 * 1 * 1) return;
    fused_variance_1_kernel1_device((float* __restrict__)T_divide,(float* __restrict__)T_multiply_red);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_reshape_cast_broadcast_to_like_where_kernel0_device_wrapper(float* __restrict__ T_where, signed char* __restrict__ placeholder, float* __restrict__ placeholder1) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_reshape_cast_broadcast_to_like_where_kernel0_device((float* __restrict__)T_where,(signed char* __restrict__)placeholder,(float* __restrict__)placeholder1);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_softmax_1_kernel3_device_wrapper(float* __restrict__ T_softmax_norm, float* __restrict__ T_softmax_exp, float* __restrict__ T_softmax_maxelem) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 256 + blockIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    fused_nn_softmax_1_kernel3_device((float* __restrict__)T_softmax_norm,(float* __restrict__)T_softmax_exp,(float* __restrict__)T_softmax_maxelem);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  void fused_nn_softmax_1_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_reshape_add_add_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_batch_matmul_4_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_batch_matmul_5_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_softmax_1_kernel1(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_reshape_add_multiply_erf_multiply_add_multiply_reshape_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_mean_1_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_subtract_add_sqrt_divide_multiply_add_1_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_batch_matmul_3_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_reshape_5_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_reshape_transpose_copy_reshape_1_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_reshape_add_reshape_transpose_reshape_transpose_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_reshape_add_reshape_transpose_transpose_reshape_transpose_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_reshape_add_reshape_transpose_divide_reshape_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_full_equal_reshape_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_cast_take_broadcast_to_like_cast_take_add_1_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_softmax_1_kernel2(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_mean_1_kernel1(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_reshape_4_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_variance_1_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_variance_1_kernel1(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_reshape_cast_broadcast_to_like_where_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_softmax_1_kernel3(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_256_1_1(int idx) {
  dim3 dim(256, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_32_4_1(int idx) {
  dim3 dim(32, 4, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_23_1_1(int idx) {
  dim3 dim(23, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_2_1_1(int idx) {
  dim3 dim(2, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_32_32_1(int idx) {
  dim3 dim(32, 32, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_16_16_1(int idx) {
  dim3 dim(16, 16, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_2_10_12(int idx) {
  dim3 dim(2, 10, 12);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_5_24_1(int idx) {
  dim3 dim(5, 24, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_15_1_1(int idx) {
  dim3 dim(15, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_10_6_12(int idx) {
  dim3 dim(10, 6, 12);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_5_40_1(int idx) {
  dim3 dim(5, 40, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

__global__ void get_3d_idx_caller(int* buf) {
    dim3 task_idx;

    task_idx = get_3d_idx_256_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_32_4_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_23_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_2_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_32_32_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_16_16_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_2_10_12(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_5_24_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_15_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_10_6_12(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_5_40_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

}

#define CALL_FRAMEWORK(idx) \
extern "C" __global__ void call_framework_##idx(\
  void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,\
  void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,\
  int cu_partition) \
{\
  asm volatile(\
    "  s_load_dwordx2 s[14:15], s[4:5], 0x0\n"\
    "  s_waitcnt lgkmcnt(0)\n"\
    "  s_setpc_b64 s[14:15]\n"\
    "  s_endpgm\n"\
  );\
}

CALL_FRAMEWORK(1)
CALL_FRAMEWORK(2)
CALL_FRAMEWORK(3)
CALL_FRAMEWORK(4)
CALL_FRAMEWORK(5)
CALL_FRAMEWORK(6)
CALL_FRAMEWORK(7)
CALL_FRAMEWORK(8)
CALL_FRAMEWORK(9)
CALL_FRAMEWORK(10)

#define MERGE_FRAMEWORK(idx) \
extern "C" __global__ void merge_framework_##idx(\
  void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,\
  void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,\
  int cu_partition) \
{\
  asm volatile(\
    "  s_load_dword s10, s[4:5], 0x40\n"\
    "  s_load_dwordx2 s[12:13], s[4:5], 0x0\n"\
    "  s_load_dwordx2 s[14:15], s[4:5], 0x20\n"\
    "  s_mul_hi_u32 s11, s6, 0x88888889\n"\
    "  s_lshr_b32 s11, s11, 5\n"\
    "  s_mul_i32 s11, s11, 60\n"\
    "  s_sub_i32 s11, s6, s11\n"\
    "  s_waitcnt lgkmcnt(0)\n"\
    "  s_cmp_ge_u32 s11, s10\n"\
    "  s_mov_b64 s[10:11], -1\n"\
    "  s_cbranch_scc1 MyBB"#idx"_3\n"\
    "; %bb.1:                                ; %Flow\n"\
    "  s_andn2_b64 vcc, exec, s[10:11]\n"\
    "  s_cbranch_vccz MyBB"#idx"_4\n"\
    "  s_endpgm\n"\
    "MyBB"#idx"_3:\n"\
    "  s_setpc_b64 s[14:15]\n"\
    "  s_endpgm\n"\
    "MyBB"#idx"_4:\n"\
    "  s_setpc_b64 s[12:13]\n"\
    "  s_endpgm\n"\
  );\
}
MERGE_FRAMEWORK(1)
MERGE_FRAMEWORK(2)
MERGE_FRAMEWORK(3)
MERGE_FRAMEWORK(4)
MERGE_FRAMEWORK(5)
MERGE_FRAMEWORK(6)
MERGE_FRAMEWORK(7)
MERGE_FRAMEWORK(8)
MERGE_FRAMEWORK(9)
MERGE_FRAMEWORK(10)
MERGE_FRAMEWORK(nostack_1)
MERGE_FRAMEWORK(nostack_2)
MERGE_FRAMEWORK(nostack_3)
MERGE_FRAMEWORK(nostack_4)
MERGE_FRAMEWORK(nostack_5)
MERGE_FRAMEWORK(nostack_6)
MERGE_FRAMEWORK(nostack_7)
MERGE_FRAMEWORK(nostack_8)
MERGE_FRAMEWORK(nostack_9)
MERGE_FRAMEWORK(nostack_10)
