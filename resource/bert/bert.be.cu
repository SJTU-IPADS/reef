#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__device__ void fused_nn_softmax_1_kernel0_device(float* __restrict__ T_softmax_maxelem, float* __restrict__ placeholder){
  if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 5760) {
    T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = -3.402823e+38f;
  }
  for (int k = 0; k < 480; ++k) {
    if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 5760) {
      T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = max(T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))], placeholder[((((((int)blockIdx.x) * 122880) + (((int)threadIdx.x) * 480)) + k))]);
    }
  }
}

__device__ void fused_reshape_add_add_kernel0_device(float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_add[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = ((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]) + placeholder2[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))]);
    }
  }
}

__device__ void fused_nn_batch_matmul_4_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute){
  float compute_local[8];
  __shared__ float placeholder_shared[720];
  __shared__ float placeholder_d_shared[300];
  float placeholder_shared_local[2];
  float placeholder_d_shared_local[4];
  for (int i_c_init = 0; i_c_init < 2; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 4; ++j_c_init) {
      compute_local[(((i_c_init * 4) + j_c_init))] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 32; ++k_outer) {
    __syncthreads();
    placeholder_shared[(((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)))] = placeholder[((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 1))] = placeholder[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 1))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 2))] = placeholder[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 2))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 15))] = placeholder[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 480))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 16))] = placeholder[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 481))];
    placeholder_shared[((((((int)threadIdx.y) * 30) + (((int)threadIdx.x) * 3)) + 17))] = placeholder[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 23040)) + (((int)threadIdx.y) * 960)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 482))];
    if (((int)threadIdx.y) < 20) {
      placeholder_d_shared[(((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 3)))] = placeholder1[((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 9600)) + (((int)threadIdx.y) * 480)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)))];
    }
    if (((int)threadIdx.y) < 20) {
      placeholder_d_shared[((((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 3)) + 1))] = placeholder1[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 9600)) + (((int)threadIdx.y) * 480)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 1))];
    }
    if (((int)threadIdx.y) < 20) {
      placeholder_d_shared[((((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 3)) + 2))] = placeholder1[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 9600)) + (((int)threadIdx.y) * 480)) + (k_outer * 15)) + (((int)threadIdx.x) * 3)) + 2))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 15; ++k_inner) {
      placeholder_shared_local[(0)] = placeholder_shared[(((((int)threadIdx.y) * 30) + k_inner))];
      placeholder_shared_local[(1)] = placeholder_shared[((((((int)threadIdx.y) * 30) + k_inner) + 15))];
      placeholder_d_shared_local[(0)] = placeholder_d_shared[(((((int)threadIdx.x) * 60) + k_inner))];
      placeholder_d_shared_local[(1)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 15))];
      placeholder_d_shared_local[(2)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 30))];
      placeholder_d_shared_local[(3)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 45))];
      compute_local[(0)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(0)], compute_local[(0)]);
      compute_local[(1)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(1)], compute_local[(1)]);
      compute_local[(2)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(2)], compute_local[(2)]);
      compute_local[(3)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(3)], compute_local[(3)]);
      compute_local[(4)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(0)], compute_local[(4)]);
      compute_local[(5)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(1)], compute_local[(5)]);
      compute_local[(6)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(2)], compute_local[(6)]);
      compute_local[(7)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(3)], compute_local[(7)]);
    }
  }
  compute[((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 1))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 2))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 3))] = compute_local[(3)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 40))] = compute_local[(4)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 41))] = compute_local[(5)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 42))] = compute_local[(6)];
  compute[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 1920)) + (((int)threadIdx.y) * 80)) + (((int)blockIdx.x) * 20)) + (((int)threadIdx.x) * 4)) + 43))] = compute_local[(7)];
}

__device__ void fused_nn_batch_matmul_5_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute){
  float compute_local[15];
  __shared__ float placeholder_shared[800];
  __shared__ float placeholder_d_shared[480];
  float placeholder_shared_local[5];
  float placeholder_d_shared_local[3];
  for (int i_c_init = 0; i_c_init < 5; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 3; ++j_c_init) {
      compute_local[(((i_c_init * 3) + j_c_init))] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 4; ++k_outer) {
    __syncthreads();
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[(((((int)threadIdx.y) * 50) + ((int)threadIdx.x)))] = placeholder[((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 10))] = placeholder[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 40))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 20))] = placeholder[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 80))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 30))] = placeholder[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 120))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_shared[((((((int)threadIdx.y) * 50) + ((int)threadIdx.x)) + 40))] = placeholder[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.y) * 3200)) + (((int)threadIdx.y) * 200)) + (k_outer * 10)) + ((int)threadIdx.x)) + 160))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_d_shared[(((((int)threadIdx.y) * 30) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 1920)) + (((int)threadIdx.y) * 120)) + (k_outer * 10)) + ((int)threadIdx.x)))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_d_shared[((((((int)threadIdx.y) * 30) + ((int)threadIdx.x)) + 10))] = placeholder1[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 1920)) + (((int)threadIdx.y) * 120)) + (k_outer * 10)) + ((int)threadIdx.x)) + 40))];
    }
    if (((int)threadIdx.x) < 10) {
      placeholder_d_shared[((((((int)threadIdx.y) * 30) + ((int)threadIdx.x)) + 20))] = placeholder1[(((((((((int)blockIdx.z) * 19200) + (((int)blockIdx.x) * 1920)) + (((int)threadIdx.y) * 120)) + (k_outer * 10)) + ((int)threadIdx.x)) + 80))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 10; ++k_inner) {
      placeholder_shared_local[(0)] = placeholder_shared[(((((int)threadIdx.y) * 50) + k_inner))];
      placeholder_shared_local[(1)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 10))];
      placeholder_shared_local[(2)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 20))];
      placeholder_shared_local[(3)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 30))];
      placeholder_shared_local[(4)] = placeholder_shared[((((((int)threadIdx.y) * 50) + k_inner) + 40))];
      placeholder_d_shared_local[(0)] = placeholder_d_shared[(((((int)threadIdx.x) * 30) + k_inner))];
      placeholder_d_shared_local[(1)] = placeholder_d_shared[((((((int)threadIdx.x) * 30) + k_inner) + 10))];
      placeholder_d_shared_local[(2)] = placeholder_d_shared[((((((int)threadIdx.x) * 30) + k_inner) + 20))];
      compute_local[(0)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(0)], compute_local[(0)]);
      compute_local[(1)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(1)], compute_local[(1)]);
      compute_local[(2)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(2)], compute_local[(2)]);
      compute_local[(3)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(0)], compute_local[(3)]);
      compute_local[(4)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(1)], compute_local[(4)]);
      compute_local[(5)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(2)], compute_local[(5)]);
      compute_local[(6)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(0)], compute_local[(6)]);
      compute_local[(7)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(1)], compute_local[(7)]);
      compute_local[(8)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(2)], compute_local[(8)]);
      compute_local[(9)] = __ocml_fma_f32(placeholder_shared_local[(3)], placeholder_d_shared_local[(0)], compute_local[(9)]);
      compute_local[(10)] = __ocml_fma_f32(placeholder_shared_local[(3)], placeholder_d_shared_local[(1)], compute_local[(10)]);
      compute_local[(11)] = __ocml_fma_f32(placeholder_shared_local[(3)], placeholder_d_shared_local[(2)], compute_local[(11)]);
      compute_local[(12)] = __ocml_fma_f32(placeholder_shared_local[(4)], placeholder_d_shared_local[(0)], compute_local[(12)]);
      compute_local[(13)] = __ocml_fma_f32(placeholder_shared_local[(4)], placeholder_d_shared_local[(1)], compute_local[(13)]);
      compute_local[(14)] = __ocml_fma_f32(placeholder_shared_local[(4)], placeholder_d_shared_local[(2)], compute_local[(14)]);
    }
  }
  compute[((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 2))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 480))] = compute_local[(3)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 481))] = compute_local[(4)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 482))] = compute_local[(5)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 960))] = compute_local[(6)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 961))] = compute_local[(7)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 962))] = compute_local[(8)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1440))] = compute_local[(9)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1441))] = compute_local[(10)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1442))] = compute_local[(11)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1920))] = compute_local[(12)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1921))] = compute_local[(13)];
  compute[(((((((((int)blockIdx.z) * 230400) + (((int)blockIdx.y) * 38400)) + (((int)threadIdx.y) * 2400)) + (((int)blockIdx.x) * 48)) + (((int)threadIdx.x) * 3)) + 1922))] = compute_local[(14)];
}

__device__ void fused_nn_softmax_1_kernel1_device(float* __restrict__ T_softmax_exp, float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem){
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 43; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    if ((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_softmax_exp[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = __ocml_exp_f32((placeholder[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] - T_softmax_maxelem[(((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]));
    }
  }
}

__device__ void fused_reshape_add_multiply_erf_multiply_add_multiply_reshape_kernel0_device(float* __restrict__ T_reshape, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = ((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]) * __ocml_fma_f32(__ocml_erf_f32(((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]) * 7.071068e-01f)), 5.000000e-01f, 5.000000e-01f));
    }
  }
}

__device__ void fused_mean_1_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder_red){
  float placeholder_red_rf[1];
  __shared__ float red_buf0[1024];
  placeholder_red_rf[(0)] = 0.000000e+00f;
  for (int k2_outer = 0; k2_outer < 15; ++k2_outer) {
    placeholder_red_rf[(0)] = (placeholder_red_rf[(0)] + placeholder[(((((((int)blockIdx.x) * 15360) + (((int)threadIdx.y) * 480)) + (k2_outer * 32)) + ((int)threadIdx.x)))]);
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = placeholder_red_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 16))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 8))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 4))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 2))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    placeholder_red[(((((int)blockIdx.x) * 32) + ((int)threadIdx.y)))] = ((volatile float*)red_buf0)[((((int)threadIdx.y) * 32))];
  }
}

__device__ void fused_subtract_add_sqrt_divide_multiply_add_1_kernel0_device(float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_add[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = __ocml_fma_f32(((placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] - placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]) / __ocml_sqrt_f32((placeholder2[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))] + 1.000000e-12f))), placeholder3[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480))], placeholder4[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480))]);
    }
  }
}

__device__ void fused_nn_batch_matmul_3_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute){
  float compute_local[9];
  __shared__ float placeholder_shared[2400];
  __shared__ float placeholder_d_shared[300];
  float placeholder_shared_local[3];
  float placeholder_d_shared_local[3];
  for (int i_c_init = 0; i_c_init < 3; ++i_c_init) {
    for (int j_c_init = 0; j_c_init < 3; ++j_c_init) {
      compute_local[(((i_c_init * 3) + j_c_init))] = 0.000000e+00f;
    }
  }
  for (int k_outer = 0; k_outer < 24; ++k_outer) {
    __syncthreads();
    placeholder_shared[(((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 1))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 2))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 3))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 20))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 480))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 21))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 481))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 22))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 482))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 23))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 483))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 40))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 960))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 41))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 961))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 42))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 962))];
    placeholder_shared[((((((int)threadIdx.y) * 60) + (((int)threadIdx.x) * 4)) + 43))] = placeholder[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 963))];
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))] = placeholder1[(((((((int)blockIdx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)))];
    }
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 1))] = placeholder1[((((((((int)blockIdx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 1))];
    }
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 2))] = placeholder1[((((((((int)blockIdx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 2))];
    }
    if (((int)threadIdx.y) < 15) {
      placeholder_d_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 3))] = placeholder1[((((((((int)blockIdx.x) * 7200) + (((int)threadIdx.y) * 480)) + (k_outer * 20)) + (((int)threadIdx.x) * 4)) + 3))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 20; ++k_inner) {
      placeholder_shared_local[(0)] = placeholder_shared[(((((int)threadIdx.y) * 60) + k_inner))];
      placeholder_shared_local[(1)] = placeholder_shared[((((((int)threadIdx.y) * 60) + k_inner) + 20))];
      placeholder_shared_local[(2)] = placeholder_shared[((((((int)threadIdx.y) * 60) + k_inner) + 40))];
      placeholder_d_shared_local[(0)] = placeholder_d_shared[(((((int)threadIdx.x) * 60) + k_inner))];
      placeholder_d_shared_local[(1)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 20))];
      placeholder_d_shared_local[(2)] = placeholder_d_shared[((((((int)threadIdx.x) * 60) + k_inner) + 40))];
      compute_local[(0)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(0)], compute_local[(0)]);
      compute_local[(1)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(1)], compute_local[(1)]);
      compute_local[(2)] = __ocml_fma_f32(placeholder_shared_local[(0)], placeholder_d_shared_local[(2)], compute_local[(2)]);
      compute_local[(3)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(0)], compute_local[(3)]);
      compute_local[(4)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(1)], compute_local[(4)]);
      compute_local[(5)] = __ocml_fma_f32(placeholder_shared_local[(1)], placeholder_d_shared_local[(2)], compute_local[(5)]);
      compute_local[(6)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(0)], compute_local[(6)]);
      compute_local[(7)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(1)], compute_local[(7)]);
      compute_local[(8)] = __ocml_fma_f32(placeholder_shared_local[(2)], placeholder_d_shared_local[(2)], compute_local[(8)]);
    }
  }
  compute[(((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 1))] = compute_local[(1)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 2))] = compute_local[(2)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 480))] = compute_local[(3)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 481))] = compute_local[(4)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 482))] = compute_local[(5)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 960))] = compute_local[(6)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 961))] = compute_local[(7)];
  compute[((((((((int)blockIdx.y) * 57600) + (((int)threadIdx.y) * 1440)) + (((int)blockIdx.x) * 15)) + (((int)threadIdx.x) * 3)) + 962))] = compute_local[(8)];
}

__device__ void fused_reshape_5_kernel0_device(float* __restrict__ T_reshape, float* __restrict__ placeholder){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 43; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))];
    }
  }
}

__device__ void fused_reshape_transpose_copy_reshape_1_kernel0_device(float* __restrict__ T_reshape, float* __restrict__ placeholder){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = placeholder[(((((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480) / 40) * 19200) + (((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480) * 40)) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 40)))];
    }
  }
}

__device__ void fused_reshape_add_reshape_transpose_reshape_transpose_kernel0_device(float* __restrict__ T_transpose, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_transpose[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = (placeholder[(((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480) * 480) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480)))] + placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480))]);
    }
  }
}

__device__ void fused_reshape_add_reshape_transpose_transpose_reshape_transpose_kernel0_device(float* __restrict__ T_transpose, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_transpose[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = (placeholder[(((((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40) * 480) + (((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 19200) * 40)) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 40)))] + placeholder1[((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40))]);
    }
  }
}

__device__ void fused_reshape_add_reshape_transpose_divide_reshape_kernel0_device(float* __restrict__ T_reshape, float* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = ((placeholder[(((((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40) * 480) + (((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 19200) * 40)) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 40)))] + placeholder1[((((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 19200) / 40))]) * 1.581139e-01f);
    }
  }
}

__device__ void fused_full_equal_reshape_kernel0_device(signed char* __restrict__ T_reshape){
  if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 480) {
    T_reshape[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = (signed char)0;
  }
}

__device__ void fused_cast_take_broadcast_to_like_cast_take_add_1_kernel0_device(float* __restrict__ T_add, float* __restrict__ placeholder, long* __restrict__ placeholder1, float* __restrict__ placeholder2, long* __restrict__ placeholder3){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_add[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = (placeholder[(((min(max(0, ((int)placeholder1[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))])), 30521) * 480) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480)))] + placeholder2[(((min(max(0, ((int)placeholder3[(((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))])), 1023) * 480) + ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480)))]);
    }
  }
}

__device__ void fused_nn_softmax_1_kernel2_device(float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_exp){
  if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 5760) {
    T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = 0.000000e+00f;
  }
  for (int k = 0; k < 480; ++k) {
    if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 5760) {
      T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = (T_softmax_maxelem[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] + T_softmax_exp[((((((int)blockIdx.x) * 122880) + (((int)threadIdx.x) * 480)) + k))]);
    }
  }
}

__device__ void fused_mean_1_kernel1_device(float* __restrict__ T_divide, float* __restrict__ placeholder_red){
  if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 480) {
    T_divide[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = (placeholder_red[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] * 2.083333e-03f);
  }
}

__device__ void fused_reshape_4_kernel0_device(float* __restrict__ T_reshape, float* __restrict__ placeholder){
  for (int ax0_ax1_fused_ax2_fused_outer = 0; ax0_ax1_fused_ax2_fused_outer < 4; ++ax0_ax1_fused_ax2_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 230400) {
      T_reshape[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = placeholder[((((ax0_ax1_fused_ax2_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))];
    }
  }
}

__device__ void fused_variance_1_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_multiply_red){
  float T_multiply_red_rf[1];
  __shared__ float red_buf0[1024];
  T_multiply_red_rf[(0)] = 0.000000e+00f;
  for (int k2_outer = 0; k2_outer < 15; ++k2_outer) {
    T_multiply_red_rf[(0)] = __ocml_fma_f32((placeholder[(((((((int)blockIdx.x) * 15360) + (((int)threadIdx.y) * 480)) + (k2_outer * 32)) + ((int)threadIdx.x)))] - placeholder1[(((((int)blockIdx.x) * 32) + ((int)threadIdx.y)))]), (placeholder[(((((((int)blockIdx.x) * 15360) + (((int)threadIdx.y) * 480)) + (k2_outer * 32)) + ((int)threadIdx.x)))] - placeholder1[(((((int)blockIdx.x) * 32) + ((int)threadIdx.y)))]), T_multiply_red_rf[(0)]);
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = T_multiply_red_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 16))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 8))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 4))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 2))]);
    ((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] = (((volatile float*)red_buf0)[(((((int)threadIdx.y) * 32) + ((int)threadIdx.x)))] + ((volatile float*)red_buf0)[((((((int)threadIdx.y) * 32) + ((int)threadIdx.x)) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    T_multiply_red[(((((int)blockIdx.x) * 32) + ((int)threadIdx.y)))] = ((volatile float*)red_buf0)[((((int)threadIdx.y) * 32))];
  }
}

__device__ void fused_variance_1_kernel1_device(float* __restrict__ T_divide, float* __restrict__ T_multiply_red){
  if (((((int)blockIdx.x) * 256) + ((int)threadIdx.x)) < 480) {
    T_divide[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = (T_multiply_red[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] * 2.083333e-03f);
  }
}

__device__ void fused_reshape_cast_broadcast_to_like_where_kernel0_device(float* __restrict__ T_where, signed char* __restrict__ placeholder, float* __restrict__ placeholder1){
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer < 43; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_where[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = ((((int)((bool)placeholder[(((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) % 480))])) != 0) ? -__int_as_float(0x7f800000) : placeholder1[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))]);
    }
  }
}

__device__ void fused_nn_softmax_1_kernel3_device(float* __restrict__ T_softmax_norm, float* __restrict__ T_softmax_exp, float* __restrict__ T_softmax_maxelem){
  for (int i0_i1_fused_i2_fused_i3_fused_outer = 0; i0_i1_fused_i2_fused_i3_fused_outer < 43; ++i0_i1_fused_i2_fused_i3_fused_outer) {
    if ((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) < 2764800) {
      T_softmax_norm[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] = (T_softmax_exp[((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)))] / T_softmax_maxelem[(((((i0_i1_fused_i2_fused_i3_fused_outer * 65536) + (((int)blockIdx.x) * 256)) + ((int)threadIdx.x)) / 480))]);
    }
  }
}


extern "C" __global__ void fused_nn_softmax_1_kernel0(int* preempted, int* task_slot, float* __restrict__ T_softmax_maxelem, float* __restrict__ placeholder) {
    if (*preempted) return;
    fused_nn_softmax_1_kernel0_device(T_softmax_maxelem, placeholder);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_reshape_add_add_kernel0(int* preempted, int* task_slot, float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2) {
    if (*preempted) return;
    fused_reshape_add_add_kernel0_device(T_add, placeholder, placeholder1, placeholder2);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_batch_matmul_4_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
    if (*preempted) return;
    fused_nn_batch_matmul_4_kernel0_device(placeholder, placeholder1, compute);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_batch_matmul_5_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
    if (*preempted) return;
    fused_nn_batch_matmul_5_kernel0_device(placeholder, placeholder1, compute);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_softmax_1_kernel1(int* preempted, int* task_slot, float* __restrict__ T_softmax_exp, float* __restrict__ placeholder, float* __restrict__ T_softmax_maxelem) {
    if (*preempted) return;
    fused_nn_softmax_1_kernel1_device(T_softmax_exp, placeholder, T_softmax_maxelem);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_reshape_add_multiply_erf_multiply_add_multiply_reshape_kernel0(int* preempted, int* task_slot, float* __restrict__ T_reshape, float* __restrict__ placeholder, float* __restrict__ placeholder1) {
    if (*preempted) return;
    fused_reshape_add_multiply_erf_multiply_add_multiply_reshape_kernel0_device(T_reshape, placeholder, placeholder1);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_mean_1_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder_red) {
    if (*preempted) return;
    fused_mean_1_kernel0_device(placeholder, placeholder_red);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_subtract_add_sqrt_divide_multiply_add_1_kernel0(int* preempted, int* task_slot, float* __restrict__ T_add, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ placeholder2, float* __restrict__ placeholder3, float* __restrict__ placeholder4) {
    if (*preempted) return;
    fused_subtract_add_sqrt_divide_multiply_add_1_kernel0_device(T_add, placeholder, placeholder1, placeholder2, placeholder3, placeholder4);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_batch_matmul_3_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ compute) {
    if (*preempted) return;
    fused_nn_batch_matmul_3_kernel0_device(placeholder, placeholder1, compute);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_reshape_5_kernel0(int* preempted, int* task_slot, float* __restrict__ T_reshape, float* __restrict__ placeholder) {
    if (*preempted) return;
    fused_reshape_5_kernel0_device(T_reshape, placeholder);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_reshape_transpose_copy_reshape_1_kernel0(int* preempted, int* task_slot, float* __restrict__ T_reshape, float* __restrict__ placeholder) {
    if (*preempted) return;
    fused_reshape_transpose_copy_reshape_1_kernel0_device(T_reshape, placeholder);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_reshape_add_reshape_transpose_reshape_transpose_kernel0(int* preempted, int* task_slot, float* __restrict__ T_transpose, float* __restrict__ placeholder, float* __restrict__ placeholder1) {
    if (*preempted) return;
    fused_reshape_add_reshape_transpose_reshape_transpose_kernel0_device(T_transpose, placeholder, placeholder1);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_reshape_add_reshape_transpose_transpose_reshape_transpose_kernel0(int* preempted, int* task_slot, float* __restrict__ T_transpose, float* __restrict__ placeholder, float* __restrict__ placeholder1) {
    if (*preempted) return;
    fused_reshape_add_reshape_transpose_transpose_reshape_transpose_kernel0_device(T_transpose, placeholder, placeholder1);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_reshape_add_reshape_transpose_divide_reshape_kernel0(int* preempted, int* task_slot, float* __restrict__ T_reshape, float* __restrict__ placeholder, float* __restrict__ placeholder1) {
    if (*preempted) return;
    fused_reshape_add_reshape_transpose_divide_reshape_kernel0_device(T_reshape, placeholder, placeholder1);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_full_equal_reshape_kernel0(int* preempted, int* task_slot, signed char* __restrict__ T_reshape) {
    if (*preempted) return;
    fused_full_equal_reshape_kernel0_device(T_reshape);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_cast_take_broadcast_to_like_cast_take_add_1_kernel0(int* preempted, int* task_slot, float* __restrict__ T_add, float* __restrict__ placeholder, long* __restrict__ placeholder1, float* __restrict__ placeholder2, long* __restrict__ placeholder3) {
    if (*preempted) return;
    fused_cast_take_broadcast_to_like_cast_take_add_1_kernel0_device(T_add, placeholder, placeholder1, placeholder2, placeholder3);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_softmax_1_kernel2(int* preempted, int* task_slot, float* __restrict__ T_softmax_maxelem, float* __restrict__ T_softmax_exp) {
    if (*preempted) return;
    fused_nn_softmax_1_kernel2_device(T_softmax_maxelem, T_softmax_exp);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_mean_1_kernel1(int* preempted, int* task_slot, float* __restrict__ T_divide, float* __restrict__ placeholder_red) {
    if (*preempted) return;
    fused_mean_1_kernel1_device(T_divide, placeholder_red);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_reshape_4_kernel0(int* preempted, int* task_slot, float* __restrict__ T_reshape, float* __restrict__ placeholder) {
    if (*preempted) return;
    fused_reshape_4_kernel0_device(T_reshape, placeholder);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_variance_1_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_multiply_red) {
    if (*preempted) return;
    fused_variance_1_kernel0_device(placeholder, placeholder1, T_multiply_red);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_variance_1_kernel1(int* preempted, int* task_slot, float* __restrict__ T_divide, float* __restrict__ T_multiply_red) {
    if (*preempted) return;
    fused_variance_1_kernel1_device(T_divide, T_multiply_red);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_reshape_cast_broadcast_to_like_where_kernel0(int* preempted, int* task_slot, float* __restrict__ T_where, signed char* __restrict__ placeholder, float* __restrict__ placeholder1) {
    if (*preempted) return;
    fused_reshape_cast_broadcast_to_like_where_kernel0_device(T_where, placeholder, placeholder1);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_softmax_1_kernel3(int* preempted, int* task_slot, float* __restrict__ T_softmax_norm, float* __restrict__ T_softmax_exp, float* __restrict__ T_softmax_maxelem) {
    if (*preempted) return;
    fused_nn_softmax_1_kernel3_device(T_softmax_norm, T_softmax_exp, T_softmax_maxelem);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        
