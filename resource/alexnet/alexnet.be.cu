#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

__device__ void fused_nn_max_pool2d_kernel0_device(float* __restrict__ placeholder, float* __restrict__ tensor){
  float tensor_local[1];
  tensor_local[(0)] = -3.402823e+38f;
  for (int dh = 0; dh < 3; ++dh) {
    for (int dw = 0; dw < 3; ++dw) {
      tensor_local[(0)] = max(tensor_local[(0)], placeholder[(((((((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) / 25) * 144) + (((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 25) / 5) * 24)) + (dh * 12)) + ((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 5) * 2)) + dw))]);
    }
  }
  tensor[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))] = tensor_local[(0)];
}

__device__ void fused_nn_softmax_kernel0_device(float* __restrict__ placeholder, float* __restrict__ T_softmax_norm){
  float normal_reduce_temp0[1];
  float red_buf0[1];
  float T_softmax_exp[16];
  float normal_reduce_temp01[1];
  float red_buf01[1];
  normal_reduce_temp0[(0)] = -3.402823e+38f;
  for (int k_inner = 0; k_inner < 16; ++k_inner) {
    if (((((int)threadIdx.x) * 16) + k_inner) < 1000) {
      normal_reduce_temp0[(0)] = max(normal_reduce_temp0[(0)], placeholder[(((((int)threadIdx.x) * 16) + k_inner))]);
    }
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = normal_reduce_temp0[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  for (int i1_inner_outer = 0; i1_inner_outer < 4; ++i1_inner_outer) {
    for (int i1_inner_inner_s = 0; i1_inner_inner_s < 4; ++i1_inner_inner_s) {
      if ((((((int)threadIdx.x) * 16) + (i1_inner_outer * 4)) + i1_inner_inner_s) < 1000) {
        T_softmax_exp[(((i1_inner_outer * 4) + i1_inner_inner_s))] = __ocml_exp_f32((placeholder[((((((int)threadIdx.x) * 16) + (i1_inner_outer * 4)) + i1_inner_inner_s))] - red_buf0[(0)]));
      }
    }
  }
  normal_reduce_temp01[(0)] = 0.000000e+00f;
  for (int k_inner1 = 0; k_inner1 < 16; ++k_inner1) {
    if (((((int)threadIdx.x) * 16) + k_inner1) < 1000) {
      normal_reduce_temp01[(0)] = (normal_reduce_temp01[(0)] + __hip_ds_bpermute(((((int)threadIdx.x) + (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63))) << 2), T_softmax_exp[(k_inner1)]));
    }
  }
  unsigned int mask1[1];
  float t01[1];
  red_buf01[(0)] = normal_reduce_temp01[(0)];
  ((int*)mask1)[(0)] = 0;
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  red_buf01[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf01[(0)]);
  for (int i1_inner_outer1 = 0; i1_inner_outer1 < 4; ++i1_inner_outer1) {
    for (int i1_inner_inner_s1 = 0; i1_inner_inner_s1 < 4; ++i1_inner_inner_s1) {
      if ((((((int)threadIdx.x) * 16) + (i1_inner_outer1 * 4)) + i1_inner_inner_s1) < 1000) {
        T_softmax_norm[((((((int)threadIdx.x) * 16) + (i1_inner_outer1 * 4)) + i1_inner_inner_s1))] = (__hip_ds_bpermute(((((int)threadIdx.x) + (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63))) << 2), T_softmax_exp[(((i1_inner_outer1 * 4) + i1_inner_inner_s1))]) / red_buf01[(0)]);
      }
    }
  }
}

__device__ void fused_nn_conv2d_nn_relu_3_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu){
  float compute[2];
  __shared__ float pad_temp_shared[36];
  __shared__ float placeholder_shared[200];
  for (int ff_init = 0; ff_init < 2; ++ff_init) {
    compute[(ff_init)] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 96; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 36) {
        if ((((((int)threadIdx.y) * 5) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 9) {
          if (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 5) {
            pad_temp_shared[(((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((2 <= ((((int)blockIdx.y) * 2) + (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 6))) && (((((int)blockIdx.y) * 2) + (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 6)) < 28)) && (2 <= ((((int)blockIdx.x) * 2) + (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)))) && (((((int)blockIdx.x) * 2) + (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)) < 28)) ? placeholder[(((((((rc_outer * 676) + (((int)blockIdx.y) * 52)) + ((((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 6) * 26)) + (((int)blockIdx.x) * 2)) + (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)) - 54))] : 0.000000e+00f);
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 13; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 25)) + ((int)threadIdx.y)) < 8) {
        if ((((((int)threadIdx.z) * 10) + (((int)threadIdx.y) * 5)) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 5)) < 40) {
          if (((((((int)threadIdx.z) * 50) + (((int)threadIdx.y) * 25)) + (((int)threadIdx.x) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 200) {
            if ((((((int)threadIdx.y) * 25) + (((int)threadIdx.x) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 50) {
              if (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 25) {
                placeholder_shared[(((((((int)threadIdx.z) * 50) + (((int)threadIdx.y) * 25)) + (((int)threadIdx.x) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = placeholder1[(((((((((int)blockIdx.z) * 19200) + (((int)threadIdx.z) * 4800)) + (((int)threadIdx.y) * 2400)) + (rc_outer * 25)) + (((int)threadIdx.x) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))];
              }
            }
          }
        }
      }
    }
    __syncthreads();
    for (int ry_inner = 0; ry_inner < 5; ++ry_inner) {
      for (int rx_inner = 0; rx_inner < 5; ++rx_inner) {
        for (int ff = 0; ff < 2; ++ff) {
          compute[(ff)] = __ocml_fma_f32(pad_temp_shared[(((((((int)threadIdx.y) * 6) + (ry_inner * 6)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[(((((((int)threadIdx.z) * 50) + (ff * 25)) + (ry_inner * 5)) + rx_inner))], compute[(ff)]);
        }
      }
    }
  }
  for (int ax1_inner_inner_inner = 0; ax1_inner_inner_inner < 2; ++ax1_inner_inner_inner) {
    T_relu[((((((((((int)blockIdx.z) * 5408) + (((int)threadIdx.z) * 1352)) + (ax1_inner_inner_inner * 676)) + (((int)blockIdx.y) * 52)) + (((int)threadIdx.y) * 26)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)))] = max(compute[(ax1_inner_inner_inner)], 0.000000e+00f);
  }
}

__device__ void fused_nn_conv2d_nn_relu_1_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu){
  float compute[1];
  __shared__ float pad_temp_shared[192];
  __shared__ float placeholder_shared[2304];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 48; ++rc_outer) {
    __syncthreads();
    if (((((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) / 6) + ((int)threadIdx.z)) < 32) {
      if ((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 192) {
        if (((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) < 6) {
          if (((int)threadIdx.x) < 3) {
            pad_temp_shared[((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))] = (((((1 <= ((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3))) && (((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3)) < 13)) && (1 <= (((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))) && ((((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 13)) ? placeholder[(((((((((rc_outer * 1152) + ((((int)threadIdx.z) >> 2) * 144)) + (((int)blockIdx.y) * 24)) + ((((int)threadIdx.z) & 3) * 12)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) - 13))] : 0.000000e+00f);
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      placeholder_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder1[(((((((((int)blockIdx.z) * 110592) + (((int)threadIdx.z) * 3456)) + (rc_outer * 72)) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((rc_inner * 24) + (((int)threadIdx.y) * 6)) + (ry_inner * 6)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[(((((((int)threadIdx.z) * 72) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner))], compute[(0)]);
        }
      }
    }
  }
  T_relu[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 144)) + (((int)blockIdx.y) * 24)) + (((int)threadIdx.y) * 12)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)))] = max(compute[(0)], 0.000000e+00f);
}

__device__ void fused_nn_dense_add_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2){
  float T_dense_rf[1];
  float red_buf0[1];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 64; ++k_outer) {
    T_dense_rf[(0)] = __ocml_fma_f32(placeholder[(((k_outer * 64) + ((int)threadIdx.x)))], placeholder1[((((((int)blockIdx.x) * 4096) + (k_outer * 64)) + ((int)threadIdx.x)))], T_dense_rf[(0)]);
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = T_dense_rf[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = red_buf0[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_add[(((int)blockIdx.x))] = (T_dense[(0)] + placeholder2[(((int)blockIdx.x))]);
  }
}

__device__ void fused_nn_max_pool2d_2_kernel0_device(float* __restrict__ placeholder, float* __restrict__ tensor){
  float tensor_local[1];
  tensor_local[(0)] = -3.402823e+38f;
  for (int dh = 0; dh < 3; ++dh) {
    for (int dw = 0; dw < 3; ++dw) {
      tensor_local[(0)] = max(tensor_local[(0)], placeholder[(((((((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) / 676) * 2916) + (((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 676) / 26) * 108)) + (dh * 54)) + ((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 26) * 2)) + dw))]);
    }
  }
  tensor[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))] = tensor_local[(0)];
}

__device__ void fused_nn_dense_add_nn_relu_1_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float T_dense_rf[1];
  float red_buf0[1];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 100; ++k_outer) {
    T_dense_rf[(0)] = __ocml_fma_f32(placeholder[(((k_outer * 64) + ((int)threadIdx.x)))], placeholder1[((((((int)blockIdx.x) * 6400) + (k_outer * 64)) + ((int)threadIdx.x)))], T_dense_rf[(0)]);
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = T_dense_rf[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = red_buf0[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_relu[(((int)blockIdx.x))] = max((T_dense[(0)] + placeholder2[(((int)blockIdx.x))]), 0.000000e+00f);
  }
}

__device__ void fused_nn_conv2d_nn_relu_4_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu){
  float compute[1];
  __shared__ float pad_temp_shared[315];
  __shared__ float placeholder_shared[24];
  compute[(0)] = 0.000000e+00f;
  for (int ry_outer = 0; ry_outer < 11; ++ry_outer) {
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 1))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 1))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 1))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 1))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 1))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 2))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 2))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 2))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 2))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 2))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 3))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 3))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 3))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 3))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 3))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 4))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 4))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 4))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 4))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 4))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 5))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 5))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 5))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 5))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 5))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 6))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 6))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 6))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 6))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 6))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 7))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 7))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 7))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 7))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 7))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 8))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 8))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 8))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 8))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 8))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 9))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 9))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 9))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 9))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 9))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 10))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 10))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 10))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 10))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 10))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
  }
  T_relu[(((((((((int)blockIdx.z) * 23328) + (((int)threadIdx.z) * 2916)) + (((int)blockIdx.y) * 324)) + (((int)threadIdx.y) * 54)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)))] = max(compute[(0)], 0.000000e+00f);
}

__device__ void fused_nn_max_pool2d_1_kernel0_device(float* __restrict__ placeholder, float* __restrict__ tensor){
  float tensor_local[1];
  tensor_local[(0)] = -3.402823e+38f;
  for (int dh = 0; dh < 3; ++dh) {
    for (int dw = 0; dw < 3; ++dw) {
      tensor_local[(0)] = max(tensor_local[(0)], placeholder[(((((((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) / 144) * 676) + (((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 144) / 12) * 52)) + (dh * 26)) + ((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 12) * 2)) + dw))]);
    }
  }
  tensor[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))] = tensor_local[(0)];
}

__device__ void fused_nn_conv2d_nn_relu_2_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu){
  float compute[1];
  __shared__ float pad_temp_shared[192];
  __shared__ float placeholder_shared[2304];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    if (((((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) / 6) + ((int)threadIdx.z)) < 32) {
      if ((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 192) {
        if (((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) < 6) {
          if (((int)threadIdx.x) < 3) {
            pad_temp_shared[((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))] = (((((1 <= ((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3))) && (((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3)) < 13)) && (1 <= (((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))) && ((((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 13)) ? placeholder[(((((((((rc_outer * 1152) + ((((int)threadIdx.z) >> 2) * 144)) + (((int)blockIdx.y) * 24)) + ((((int)threadIdx.z) & 3) * 12)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) - 13))] : 0.000000e+00f);
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      placeholder_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder1[(((((((((int)blockIdx.z) * 73728) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((rc_inner * 24) + (((int)threadIdx.y) * 6)) + (ry_inner * 6)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[(((((((int)threadIdx.z) * 72) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner))], compute[(0)]);
        }
      }
    }
  }
  T_relu[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 144)) + (((int)blockIdx.y) * 24)) + (((int)threadIdx.y) * 12)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)))] = max(compute[(0)], 0.000000e+00f);
}

__device__ void fused_nn_batch_flatten_kernel0_device(float* __restrict__ tensor, float* __restrict__ placeholder){
  tensor[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))] = placeholder[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))];
}

__device__ void fused_nn_conv2d_nn_relu_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu){
  float compute[1];
  __shared__ float pad_temp_shared[192];
  __shared__ float placeholder_shared[2304];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 48; ++rc_outer) {
    __syncthreads();
    if (((((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) / 6) + ((int)threadIdx.z)) < 32) {
      if ((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 192) {
        if (((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) < 6) {
          if (((int)threadIdx.x) < 3) {
            pad_temp_shared[((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))] = (((((1 <= ((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3))) && (((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3)) < 13)) && (1 <= (((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))) && ((((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 13)) ? placeholder[(((((((((rc_outer * 1152) + ((((int)threadIdx.z) >> 2) * 144)) + (((int)blockIdx.y) * 24)) + ((((int)threadIdx.z) & 3) * 12)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) - 13))] : 0.000000e+00f);
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      placeholder_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder1[(((((((((int)blockIdx.z) * 110592) + (((int)threadIdx.z) * 3456)) + (rc_outer * 72)) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((rc_inner * 24) + (((int)threadIdx.y) * 6)) + (ry_inner * 6)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[(((((((int)threadIdx.z) * 72) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner))], compute[(0)]);
        }
      }
    }
  }
  T_relu[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 144)) + (((int)blockIdx.y) * 24)) + (((int)threadIdx.y) * 12)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)))] = max(compute[(0)], 0.000000e+00f);
}

__device__ void fused_nn_dense_add_nn_relu_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float T_dense_rf[1];
  float red_buf0[1];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 64; ++k_outer) {
    T_dense_rf[(0)] = __ocml_fma_f32(placeholder[(((k_outer * 64) + ((int)threadIdx.x)))], placeholder1[((((((int)blockIdx.x) * 4096) + (k_outer * 64)) + ((int)threadIdx.x)))], T_dense_rf[(0)]);
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = T_dense_rf[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = red_buf0[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_relu[(((int)blockIdx.x))] = max((T_dense[(0)] + placeholder2[(((int)blockIdx.x))]), 0.000000e+00f);
  }
}


extern "C" __global__ void fused_nn_max_pool2d_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ tensor) {
    if (*preempted) return;
    fused_nn_max_pool2d_kernel0_device(placeholder, tensor);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_softmax_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ T_softmax_norm) {
    if (*preempted) return;
    fused_nn_softmax_kernel0_device(placeholder, T_softmax_norm);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_conv2d_nn_relu_3_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
    if (*preempted) return;
    fused_nn_conv2d_nn_relu_3_kernel0_device(placeholder, placeholder1, T_relu);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_conv2d_nn_relu_1_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
    if (*preempted) return;
    fused_nn_conv2d_nn_relu_1_kernel0_device(placeholder, placeholder1, T_relu);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_dense_add_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
    if (*preempted) return;
    fused_nn_dense_add_kernel0_device(placeholder, placeholder1, T_add, placeholder2);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_max_pool2d_2_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ tensor) {
    if (*preempted) return;
    fused_nn_max_pool2d_2_kernel0_device(placeholder, tensor);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_dense_add_nn_relu_1_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    if (*preempted) return;
    fused_nn_dense_add_nn_relu_1_kernel0_device(placeholder, placeholder1, T_relu, placeholder2);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_conv2d_nn_relu_4_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
    if (*preempted) return;
    fused_nn_conv2d_nn_relu_4_kernel0_device(placeholder, placeholder1, T_relu);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_max_pool2d_1_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ tensor) {
    if (*preempted) return;
    fused_nn_max_pool2d_1_kernel0_device(placeholder, tensor);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_conv2d_nn_relu_2_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
    if (*preempted) return;
    fused_nn_conv2d_nn_relu_2_kernel0_device(placeholder, placeholder1, T_relu);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_batch_flatten_kernel0(int* preempted, int* task_slot, float* __restrict__ tensor, float* __restrict__ placeholder) {
    if (*preempted) return;
    fused_nn_batch_flatten_kernel0_device(tensor, placeholder);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_conv2d_nn_relu_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
    if (*preempted) return;
    fused_nn_conv2d_nn_relu_kernel0_device(placeholder, placeholder1, T_relu);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void fused_nn_dense_add_nn_relu_kernel0(int* preempted, int* task_slot, float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    if (*preempted) return;
    fused_nn_dense_add_nn_relu_kernel0_device(placeholder, placeholder1, T_relu, placeholder2);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        
