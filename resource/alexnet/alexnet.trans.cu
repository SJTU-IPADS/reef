#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define CU_NUM 60

__device__ __forceinline__ bool is_first_thread() {
  return threadIdx.x == 0;
}

__device__ __forceinline__ unsigned int get_cu_id() {
  return blockIdx.x % CU_NUM;
}

__device__ __forceinline__ dim3 get_3d_idx(int idx, dim3 dim) {
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

__device__ void fused_nn_max_pool2d_kernel0_device(float* __restrict__ placeholder, float* __restrict__ tensor){
  float tensor_local[1];
  tensor_local[(0)] = -3.402823e+38f;
  for (int dh = 0; dh < 3; ++dh) {
    for (int dw = 0; dw < 3; ++dw) {
      tensor_local[(0)] = max(tensor_local[(0)], placeholder[(((((((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) / 25) * 144) + (((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 25) / 5) * 24)) + (dh * 12)) + ((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 5) * 2)) + dw))]);
    }
  }
  tensor[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))] = tensor_local[(0)];
}

__device__ void fused_nn_softmax_kernel0_device(float* __restrict__ placeholder, float* __restrict__ T_softmax_norm){
  float normal_reduce_temp0[1];
  float red_buf0[1];
  float T_softmax_exp[16];
  float normal_reduce_temp01[1];
  float red_buf01[1];
  normal_reduce_temp0[(0)] = -3.402823e+38f;
  for (int k_inner = 0; k_inner < 16; ++k_inner) {
    if (((((int)threadIdx.x) * 16) + k_inner) < 1000) {
      normal_reduce_temp0[(0)] = max(normal_reduce_temp0[(0)], placeholder[(((((int)threadIdx.x) * 16) + k_inner))]);
    }
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = normal_reduce_temp0[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  for (int i1_inner_outer = 0; i1_inner_outer < 4; ++i1_inner_outer) {
    for (int i1_inner_inner_s = 0; i1_inner_inner_s < 4; ++i1_inner_inner_s) {
      if ((((((int)threadIdx.x) * 16) + (i1_inner_outer * 4)) + i1_inner_inner_s) < 1000) {
        T_softmax_exp[(((i1_inner_outer * 4) + i1_inner_inner_s))] = __ocml_exp_f32((placeholder[((((((int)threadIdx.x) * 16) + (i1_inner_outer * 4)) + i1_inner_inner_s))] - red_buf0[(0)]));
      }
    }
  }
  normal_reduce_temp01[(0)] = 0.000000e+00f;
  for (int k_inner1 = 0; k_inner1 < 16; ++k_inner1) {
    if (((((int)threadIdx.x) * 16) + k_inner1) < 1000) {
      normal_reduce_temp01[(0)] = (normal_reduce_temp01[(0)] + __hip_ds_bpermute(((((int)threadIdx.x) + (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63))) << 2), T_softmax_exp[(k_inner1)]));
    }
  }
  unsigned int mask1[1];
  float t01[1];
  red_buf01[(0)] = normal_reduce_temp01[(0)];
  ((int*)mask1)[(0)] = 0;
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  red_buf01[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf01[(0)]);
  for (int i1_inner_outer1 = 0; i1_inner_outer1 < 4; ++i1_inner_outer1) {
    for (int i1_inner_inner_s1 = 0; i1_inner_inner_s1 < 4; ++i1_inner_inner_s1) {
      if ((((((int)threadIdx.x) * 16) + (i1_inner_outer1 * 4)) + i1_inner_inner_s1) < 1000) {
        T_softmax_norm[((((((int)threadIdx.x) * 16) + (i1_inner_outer1 * 4)) + i1_inner_inner_s1))] = (__hip_ds_bpermute(((((int)threadIdx.x) + (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63))) << 2), T_softmax_exp[(((i1_inner_outer1 * 4) + i1_inner_inner_s1))]) / red_buf01[(0)]);
      }
    }
  }
}

__device__ void fused_nn_conv2d_nn_relu_3_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu){
  float compute[2];
  __shared__ float pad_temp_shared[36];
  __shared__ float placeholder_shared[200];
  for (int ff_init = 0; ff_init < 2; ++ff_init) {
    compute[(ff_init)] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 96; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 3; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 36) {
        if ((((((int)threadIdx.y) * 5) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 9) {
          if (((((int)threadIdx.x) * 3) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 5) {
            pad_temp_shared[(((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((2 <= ((((int)blockIdx.y) * 2) + (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 6))) && (((((int)blockIdx.y) * 2) + (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 6)) < 28)) && (2 <= ((((int)blockIdx.x) * 2) + (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)))) && (((((int)blockIdx.x) * 2) + (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)) < 28)) ? placeholder[(((((((rc_outer * 676) + (((int)blockIdx.y) * 52)) + ((((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 6) * 26)) + (((int)blockIdx.x) * 2)) + (((((((int)threadIdx.z) * 9) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 6)) - 54))] : 0.000000e+00f);
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 13; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if ((((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 25)) + ((int)threadIdx.y)) < 8) {
        if ((((((int)threadIdx.z) * 10) + (((int)threadIdx.y) * 5)) + (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 5)) < 40) {
          if (((((((int)threadIdx.z) * 50) + (((int)threadIdx.y) * 25)) + (((int)threadIdx.x) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 200) {
            if ((((((int)threadIdx.y) * 25) + (((int)threadIdx.x) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 50) {
              if (((((int)threadIdx.x) * 13) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 25) {
                placeholder_shared[(((((((int)threadIdx.z) * 50) + (((int)threadIdx.y) * 25)) + (((int)threadIdx.x) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = placeholder1[(((((((((int)blockIdx.z) * 19200) + (((int)threadIdx.z) * 4800)) + (((int)threadIdx.y) * 2400)) + (rc_outer * 25)) + (((int)threadIdx.x) * 13)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))];
              }
            }
          }
        }
      }
    }
    __syncthreads();
    for (int ry_inner = 0; ry_inner < 5; ++ry_inner) {
      for (int rx_inner = 0; rx_inner < 5; ++rx_inner) {
        for (int ff = 0; ff < 2; ++ff) {
          compute[(ff)] = __ocml_fma_f32(pad_temp_shared[(((((((int)threadIdx.y) * 6) + (ry_inner * 6)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[(((((((int)threadIdx.z) * 50) + (ff * 25)) + (ry_inner * 5)) + rx_inner))], compute[(ff)]);
        }
      }
    }
  }
  for (int ax1_inner_inner_inner = 0; ax1_inner_inner_inner < 2; ++ax1_inner_inner_inner) {
    T_relu[((((((((((int)blockIdx.z) * 5408) + (((int)threadIdx.z) * 1352)) + (ax1_inner_inner_inner * 676)) + (((int)blockIdx.y) * 52)) + (((int)threadIdx.y) * 26)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)))] = max(compute[(ax1_inner_inner_inner)], 0.000000e+00f);
  }
}

__device__ void fused_nn_conv2d_nn_relu_1_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu){
  float compute[1];
  __shared__ float pad_temp_shared[192];
  __shared__ float placeholder_shared[2304];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 48; ++rc_outer) {
    __syncthreads();
    if (((((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) / 6) + ((int)threadIdx.z)) < 32) {
      if ((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 192) {
        if (((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) < 6) {
          if (((int)threadIdx.x) < 3) {
            pad_temp_shared[((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))] = (((((1 <= ((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3))) && (((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3)) < 13)) && (1 <= (((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))) && ((((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 13)) ? placeholder[(((((((((rc_outer * 1152) + ((((int)threadIdx.z) >> 2) * 144)) + (((int)blockIdx.y) * 24)) + ((((int)threadIdx.z) & 3) * 12)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) - 13))] : 0.000000e+00f);
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      placeholder_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder1[(((((((((int)blockIdx.z) * 110592) + (((int)threadIdx.z) * 3456)) + (rc_outer * 72)) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((rc_inner * 24) + (((int)threadIdx.y) * 6)) + (ry_inner * 6)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[(((((((int)threadIdx.z) * 72) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner))], compute[(0)]);
        }
      }
    }
  }
  T_relu[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 144)) + (((int)blockIdx.y) * 24)) + (((int)threadIdx.y) * 12)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)))] = max(compute[(0)], 0.000000e+00f);
}

__device__ void fused_nn_dense_add_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2){
  float T_dense_rf[1];
  float red_buf0[1];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 64; ++k_outer) {
    T_dense_rf[(0)] = __ocml_fma_f32(placeholder[(((k_outer * 64) + ((int)threadIdx.x)))], placeholder1[((((((int)blockIdx.x) * 4096) + (k_outer * 64)) + ((int)threadIdx.x)))], T_dense_rf[(0)]);
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = T_dense_rf[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = red_buf0[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_add[(((int)blockIdx.x))] = (T_dense[(0)] + placeholder2[(((int)blockIdx.x))]);
  }
}

__device__ void fused_nn_max_pool2d_2_kernel0_device(float* __restrict__ placeholder, float* __restrict__ tensor){
  float tensor_local[1];
  tensor_local[(0)] = -3.402823e+38f;
  for (int dh = 0; dh < 3; ++dh) {
    for (int dw = 0; dw < 3; ++dw) {
      tensor_local[(0)] = max(tensor_local[(0)], placeholder[(((((((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) / 676) * 2916) + (((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 676) / 26) * 108)) + (dh * 54)) + ((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 26) * 2)) + dw))]);
    }
  }
  tensor[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))] = tensor_local[(0)];
}

__device__ void fused_nn_dense_add_nn_relu_1_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float T_dense_rf[1];
  float red_buf0[1];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 100; ++k_outer) {
    T_dense_rf[(0)] = __ocml_fma_f32(placeholder[(((k_outer * 64) + ((int)threadIdx.x)))], placeholder1[((((((int)blockIdx.x) * 6400) + (k_outer * 64)) + ((int)threadIdx.x)))], T_dense_rf[(0)]);
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = T_dense_rf[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = red_buf0[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_relu[(((int)blockIdx.x))] = max((T_dense[(0)] + placeholder2[(((int)blockIdx.x))]), 0.000000e+00f);
  }
}

__device__ void fused_nn_conv2d_nn_relu_4_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu){
  float compute[1];
  __shared__ float pad_temp_shared[315];
  __shared__ float placeholder_shared[24];
  compute[(0)] = 0.000000e+00f;
  for (int ry_outer = 0; ry_outer < 11; ++ry_outer) {
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 1))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 1))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 1))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 1))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 1))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 2))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 2))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 2))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 2))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 2))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 3))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 3))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 3))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 3))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 3))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 4))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 4))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 4))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 4))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 4))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 5))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 5))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 5))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 5))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 5))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 6))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 6))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 6))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 6))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 6))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 7))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 7))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 7))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 7))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 7))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 8))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 8))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 8))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 8))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 8))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 9))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 9))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 9))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 9))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 9))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
    __syncthreads();
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 315) {
        pad_temp_shared[((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) % 5)) + 10))];
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 314) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 39) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 1) % 5)) + 10))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 313) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 38) {
          pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 2) % 5)) + 10))];
        }
      }
    }
    if (((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) < 63) {
      if ((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) < 312) {
        if (((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) < 37) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 40) + (((int)threadIdx.y) * 7)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[(((((((((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) / 21) * 50176) + (((int)blockIdx.y) * 5376)) + (ry_outer * 224)) + ((((((int)threadIdx.z) * 8) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) / 5)) % 21) * 224)) + (((int)blockIdx.x) * 8)) + ((((((int)threadIdx.y) * 7) + (((int)threadIdx.x) * 4)) + 3) % 5)) + 10))];
          }
        }
      }
    }
    if ((((((int)threadIdx.x) + ((int)threadIdx.y)) / 3) + ((int)threadIdx.z)) < 8) {
      if ((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)) < 24) {
        if ((((int)threadIdx.x) + ((int)threadIdx.y)) < 3) {
          if (((int)threadIdx.x) < 1) {
            placeholder_shared[((((((int)threadIdx.z) * 3) + ((int)threadIdx.x)) + ((int)threadIdx.y)))] = placeholder1[(((((((((int)blockIdx.z) * 2904) + (((int)threadIdx.z) * 363)) + (((int)threadIdx.x) * 121)) + (((int)threadIdx.y) * 121)) + (ry_outer * 11)) + 10))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 105))], placeholder_shared[(((((int)threadIdx.z) * 3) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 20) + (((int)threadIdx.x) * 4)) + 210))], placeholder_shared[(((((int)threadIdx.z) * 3) + 2))], compute[(0)]);
  }
  T_relu[(((((((((int)blockIdx.z) * 23328) + (((int)threadIdx.z) * 2916)) + (((int)blockIdx.y) * 324)) + (((int)threadIdx.y) * 54)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)))] = max(compute[(0)], 0.000000e+00f);
}

__device__ void fused_nn_max_pool2d_1_kernel0_device(float* __restrict__ placeholder, float* __restrict__ tensor){
  float tensor_local[1];
  tensor_local[(0)] = -3.402823e+38f;
  for (int dh = 0; dh < 3; ++dh) {
    for (int dw = 0; dw < 3; ++dw) {
      tensor_local[(0)] = max(tensor_local[(0)], placeholder[(((((((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) / 144) * 676) + (((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 144) / 12) * 52)) + (dh * 26)) + ((((((int)blockIdx.x) * 128) + ((int)threadIdx.x)) % 12) * 2)) + dw))]);
    }
  }
  tensor[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))] = tensor_local[(0)];
}

__device__ void fused_nn_conv2d_nn_relu_2_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu){
  float compute[1];
  __shared__ float pad_temp_shared[192];
  __shared__ float placeholder_shared[2304];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    if (((((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) / 6) + ((int)threadIdx.z)) < 32) {
      if ((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 192) {
        if (((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) < 6) {
          if (((int)threadIdx.x) < 3) {
            pad_temp_shared[((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))] = (((((1 <= ((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3))) && (((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3)) < 13)) && (1 <= (((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))) && ((((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 13)) ? placeholder[(((((((((rc_outer * 1152) + ((((int)threadIdx.z) >> 2) * 144)) + (((int)blockIdx.y) * 24)) + ((((int)threadIdx.z) & 3) * 12)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) - 13))] : 0.000000e+00f);
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      placeholder_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder1[(((((((((int)blockIdx.z) * 73728) + (((int)threadIdx.z) * 2304)) + (rc_outer * 72)) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((rc_inner * 24) + (((int)threadIdx.y) * 6)) + (ry_inner * 6)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[(((((((int)threadIdx.z) * 72) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner))], compute[(0)]);
        }
      }
    }
  }
  T_relu[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 144)) + (((int)blockIdx.y) * 24)) + (((int)threadIdx.y) * 12)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)))] = max(compute[(0)], 0.000000e+00f);
}

__device__ void fused_nn_batch_flatten_kernel0_device(float* __restrict__ tensor, float* __restrict__ placeholder){
  tensor[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))] = placeholder[(((((int)blockIdx.x) * 128) + ((int)threadIdx.x)))];
}

__device__ void fused_nn_conv2d_nn_relu_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu){
  float compute[1];
  __shared__ float pad_temp_shared[192];
  __shared__ float placeholder_shared[2304];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 48; ++rc_outer) {
    __syncthreads();
    if (((((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) / 6) + ((int)threadIdx.z)) < 32) {
      if ((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 192) {
        if (((((int)threadIdx.y) * 3) + ((int)threadIdx.x)) < 6) {
          if (((int)threadIdx.x) < 3) {
            pad_temp_shared[((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))] = (((((1 <= ((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3))) && (((((int)blockIdx.y) * 2) + (((int)threadIdx.z) & 3)) < 13)) && (1 <= (((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)))) && ((((((int)blockIdx.x) * 4) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) < 13)) ? placeholder[(((((((((rc_outer * 1152) + ((((int)threadIdx.z) >> 2) * 144)) + (((int)blockIdx.y) * 24)) + ((((int)threadIdx.z) & 3) * 12)) + (((int)blockIdx.x) * 4)) + (((int)threadIdx.y) * 3)) + ((int)threadIdx.x)) - 13))] : 0.000000e+00f);
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      placeholder_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = placeholder1[(((((((((int)blockIdx.z) * 110592) + (((int)threadIdx.z) * 3456)) + (rc_outer * 72)) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))];
    }
    __syncthreads();
    for (int rc_inner = 0; rc_inner < 8; ++rc_inner) {
      for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
        for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
          compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((rc_inner * 24) + (((int)threadIdx.y) * 6)) + (ry_inner * 6)) + ((int)threadIdx.x)) + rx_inner))], placeholder_shared[(((((((int)threadIdx.z) * 72) + (rc_inner * 9)) + (ry_inner * 3)) + rx_inner))], compute[(0)]);
        }
      }
    }
  }
  T_relu[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 144)) + (((int)blockIdx.y) * 24)) + (((int)threadIdx.y) * 12)) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)))] = max(compute[(0)], 0.000000e+00f);
}

__device__ void fused_nn_dense_add_nn_relu_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float T_dense_rf[1];
  float red_buf0[1];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 64; ++k_outer) {
    T_dense_rf[(0)] = __ocml_fma_f32(placeholder[(((k_outer * 64) + ((int)threadIdx.x)))], placeholder1[((((((int)blockIdx.x) * 4096) + (k_outer * 64)) + ((int)threadIdx.x)))], T_dense_rf[(0)]);
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = T_dense_rf[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = red_buf0[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_relu[(((int)blockIdx.x))] = max((T_dense[(0)] + placeholder2[(((int)blockIdx.x))]), 0.000000e+00f);
  }
}


extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_max_pool2d_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ tensor) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 128 + threadIdx.z * 1 * 128 >= 128 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 50 + blockIdx.z * 1 * 50 >= 50 * 1 * 1) return;
    fused_nn_max_pool2d_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)tensor);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(35))) __attribute__((amdgpu_num_sgpr(54))) void fused_nn_softmax_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ T_softmax_norm) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 64 + threadIdx.z * 1 * 64 >= 64 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 1 * 1 >= 1 * 1 * 1) return;
    fused_nn_softmax_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)T_softmax_norm);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(45))) __attribute__((amdgpu_num_sgpr(73))) void fused_nn_conv2d_nn_relu_3_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 2 + threadIdx.z * 2 * 2 >= 2 * 2 * 4) return;
    // if (blockIdx.x + blockIdx.y * 13 + blockIdx.z * 13 * 13 >= 13 * 13 * 32) return;
    fused_nn_conv2d_nn_relu_3_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(35))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_nn_relu_1_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 4 + threadIdx.z * 2 * 4 >= 4 * 2 * 32) return;
    // if (blockIdx.x + blockIdx.y * 3 + blockIdx.z * 6 * 3 >= 3 * 6 * 12) return;
    fused_nn_conv2d_nn_relu_1_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_dense_add_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 64 + threadIdx.z * 1 * 64 >= 64 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 1000 + blockIdx.z * 1 * 1000 >= 1000 * 1 * 1) return;
    fused_nn_dense_add_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_add,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_max_pool2d_2_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ tensor) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 128 + threadIdx.z * 1 * 128 >= 128 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 507 + blockIdx.z * 1 * 507 >= 507 * 1 * 1) return;
    fused_nn_max_pool2d_2_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)tensor);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_dense_add_nn_relu_1_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 64 + threadIdx.z * 1 * 64 >= 64 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 4096 + blockIdx.z * 1 * 4096 >= 4096 * 1 * 1) return;
    fused_nn_dense_add_nn_relu_1_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(27))) __attribute__((amdgpu_num_sgpr(40))) void fused_nn_conv2d_nn_relu_4_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 2 + threadIdx.z * 6 * 2 >= 2 * 6 * 8) return;
    // if (blockIdx.x + blockIdx.y * 27 + blockIdx.z * 9 * 27 >= 27 * 9 * 12) return;
    fused_nn_conv2d_nn_relu_4_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_max_pool2d_1_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ tensor) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 128 + threadIdx.z * 1 * 128 >= 128 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 288 + blockIdx.z * 1 * 288 >= 288 * 1 * 1) return;
    fused_nn_max_pool2d_1_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)tensor);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(35))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_nn_relu_2_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 4 + threadIdx.z * 2 * 4 >= 4 * 2 * 32) return;
    // if (blockIdx.x + blockIdx.y * 3 + blockIdx.z * 6 * 3 >= 3 * 6 * 12) return;
    fused_nn_conv2d_nn_relu_2_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_batch_flatten_kernel0_device_wrapper(float* __restrict__ tensor, float* __restrict__ placeholder) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 128 + threadIdx.z * 1 * 128 >= 128 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 50 + blockIdx.z * 1 * 50 >= 50 * 1 * 1) return;
    fused_nn_batch_flatten_kernel0_device((float* __restrict__)tensor,(float* __restrict__)placeholder);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(35))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_nn_relu_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 4 + threadIdx.z * 2 * 4 >= 4 * 2 * 32) return;
    // if (blockIdx.x + blockIdx.y * 3 + blockIdx.z * 6 * 3 >= 3 * 6 * 8) return;
    fused_nn_conv2d_nn_relu_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_dense_add_nn_relu_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 64 + threadIdx.z * 1 * 64 >= 64 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 4096 + blockIdx.z * 1 * 4096 >= 4096 * 1 * 1) return;
    fused_nn_dense_add_nn_relu_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  void fused_nn_max_pool2d_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_softmax_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_nn_relu_3_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_nn_relu_1_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_dense_add_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_max_pool2d_2_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_dense_add_nn_relu_1_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_nn_relu_4_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_max_pool2d_1_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_nn_relu_2_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_batch_flatten_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_nn_relu_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_dense_add_nn_relu_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_128_1_1(int idx) {
  dim3 dim(128, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_288_1_1(int idx) {
  dim3 dim(288, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_4096_1_1(int idx) {
  dim3 dim(4096, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_64_1_1(int idx) {
  dim3 dim(64, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_27_9_12(int idx) {
  dim3 dim(27, 9, 12);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_1000_1_1(int idx) {
  dim3 dim(1000, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_2_6_8(int idx) {
  dim3 dim(2, 6, 8);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_507_1_1(int idx) {
  dim3 dim(507, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_2_2_4(int idx) {
  dim3 dim(2, 2, 4);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_3_6_8(int idx) {
  dim3 dim(3, 6, 8);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_4_2_32(int idx) {
  dim3 dim(4, 2, 32);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_3_6_12(int idx) {
  dim3 dim(3, 6, 12);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_13_13_32(int idx) {
  dim3 dim(13, 13, 32);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_1_1_1(int idx) {
  dim3 dim(1, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_50_1_1(int idx) {
  dim3 dim(50, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

__global__ void get_3d_idx_caller(int* buf) {
    dim3 task_idx;

    task_idx = get_3d_idx_128_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_288_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_4096_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_64_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_27_9_12(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_1000_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_2_6_8(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_507_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_2_2_4(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_3_6_8(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_4_2_32(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_3_6_12(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_13_13_32(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_1_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_50_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

}

#define CALL_FRAMEWORK(idx) \
extern "C" __global__ void call_framework_##idx(\
  void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,\
  void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,\
  int cu_partition) \
{\
  asm volatile(\
    "  s_load_dwordx2 s[14:15], s[4:5], 0x0\n"\
    "  s_waitcnt lgkmcnt(0)\n"\
    "  s_setpc_b64 s[14:15]\n"\
    "  s_endpgm\n"\
  );\
}

CALL_FRAMEWORK(1)
CALL_FRAMEWORK(2)
CALL_FRAMEWORK(3)
CALL_FRAMEWORK(4)
CALL_FRAMEWORK(5)
CALL_FRAMEWORK(6)
CALL_FRAMEWORK(7)
CALL_FRAMEWORK(8)
CALL_FRAMEWORK(9)
CALL_FRAMEWORK(10)

#define MERGE_FRAMEWORK(idx) \
extern "C" __global__ void merge_framework_##idx(\
  void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,\
  void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,\
  int cu_partition) \
{\
  asm volatile(\
    "  s_load_dword s10, s[4:5], 0x40\n"\
    "  s_load_dwordx2 s[12:13], s[4:5], 0x0\n"\
    "  s_load_dwordx2 s[14:15], s[4:5], 0x20\n"\
    "  s_mul_hi_u32 s11, s6, 0x88888889\n"\
    "  s_lshr_b32 s11, s11, 5\n"\
    "  s_mul_i32 s11, s11, 60\n"\
    "  s_sub_i32 s11, s6, s11\n"\
    "  s_waitcnt lgkmcnt(0)\n"\
    "  s_cmp_ge_u32 s11, s10\n"\
    "  s_mov_b64 s[10:11], -1\n"\
    "  s_cbranch_scc1 MyBB"#idx"_3\n"\
    "; %bb.1:                                ; %Flow\n"\
    "  s_andn2_b64 vcc, exec, s[10:11]\n"\
    "  s_cbranch_vccz MyBB"#idx"_4\n"\
    "  s_endpgm\n"\
    "MyBB"#idx"_3:\n"\
    "  s_setpc_b64 s[14:15]\n"\
    "  s_endpgm\n"\
    "MyBB"#idx"_4:\n"\
    "  s_setpc_b64 s[12:13]\n"\
    "  s_endpgm\n"\
  );\
}
MERGE_FRAMEWORK(1)
MERGE_FRAMEWORK(2)
MERGE_FRAMEWORK(3)
MERGE_FRAMEWORK(4)
MERGE_FRAMEWORK(5)
MERGE_FRAMEWORK(6)
MERGE_FRAMEWORK(7)
MERGE_FRAMEWORK(8)
MERGE_FRAMEWORK(9)
MERGE_FRAMEWORK(10)
MERGE_FRAMEWORK(nostack_1)
MERGE_FRAMEWORK(nostack_2)
MERGE_FRAMEWORK(nostack_3)
MERGE_FRAMEWORK(nostack_4)
MERGE_FRAMEWORK(nostack_5)
MERGE_FRAMEWORK(nostack_6)
MERGE_FRAMEWORK(nostack_7)
MERGE_FRAMEWORK(nostack_8)
MERGE_FRAMEWORK(nostack_9)
MERGE_FRAMEWORK(nostack_10)
