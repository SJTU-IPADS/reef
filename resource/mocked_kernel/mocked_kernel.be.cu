#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define NUM_BLOCKS 64
#define NUM_TREHAD_PER_BLOCK 128
#define BLOCKDIM_X 4
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 4

__device__ void multiply_device(float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp){
    __shared__ float buffer[1024 * 32 / 4];
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    temp[arrayOffset] = a[threadOffset] * b[arrayOffset];
}

__device__ void add_device(float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp){
    __shared__ float buffer[1024 * 32 / 4];
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    temp[arrayOffset] = a[threadOffset] + b[arrayOffset];
}

extern "C" __global__ void multiply(int* preempted, int* task_slot, float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp) {
    if (*preempted) return;
    multiply_device(a, b, temp);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        

extern "C" __global__ void add(int* preempted, int* task_slot, float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp) {
    if (*preempted) return;
    add_device(a, b, temp);
    if (threadIdx.x + threadIdx.y + threadIdx.z == 0)
        atomicAdd(task_slot, 1);
}        
