#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define NUM_BLOCKS 64
#define NUM_TREHAD_PER_BLOCK 128
#define BLOCKDIM_X 4
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 4

extern "C" __global__ void multiply(float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp) {
    __shared__ float buffer[1024 * 32 / 4];
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    temp[arrayOffset] = a[threadOffset] * b[arrayOffset];
}

extern "C" __global__ void add(float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp) {
    __shared__ float buffer[1024 * 32 / 4];
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    temp[arrayOffset] = a[threadOffset] + b[arrayOffset];
}
