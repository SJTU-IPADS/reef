#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define NUM_BLOCKS 64
#define NUM_TREHAD_PER_BLOCK 128
#define BLOCKDIM_X 4
#define BLOCKDIM_Y 8
#define BLOCKDIM_Z 4

#define CU_NUM 60

__device__ __forceinline__ bool is_first_thread() {
  return threadIdx.x == 0;
}

__device__ __forceinline__ unsigned int get_cu_id() {
  return blockIdx.x % CU_NUM;
}

__device__ __forceinline__ dim3 get_3d_idx(int idx, dim3 dim) {
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

__device__ void multiply_device(float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp){
    __shared__ float buffer[1024 * 32 / 4];
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    temp[arrayOffset] = a[threadOffset] * b[arrayOffset];
}

__device__ void add_device(float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp){
    __shared__ float buffer[1024 * 32 / 4];
    int blockOffset = blockIdx.x;
    int blockSize = NUM_TREHAD_PER_BLOCK;
    int threadOffset = threadIdx.x + threadIdx.y * BLOCKDIM_X + threadIdx.z * BLOCKDIM_X * BLOCKDIM_Y;
    int arrayOffset = blockOffset * blockSize + threadOffset;

    temp[arrayOffset] = a[threadOffset] + b[arrayOffset];
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void multiply_device_wrapper(float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 4 + threadIdx.z * 8 * 4 >= 4 * 8 * 4) return;
    // if (blockIdx.x + blockIdx.y * 64 + blockIdx.z * 1 * 64 >= 64 * 1 * 1) return;
    multiply_device((float* __restrict__)a,(float* __restrict__)b,(float* __restrict__)temp);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void add_device_wrapper(float* __restrict__ a, float* __restrict__ b, float* __restrict__ temp) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 4 + threadIdx.z * 8 * 4 >= 4 * 8 * 4) return;
    // if (blockIdx.x + blockIdx.y * 64 + blockIdx.z * 1 * 64 >= 64 * 1 * 1) return;
    add_device((float* __restrict__)a,(float* __restrict__)b,(float* __restrict__)temp);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  void multiply(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void add(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_64_1_1(int idx) {
  dim3 dim(64, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_4_8_4(int idx) {
  dim3 dim(4, 8, 4);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

__global__ void get_3d_idx_caller(int* buf) {
    dim3 task_idx;

    task_idx = get_3d_idx_64_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_4_8_4(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

}

#define CALL_FRAMEWORK(idx) \
extern "C" __global__ void call_framework_##idx(\
  void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,\
  void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,\
  int cu_partition) \
{\
  asm volatile(\
    "  s_load_dwordx2 s[14:15], s[4:5], 0x0\n"\
    "  s_waitcnt lgkmcnt(0)\n"\
    "  s_setpc_b64 s[14:15]\n"\
    "  s_endpgm\n"\
  );\
}

CALL_FRAMEWORK(1)
CALL_FRAMEWORK(2)
CALL_FRAMEWORK(3)
CALL_FRAMEWORK(4)
CALL_FRAMEWORK(5)
CALL_FRAMEWORK(6)
CALL_FRAMEWORK(7)
CALL_FRAMEWORK(8)
CALL_FRAMEWORK(9)
CALL_FRAMEWORK(10)

#define MERGE_FRAMEWORK(idx) \
extern "C" __global__ void merge_framework_##idx(\
  void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,\
  void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,\
  int cu_partition) \
{\
  asm volatile(\
    "  s_load_dword s10, s[4:5], 0x40\n"\
    "  s_load_dwordx2 s[12:13], s[4:5], 0x0\n"\
    "  s_load_dwordx2 s[14:15], s[4:5], 0x20\n"\
    "  s_mul_hi_u32 s11, s6, 0x88888889\n"\
    "  s_lshr_b32 s11, s11, 5\n"\
    "  s_mul_i32 s11, s11, 60\n"\
    "  s_sub_i32 s11, s6, s11\n"\
    "  s_waitcnt lgkmcnt(0)\n"\
    "  s_cmp_ge_u32 s11, s10\n"\
    "  s_mov_b64 s[10:11], -1\n"\
    "  s_cbranch_scc1 MyBB"#idx"_3\n"\
    "; %bb.1:                                ; %Flow\n"\
    "  s_andn2_b64 vcc, exec, s[10:11]\n"\
    "  s_cbranch_vccz MyBB"#idx"_4\n"\
    "  s_endpgm\n"\
    "MyBB"#idx"_3:\n"\
    "  s_setpc_b64 s[14:15]\n"\
    "  s_endpgm\n"\
    "MyBB"#idx"_4:\n"\
    "  s_setpc_b64 s[12:13]\n"\
    "  s_endpgm\n"\
  );\
}
MERGE_FRAMEWORK(1)
MERGE_FRAMEWORK(2)
MERGE_FRAMEWORK(3)
MERGE_FRAMEWORK(4)
MERGE_FRAMEWORK(5)
MERGE_FRAMEWORK(6)
MERGE_FRAMEWORK(7)
MERGE_FRAMEWORK(8)
MERGE_FRAMEWORK(9)
MERGE_FRAMEWORK(10)
MERGE_FRAMEWORK(nostack_1)
MERGE_FRAMEWORK(nostack_2)
MERGE_FRAMEWORK(nostack_3)
MERGE_FRAMEWORK(nostack_4)
MERGE_FRAMEWORK(nostack_5)
MERGE_FRAMEWORK(nostack_6)
MERGE_FRAMEWORK(nostack_7)
MERGE_FRAMEWORK(nostack_8)
MERGE_FRAMEWORK(nostack_9)
MERGE_FRAMEWORK(nostack_10)
