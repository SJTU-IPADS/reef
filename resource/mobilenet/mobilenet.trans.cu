#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#define CU_NUM 60

__device__ __forceinline__ bool is_first_thread() {
  return threadIdx.x == 0;
}

__device__ __forceinline__ unsigned int get_cu_id() {
  return blockIdx.x % CU_NUM;
}

__device__ __forceinline__ dim3 get_3d_idx(int idx, dim3 dim) {
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

__device__ void fused_nn_conv2d_add_nn_relu_12_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float compute[8];
  __shared__ float pad_temp_shared[512];
  __shared__ float placeholder_shared[1024];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder[(((((((rc_outer * 50176) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[((((((((rc_outer * 50176) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))];
    placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + (((int)threadIdx.y) * 128)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder1[(((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + (((int)threadIdx.y) * 128)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 1))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder1[(((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + (((int)threadIdx.y) * 128)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 2))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder1[(((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + (((int)threadIdx.y) * 128)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 3))];
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(7)]);
  }
  T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 50176))] = max((compute[(2)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100352))] = max((compute[(4)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 150528))] = max((compute[(6)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 50177))] = max((compute[(3)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100353))] = max((compute[(5)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 150529))] = max((compute[(7)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_7_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  __shared__ float PaddedInput_shared[841];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 841) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 14) + ((int)threadIdx.y)) < 61) {
        PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] = (((29 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x))) && (1 <= ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 29))) ? placeholder[(((((((int)blockIdx.z) * 784) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 29) * 28)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 29)) - 29))] : 0.000000e+00f);
      }
    }
  }
  if (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 14) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 14) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  for (int ax2 = 0; ax2 < 3; ++ax2) {
    for (int ax3 = 0; ax3 < 3; ++ax3) {
      PaddedInput_shared_local[(((ax2 * 3) + ax3))] = PaddedInput_shared[(((((((int)threadIdx.y) * 58) + (ax2 * 29)) + (((int)threadIdx.x) * 2)) + ax3))];
    }
  }
  for (int ax21 = 0; ax21 < 3; ++ax21) {
    for (int ax31 = 0; ax31 < 3; ++ax31) {
      placeholder_shared_local[(((ax21 * 3) + ax31))] = placeholder_shared[(((ax21 * 3) + ax31))];
    }
  }
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  for (int di = 0; di < 3; ++di) {
    for (int dj = 0; dj < 3; ++dj) {
      DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(((di * 3) + dj))], placeholder_shared_local[(((di * 3) + dj))], DepthwiseConv2d[(0)]);
    }
  }
  T_relu[((((((int)blockIdx.z) * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_6_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float compute[2];
  __shared__ float pad_temp_shared[448];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 14) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = placeholder[(((((((rc_outer * 3136) + ((((int)threadIdx.z) >> 1) * 196)) + (((int)blockIdx.y) * 28)) + ((((int)threadIdx.z) & 1) * 14)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
    if (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) >> 4) + ((int)threadIdx.z)) < 32) {
      if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) < 512) {
        if (((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) < 16) {
          if (((int)threadIdx.x) < 4) {
            placeholder_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 256)) + (rc_outer * 16)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))];
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1) >> 4) + ((int)threadIdx.z)) < 32) {
      if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) < 511) {
        if (((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) < 15) {
          if (((int)threadIdx.x) < 4) {
            placeholder_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder1[(((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 256)) + (rc_outer * 16)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 28))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 29))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 56))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 57))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 84))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 85))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 140))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 141))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 168))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 169))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 197))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 252))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 253))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 280))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 281))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 308))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 309))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 364))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 365))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 393))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 420))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 421))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
  }
  T_relu[((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_14_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float compute[8];
  __shared__ float pad_temp_shared[512];
  __shared__ float placeholder_shared[1024];
  compute[(0)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder[(((((((rc_outer * 50176) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[((((((((rc_outer * 50176) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))];
    placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)))] = placeholder1[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 64)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)))];
    placeholder_shared[(((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder1[(((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 64)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 1))];
    placeholder_shared[(((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder1[(((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 64)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 2))];
    placeholder_shared[(((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder1[(((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 64)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 3))];
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 16))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 16))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 17))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 17))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 48))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 48))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 80))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 80))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 81))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 81))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 144))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 144))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 145))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 145))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 176))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 176))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 177))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 177))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 208))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 208))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 209))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 209))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 240))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 240))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 241))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 241))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 272))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 272))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 273))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 273))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 304))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 304))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 305))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 305))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 368))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 368))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 369))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 369))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 400))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 400))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 401))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 401))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 432))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 432))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 433))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 433))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 464))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 464))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 465))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 465))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 496))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 496))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 497))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 497))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(7)]);
  }
  T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100352))] = max((compute[(4)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 112))] = max((compute[(2)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100464))] = max((compute[(6)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100353))] = max((compute[(5)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 113))] = max((compute[(3)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100465))] = max((compute[(7)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_18_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float compute[7];
  __shared__ float pad_temp_shared[495];
  __shared__ float placeholder_shared[288];
  for (int yy_init = 0; yy_init < 7; ++yy_init) {
    compute[(yy_init)] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 3; ++rc_outer) {
    __syncthreads();
    if (((((int)threadIdx.z) * 16) + ((int)threadIdx.x)) < 495) {
      pad_temp_shared[(((((int)threadIdx.z) * 16) + ((int)threadIdx.x)))] = (((1 <= ((((int)blockIdx.y) * 14) + (((((int)threadIdx.z) * 16) + ((int)threadIdx.x)) / 33))) && (1 <= ((((int)blockIdx.x) * 32) + (((((int)threadIdx.z) * 16) + ((int)threadIdx.x)) % 33)))) ? placeholder[(((((((rc_outer * 50176) + (((int)blockIdx.y) * 3136)) + ((((((int)threadIdx.z) * 16) + ((int)threadIdx.x)) / 33) * 224)) + (((int)blockIdx.x) * 32)) + (((((int)threadIdx.z) * 16) + ((int)threadIdx.x)) % 33)) - 225))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.x) / 9) + ((int)threadIdx.z)) < 32) {
      if (((((int)threadIdx.z) * 3) + (((int)threadIdx.x) / 3)) < 96) {
        if (((((int)threadIdx.z) * 9) + ((int)threadIdx.x)) < 288) {
          if (((int)threadIdx.x) < 9) {
            placeholder_shared[(((((int)threadIdx.z) * 9) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 27) + (rc_outer * 9)) + ((int)threadIdx.x)))];
          }
        }
      }
    }
    __syncthreads();
    for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
      for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
        for (int yy = 0; yy < 7; ++yy) {
          compute[(yy)] = __ocml_fma_f32(pad_temp_shared[(((((yy * 66) + (ry_inner * 33)) + (((int)threadIdx.x) * 2)) + rx_inner))], placeholder_shared[((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner))], compute[(yy)]);
        }
      }
    }
  }
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 7; ++ax2_inner_inner_inner) {
    T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 784)) + (ax2_inner_inner_inner * 112)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)))] = max((compute[(ax2_inner_inner_inner)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  }
}

__device__ void fused_nn_conv2d_add_nn_relu_17_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  __shared__ float PaddedInput_shared[1824];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[18];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[4];
  PaddedInput_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = ((((1 <= ((((int)blockIdx.y) * 14) + (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) / 114))) && (1 <= (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 114))) && ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) / 114) * 112)) + (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 392))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 50) % 114)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 50) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 392) / 114) * 112)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 50) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 100) % 114)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 100) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784) / 114) * 112)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 100) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1176))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 36) % 114)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 36) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1176) / 114) * 112)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 36) % 114)) - 113))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 256) {
    if (((int)threadIdx.y) < 10) {
      PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568))] = ((((((((int)blockIdx.y) * 14) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568) / 114)) < 113) && (1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 86) % 114))) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 86) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568) / 114) * 112)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 86) % 114)) - 113))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 28) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 3))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 4))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 5))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 114))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 115))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 116))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 117))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 118))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 119))];
  PaddedInput_shared_local[(12)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 228))];
  PaddedInput_shared_local[(13)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 229))];
  PaddedInput_shared_local[(14)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 230))];
  PaddedInput_shared_local[(15)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 231))];
  PaddedInput_shared_local[(16)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 232))];
  PaddedInput_shared_local[(17)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 233))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(12)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(13)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(14)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(0)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(1)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(2)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(3)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(4)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(5)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(13)], placeholder_shared_local[(6)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(14)], placeholder_shared_local[(7)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(15)], placeholder_shared_local[(8)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(0)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(1)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(2)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(3)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(4)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(5)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(14)], placeholder_shared_local[(6)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(15)], placeholder_shared_local[(7)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(16)], placeholder_shared_local[(8)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(0)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(1)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(2)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(3)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(4)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(11)], placeholder_shared_local[(5)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(15)], placeholder_shared_local[(6)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(16)], placeholder_shared_local[(7)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(17)], placeholder_shared_local[(8)], DepthwiseConv2d[(3)]);
  T_relu[(((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 4)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 4)) + 1))] = max((DepthwiseConv2d[(1)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 4)) + 2))] = max((DepthwiseConv2d[(2)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 4)) + 3))] = max((DepthwiseConv2d[(3)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_11_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  __shared__ float PaddedInput_shared[3249];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  PaddedInput_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = (((57 <= ((((int)threadIdx.y) * 28) + ((int)threadIdx.x))) && (1 <= (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 57))) ? placeholder[(((((((int)blockIdx.z) * 3136) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) / 57) * 56)) + (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 43) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 43) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 29) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 29) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2352))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 15) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2352) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 15) % 57)) - 57))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 113) {
    if (((int)threadIdx.y) < 5) {
      PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 3136))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 3136) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1) % 57)) - 57))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 28) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 57))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 58))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 59))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 114))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 115))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 116))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  T_relu[((((((int)blockIdx.z) * 784) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_16_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float compute[16];
  __shared__ float pad_temp_shared[1024];
  __shared__ float placeholder_shared[1024];
  compute[(0)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(8)] = 0.000000e+00f;
  compute[(12)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(9)] = 0.000000e+00f;
  compute[(13)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(10)] = 0.000000e+00f;
  compute[(14)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  compute[(11)] = 0.000000e+00f;
  compute[(15)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((rc_outer * 200704) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[((((((((rc_outer * 200704) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 1))];
    pad_temp_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[((((((((rc_outer * 200704) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 2))];
    pad_temp_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[((((((((rc_outer * 200704) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 3))];
    placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)))] = placeholder1[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder1[((((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 1))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder1[((((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 2))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder1[((((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 3))];
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 2))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 2))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 2))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 2))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 3))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 3))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 3))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 3))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 66))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 66))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 66))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 66))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 67))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 67))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 67))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 67))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 130))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 130))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 130))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 130))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 131))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 131))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 131))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 131))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 194))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 194))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 194))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 194))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 195))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 195))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 195))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 195))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 258))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 258))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 258))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 258))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 259))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 259))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 259))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 259))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 322))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 322))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 322))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 322))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 323))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 323))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 323))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 323))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 386))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 386))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 386))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 386))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 387))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 387))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 387))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 387))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 450))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 450))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 450))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 450))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 451))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 451))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 451))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 451))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 512))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 512))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 512))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 512))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 513))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 513))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 513))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 513))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 514))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 514))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 514))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 514))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 515))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 515))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 515))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 515))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 576))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 576))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 576))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 576))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 577))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 577))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 577))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 577))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 578))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 578))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 578))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 578))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 579))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 579))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 579))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 579))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 640))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 640))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 640))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 640))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 641))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 641))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 641))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 641))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 642))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 642))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 642))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 642))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 643))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 643))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 643))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 643))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 704))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 704))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 704))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 704))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 705))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 705))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 705))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 705))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 706))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 706))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 706))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 706))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 707))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 707))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 707))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 707))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 768))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 768))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 768))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 768))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 769))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 769))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 769))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 769))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 770))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 770))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 770))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 770))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 771))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 771))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 771))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 771))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 832))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 832))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 832))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 832))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 834))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 834))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 834))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 834))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 835))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 835))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 835))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 835))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 898))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 898))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 898))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 898))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 899))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 899))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 899))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 899))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 960))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 960))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 960))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 960))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 961))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 961))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 961))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 961))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 962))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 962))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 962))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 962))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 963))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 963))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 963))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 963))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(15)]);
  }
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)))] = max((compute[(0)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 200704))] = max((compute[(4)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 401408))] = max((compute[(8)] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 602112))] = max((compute[(12)] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 1))] = max((compute[(1)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 200705))] = max((compute[(5)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 401409))] = max((compute[(9)] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 602113))] = max((compute[(13)] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 2))] = max((compute[(2)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 200706))] = max((compute[(6)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 401410))] = max((compute[(10)] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 602114))] = max((compute[(14)] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 3))] = max((compute[(3)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 200707))] = max((compute[(7)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 401411))] = max((compute[(11)] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 602115))] = max((compute[(15)] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_2_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float compute[1];
  __shared__ float pad_temp_shared[1568];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))] = placeholder[(((((rc_outer * 1568) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[(((((rc_outer * 1568) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 14)) + ((((int)threadIdx.x) * 2) + 1)))];
    if (((((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 5) + ((int)threadIdx.z)) < 16) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)) < 512) {
        if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 5) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + (rc_outer * 32)) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[((((int)threadIdx.z) * 32))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 32) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))], placeholder_shared[(((((int)threadIdx.z) * 32) + 2))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))], placeholder_shared[(((((int)threadIdx.z) * 32) + 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 32) + 4))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 245))], placeholder_shared[(((((int)threadIdx.z) * 32) + 5))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 294))], placeholder_shared[(((((int)threadIdx.z) * 32) + 6))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 343))], placeholder_shared[(((((int)threadIdx.z) * 32) + 7))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 32) + 8))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))], placeholder_shared[(((((int)threadIdx.z) * 32) + 9))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 490))], placeholder_shared[(((((int)threadIdx.z) * 32) + 10))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 539))], placeholder_shared[(((((int)threadIdx.z) * 32) + 11))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 588))], placeholder_shared[(((((int)threadIdx.z) * 32) + 12))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))], placeholder_shared[(((((int)threadIdx.z) * 32) + 13))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 686))], placeholder_shared[(((((int)threadIdx.z) * 32) + 14))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 735))], placeholder_shared[(((((int)threadIdx.z) * 32) + 15))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 32) + 16))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 32) + 17))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 882))], placeholder_shared[(((((int)threadIdx.z) * 32) + 18))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 931))], placeholder_shared[(((((int)threadIdx.z) * 32) + 19))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 980))], placeholder_shared[(((((int)threadIdx.z) * 32) + 20))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1029))], placeholder_shared[(((((int)threadIdx.z) * 32) + 21))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1078))], placeholder_shared[(((((int)threadIdx.z) * 32) + 22))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1127))], placeholder_shared[(((((int)threadIdx.z) * 32) + 23))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1176))], placeholder_shared[(((((int)threadIdx.z) * 32) + 24))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1225))], placeholder_shared[(((((int)threadIdx.z) * 32) + 25))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1274))], placeholder_shared[(((((int)threadIdx.z) * 32) + 26))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1323))], placeholder_shared[(((((int)threadIdx.z) * 32) + 27))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1372))], placeholder_shared[(((((int)threadIdx.z) * 32) + 28))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1421))], placeholder_shared[(((((int)threadIdx.z) * 32) + 29))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1470))], placeholder_shared[(((((int)threadIdx.z) * 32) + 30))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1519))], placeholder_shared[(((((int)threadIdx.z) * 32) + 31))], compute[(0)]);
  }
  T_relu[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

__device__ void fused_nn_batch_flatten_kernel0_device(float* __restrict__ tensor, float* __restrict__ placeholder){
  tensor[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = placeholder[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))];
}

__device__ void fused_nn_conv2d_add_nn_relu_13_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  __shared__ float PaddedInput_shared[3364];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[16];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[4];
  PaddedInput_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = ((((58 <= ((((int)threadIdx.y) * 28) + ((int)threadIdx.x))) && (1 <= (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 58))) && ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 58) < 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) / 58) * 56)) + (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 30) % 58)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 30) % 58) < 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784) / 58) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 30) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2) % 58)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2) % 58) < 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568) / 58) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2352))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 32) % 58)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 32) % 58) < 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2352) / 58) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 32) % 58)) - 57))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 228) {
    if (((int)threadIdx.y) < 9) {
      PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 3136))] = ((((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 170) && (1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 4) % 58))) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 4) % 58) < 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 3136) / 58) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 4) % 58)) - 57))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 28) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 3))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 58))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 59))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 60))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 61))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 116))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 117))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 118))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 119))];
  PaddedInput_shared_local[(12)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 174))];
  PaddedInput_shared_local[(13)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 175))];
  PaddedInput_shared_local[(14)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 176))];
  PaddedInput_shared_local[(15)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 177))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(0)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(1)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(2)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(3)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(4)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(5)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(6)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(7)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(11)], placeholder_shared_local[(8)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(0)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(1)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(2)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(3)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(4)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(5)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(12)], placeholder_shared_local[(6)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(13)], placeholder_shared_local[(7)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(14)], placeholder_shared_local[(8)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(0)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(1)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(2)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(3)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(4)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(11)], placeholder_shared_local[(5)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(13)], placeholder_shared_local[(6)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(14)], placeholder_shared_local[(7)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(15)], placeholder_shared_local[(8)], DepthwiseConv2d[(3)]);
  T_relu[((((((int)blockIdx.z) * 3136) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 2)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 2)) + 1))] = max((DepthwiseConv2d[(1)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 2)) + 56))] = max((DepthwiseConv2d[(2)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 2)) + 57))] = max((DepthwiseConv2d[(3)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_8_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float compute[4];
  __shared__ float pad_temp_shared[1792];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 112) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = placeholder[((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 112) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[(((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 4)) < 32) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) < 512) {
        if (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 8) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + ((((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 4) * 256)) + (rc_outer * 16)) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 560))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 560))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 561))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 561))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 672))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 672))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 673))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 673))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 785))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 785))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1008))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1008))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1009))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1009))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1120))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1120))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1121))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1121))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1232))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1232))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1233))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1233))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1344))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1344))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1345))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1345))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1456))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1456))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1457))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1457))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1568))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1568))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1569))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1569))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1680))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1680))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1681))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1681))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(3)]);
  }
  T_relu[((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12544))] = max((compute[(2)] + placeholder2[((((((int)blockIdx.z) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12545))] = max((compute[(3)] + placeholder2[((((((int)blockIdx.z) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
}

__device__ void fused_nn_global_avg_pool2d_kernel0_device(float* __restrict__ placeholder, float* __restrict__ tensor){
  float tensor1[1];
  tensor1[(0)] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 7; ++rv0) {
    for (int rv1 = 0; rv1 < 7; ++rv1) {
      if (((int)threadIdx.y) < 1) {
        tensor1[(0)] = (tensor1[(0)] + placeholder[((((((((int)threadIdx.y) * 50176) + (((int)blockIdx.x) * 392)) + (((int)threadIdx.x) * 49)) + (rv0 * 7)) + rv1))]);
      }
    }
  }
  if (((int)threadIdx.y) < 1) {
    tensor[((((((int)threadIdx.y) * 1024) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)))] = (tensor1[(0)] * 2.040816e-02f);
  }
}

__device__ void fused_nn_conv2d_add_nn_relu_9_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  __shared__ float PaddedInput_shared[1024];
  __shared__ float placeholder_shared[36];
  float PaddedInput_shared_local[12];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[2];
  PaddedInput_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)) < 29)) ? placeholder[((((((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)) - 29))] : 0.000000e+00f);
  PaddedInput_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392))] = (((((1 <= ((((int)blockIdx.y) * 14) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 136) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 136) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15)))) && (((((int)blockIdx.x) * 14) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15)) < 29)) ? placeholder[((((((((((int)blockIdx.z) * 3136) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + (((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 136) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15)) - 29))] : 0.000000e+00f);
  if ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 240) {
    if (((((int)threadIdx.z) * 14) + ((int)threadIdx.y)) < 35) {
      if (((int)threadIdx.z) < 3) {
        PaddedInput_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 784))] = ((((((((int)blockIdx.y) * 14) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 16) >> 4)) < 29) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)) < 29)) ? placeholder[((((((((((int)blockIdx.z) * 3136) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 784) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 16) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)) - 29))] : 0.000000e+00f);
      }
    }
  }
  if ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 36) {
    if (((((int)threadIdx.z) * 14) + ((int)threadIdx.y)) < 6) {
      if (((int)threadIdx.z) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = placeholder1[(((((((int)threadIdx.z) * 98) + (((int)blockIdx.z) * 36)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
      }
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 16))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 17))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 18))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 19))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 32))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 33))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 34))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 35))];
  placeholder_shared_local[(0)] = placeholder_shared[((((int)threadIdx.z) * 9))];
  placeholder_shared_local[(1)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 1))];
  placeholder_shared_local[(2)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 2))];
  placeholder_shared_local[(3)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 3))];
  placeholder_shared_local[(4)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 4))];
  placeholder_shared_local[(5)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 5))];
  placeholder_shared_local[(6)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 6))];
  placeholder_shared_local[(7)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 7))];
  placeholder_shared_local[(8)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 8))];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(0)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(1)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(2)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(3)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(4)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(5)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(6)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(7)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(11)], placeholder_shared_local[(8)], DepthwiseConv2d[(1)]);
  T_relu[(((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((((int)blockIdx.z) * 4) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = max((DepthwiseConv2d[(1)] + placeholder2[(((((int)blockIdx.z) * 4) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

__device__ void fused_nn_dense_add_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2){
  float T_dense_rf[1];
  float red_buf0[1];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 16; ++k_outer) {
    T_dense_rf[(0)] = __ocml_fma_f32(placeholder[(((k_outer * 64) + ((int)threadIdx.x)))], placeholder1[((((((int)blockIdx.x) * 1024) + (k_outer * 64)) + ((int)threadIdx.x)))], T_dense_rf[(0)]);
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = T_dense_rf[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = red_buf0[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_add[(((int)blockIdx.x))] = (T_dense[(0)] + placeholder2[(((int)blockIdx.x))]);
  }
}

__device__ void fused_nn_conv2d_add_nn_relu_15_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  __shared__ float PaddedInput_shared[3277];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) < 3277) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 14) + ((int)threadIdx.y)) < 59) {
        PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)))] = (((1 <= ((((int)blockIdx.y) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) / 113))) && (1 <= ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) % 113))) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 3136)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) / 113) * 112)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) % 113)) - 113))] : 0.000000e+00f);
      }
    }
  }
  if (((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 56) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 56) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  for (int ax2 = 0; ax2 < 3; ++ax2) {
    for (int ax3 = 0; ax3 < 3; ++ax3) {
      PaddedInput_shared_local[(((ax2 * 3) + ax3))] = PaddedInput_shared[(((((((int)threadIdx.y) * 226) + (ax2 * 113)) + (((int)threadIdx.x) * 2)) + ax3))];
    }
  }
  for (int ax21 = 0; ax21 < 3; ++ax21) {
    for (int ax31 = 0; ax31 < 3; ++ax31) {
      placeholder_shared_local[(((ax21 * 3) + ax31))] = placeholder_shared[(((ax21 * 3) + ax31))];
    }
  }
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  for (int di = 0; di < 3; ++di) {
    for (int dj = 0; dj < 3; ++dj) {
      DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(((di * 3) + dj))], placeholder_shared_local[(((di * 3) + dj))], DepthwiseConv2d[(0)]);
    }
  }
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 784)) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

__device__ void fused_nn_softmax_kernel0_device(float* __restrict__ placeholder, float* __restrict__ T_softmax_norm){
  float normal_reduce_temp0[1];
  float red_buf0[1];
  float T_softmax_exp[16];
  float normal_reduce_temp01[1];
  float red_buf01[1];
  normal_reduce_temp0[(0)] = -3.402823e+38f;
  for (int k_inner = 0; k_inner < 16; ++k_inner) {
    if (((((int)threadIdx.x) * 16) + k_inner) < 1000) {
      normal_reduce_temp0[(0)] = max(normal_reduce_temp0[(0)], placeholder[(((((int)threadIdx.x) * 16) + k_inner))]);
    }
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = normal_reduce_temp0[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  for (int i1_inner_outer = 0; i1_inner_outer < 4; ++i1_inner_outer) {
    for (int i1_inner_inner_s = 0; i1_inner_inner_s < 4; ++i1_inner_inner_s) {
      if ((((((int)threadIdx.x) * 16) + (i1_inner_outer * 4)) + i1_inner_inner_s) < 1000) {
        T_softmax_exp[(((i1_inner_outer * 4) + i1_inner_inner_s))] = __ocml_exp_f32((placeholder[((((((int)threadIdx.x) * 16) + (i1_inner_outer * 4)) + i1_inner_inner_s))] - red_buf0[(0)]));
      }
    }
  }
  normal_reduce_temp01[(0)] = 0.000000e+00f;
  for (int k_inner1 = 0; k_inner1 < 16; ++k_inner1) {
    if (((((int)threadIdx.x) * 16) + k_inner1) < 1000) {
      normal_reduce_temp01[(0)] = (normal_reduce_temp01[(0)] + __hip_ds_bpermute(((((int)threadIdx.x) + (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63))) << 2), T_softmax_exp[(k_inner1)]));
    }
  }
  unsigned int mask1[1];
  float t01[1];
  red_buf01[(0)] = normal_reduce_temp01[(0)];
  ((int*)mask1)[(0)] = 0;
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  red_buf01[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf01[(0)]);
  for (int i1_inner_outer1 = 0; i1_inner_outer1 < 4; ++i1_inner_outer1) {
    for (int i1_inner_inner_s1 = 0; i1_inner_inner_s1 < 4; ++i1_inner_inner_s1) {
      if ((((((int)threadIdx.x) * 16) + (i1_inner_outer1 * 4)) + i1_inner_inner_s1) < 1000) {
        T_softmax_norm[((((((int)threadIdx.x) * 16) + (i1_inner_outer1 * 4)) + i1_inner_inner_s1))] = (__hip_ds_bpermute(((((int)threadIdx.x) + (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63))) << 2), T_softmax_exp[(((i1_inner_outer1 * 4) + i1_inner_inner_s1))]) / red_buf01[(0)]);
      }
    }
  }
}

__device__ void fused_nn_conv2d_add_nn_relu_3_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  __shared__ float PaddedInput_shared[1800];
  __shared__ float placeholder_shared[72];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  PaddedInput_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = (((15 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 225)) && (1 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 15))) ? placeholder[((((((((int)blockIdx.z) * 1568) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 225) * 196)) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 225) / 15) * 14)) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[(((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392))] = (((15 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 167) % 225)) && (1 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 2) % 15))) ? placeholder[((((((((int)blockIdx.z) * 1568) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392) / 225) * 196)) + (((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 167) % 225) / 15) * 14)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 2) % 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[(((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 784))] = (((15 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 109) % 225)) && (1 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 4) % 15))) ? placeholder[((((((((int)blockIdx.z) * 1568) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 784) / 225) * 196)) + (((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 109) % 225) / 15) * 14)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 4) % 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[(((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 1176))] = (((15 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 51) % 225)) && (1 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 6) % 15))) ? placeholder[((((((((int)blockIdx.z) * 1568) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 1176) / 225) * 196)) + (((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 51) % 225) / 15) * 14)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 6) % 15)) - 15))] : 0.000000e+00f);
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 232) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 34) {
      if (((int)threadIdx.z) < 5) {
        PaddedInput_shared[(((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 1568))] = (((15 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 218) % 225)) && (1 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) % 15))) ? placeholder[((((((((int)blockIdx.z) * 1568) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 1568) / 225) * 196)) + (((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 218) % 225) / 15) * 14)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) % 15)) - 15))] : 0.000000e+00f);
      }
    }
  }
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 72) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 11) {
      if (((int)threadIdx.z) < 2) {
        placeholder_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = placeholder1[(((((((int)blockIdx.z) * 72) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
      }
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 15))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 16))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 17))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 30))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 31))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 32))];
  placeholder_shared_local[(0)] = placeholder_shared[((((int)threadIdx.z) * 9))];
  placeholder_shared_local[(1)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 1))];
  placeholder_shared_local[(2)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 2))];
  placeholder_shared_local[(3)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 3))];
  placeholder_shared_local[(4)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 4))];
  placeholder_shared_local[(5)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 5))];
  placeholder_shared_local[(6)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 6))];
  placeholder_shared_local[(7)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 7))];
  placeholder_shared_local[(8)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 8))];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  T_relu[(((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((((int)blockIdx.z) * 8) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_4_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float compute[2];
  __shared__ float pad_temp_shared[784];
  __shared__ float placeholder_shared[256];
  compute[(0)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))] = placeholder[((((((((rc_outer * 3136) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.y) * 2) + ((((int)threadIdx.x) * 2) / 7)) / 7) * 196)) + (((int)blockIdx.y) * 98)) + ((((((int)threadIdx.y) * 2) + ((((int)threadIdx.x) * 2) / 7)) % 7) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 2) % 7)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[((((((((rc_outer * 3136) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) / 7)) / 7) * 196)) + (((int)blockIdx.y) * 98)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) / 7)) % 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 2) + 1) % 7)))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 4)) < 16) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)) < 256) {
        if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 5) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 1024)) + ((((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 4) * 512)) + (rc_outer * 16)) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 128))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 129))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))], placeholder_shared[(((((int)threadIdx.z) * 16) + 130))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))], placeholder_shared[(((((int)threadIdx.z) * 16) + 131))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 16) + 132))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 245))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 245))], placeholder_shared[(((((int)threadIdx.z) * 16) + 133))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 294))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 294))], placeholder_shared[(((((int)threadIdx.z) * 16) + 134))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 343))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 343))], placeholder_shared[(((((int)threadIdx.z) * 16) + 135))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 16) + 136))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))], placeholder_shared[(((((int)threadIdx.z) * 16) + 137))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 490))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 490))], placeholder_shared[(((((int)threadIdx.z) * 16) + 138))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 539))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 539))], placeholder_shared[(((((int)threadIdx.z) * 16) + 139))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 588))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 588))], placeholder_shared[(((((int)threadIdx.z) * 16) + 140))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))], placeholder_shared[(((((int)threadIdx.z) * 16) + 141))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 686))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 686))], placeholder_shared[(((((int)threadIdx.z) * 16) + 142))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 735))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 735))], placeholder_shared[(((((int)threadIdx.z) * 16) + 143))], compute[(1)]);
  }
  T_relu[(((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 1568))] = max((compute[(1)] + placeholder2[((((((int)blockIdx.z) * 16) + ((int)threadIdx.z)) + 8))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float compute[1];
  __shared__ float pad_temp_shared[1568];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))] = placeholder[(((((rc_outer * 1568) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[(((((rc_outer * 1568) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 14)) + ((((int)threadIdx.x) * 2) + 1)))];
    if (((((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 5) + ((int)threadIdx.z)) < 16) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)) < 512) {
        if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 5) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) + (rc_outer * 32)) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[((((int)threadIdx.z) * 32))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 32) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))], placeholder_shared[(((((int)threadIdx.z) * 32) + 2))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))], placeholder_shared[(((((int)threadIdx.z) * 32) + 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 32) + 4))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 245))], placeholder_shared[(((((int)threadIdx.z) * 32) + 5))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 294))], placeholder_shared[(((((int)threadIdx.z) * 32) + 6))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 343))], placeholder_shared[(((((int)threadIdx.z) * 32) + 7))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 32) + 8))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))], placeholder_shared[(((((int)threadIdx.z) * 32) + 9))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 490))], placeholder_shared[(((((int)threadIdx.z) * 32) + 10))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 539))], placeholder_shared[(((((int)threadIdx.z) * 32) + 11))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 588))], placeholder_shared[(((((int)threadIdx.z) * 32) + 12))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))], placeholder_shared[(((((int)threadIdx.z) * 32) + 13))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 686))], placeholder_shared[(((((int)threadIdx.z) * 32) + 14))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 735))], placeholder_shared[(((((int)threadIdx.z) * 32) + 15))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 32) + 16))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 32) + 17))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 882))], placeholder_shared[(((((int)threadIdx.z) * 32) + 18))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 931))], placeholder_shared[(((((int)threadIdx.z) * 32) + 19))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 980))], placeholder_shared[(((((int)threadIdx.z) * 32) + 20))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1029))], placeholder_shared[(((((int)threadIdx.z) * 32) + 21))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1078))], placeholder_shared[(((((int)threadIdx.z) * 32) + 22))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1127))], placeholder_shared[(((((int)threadIdx.z) * 32) + 23))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1176))], placeholder_shared[(((((int)threadIdx.z) * 32) + 24))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1225))], placeholder_shared[(((((int)threadIdx.z) * 32) + 25))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1274))], placeholder_shared[(((((int)threadIdx.z) * 32) + 26))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1323))], placeholder_shared[(((((int)threadIdx.z) * 32) + 27))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1372))], placeholder_shared[(((((int)threadIdx.z) * 32) + 28))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1421))], placeholder_shared[(((((int)threadIdx.z) * 32) + 29))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1470))], placeholder_shared[(((((int)threadIdx.z) * 32) + 30))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1519))], placeholder_shared[(((((int)threadIdx.z) * 32) + 31))], compute[(0)]);
  }
  T_relu[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_1_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  __shared__ float PaddedInput_shared[324];
  __shared__ float placeholder_shared[36];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  PaddedInput_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = (((((9 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81)) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) < 72)) && (1 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9))) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) < 8)) ? placeholder[((((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 81) * 49)) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) / 9) * 7)) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9)) - 8))] : 0.000000e+00f);
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 128) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 19) {
      if (((int)threadIdx.z) < 3) {
        PaddedInput_shared[(((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196))] = (((((9 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 34) % 81)) && ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 34) % 81) < 72)) && (1 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 7) % 9))) && ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 7) % 9) < 8)) ? placeholder[((((((((int)blockIdx.z) * 196) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196) / 81) * 49)) + (((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 34) % 81) / 9) * 7)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 7) % 9)) - 8))] : 0.000000e+00f);
      }
    }
  }
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 36) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 6) {
      if (((int)threadIdx.z) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = placeholder1[(((((((int)threadIdx.z) * 49) + (((int)blockIdx.z) * 36)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
      }
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 9))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 10))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 11))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 18))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 19))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 20))];
  placeholder_shared_local[(0)] = placeholder_shared[((((int)threadIdx.z) * 9))];
  placeholder_shared_local[(1)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 1))];
  placeholder_shared_local[(2)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 2))];
  placeholder_shared_local[(3)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 3))];
  placeholder_shared_local[(4)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 4))];
  placeholder_shared_local[(5)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 5))];
  placeholder_shared_local[(6)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 6))];
  placeholder_shared_local[(7)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 7))];
  placeholder_shared_local[(8)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 8))];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  T_relu[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((((int)blockIdx.z) * 4) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_5_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  __shared__ float PaddedInput_shared[512];
  __shared__ float placeholder_shared[18];
  float PaddedInput_shared_local[12];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[2];
  PaddedInput_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = ((((16 <= (((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x))) && (1 <= ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15))) && (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15) < 15)) ? placeholder[(((((((int)blockIdx.z) * 392) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 4) * 14)) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196))] = (((((16 <= (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196) & 255)) && ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196) & 255) < 240)) && (1 <= (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 4) & 15))) && ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 4) & 15) < 15)) ? placeholder[((((((((int)blockIdx.z) * 392) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196) >> 8) * 196)) + (((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196) & 255) >> 4) * 14)) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 4) & 15)) - 15))] : 0.000000e+00f);
  if ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 120) {
    if (((((int)threadIdx.z) * 14) + ((int)threadIdx.y)) < 18) {
      PaddedInput_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392))] = (((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 104) && (1 <= (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15))) && ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15) < 15)) ? placeholder[((((((((int)blockIdx.z) * 392) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392) >> 8) * 196)) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 136) >> 4) * 14)) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15)) - 15))] : 0.000000e+00f);
    }
  }
  if ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 18) {
    if (((((int)threadIdx.z) * 14) + ((int)threadIdx.y)) < 3) {
      if (((int)threadIdx.z) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = placeholder1[(((((((int)threadIdx.z) * 98) + (((int)blockIdx.z) * 18)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
      }
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 16))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 17))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 18))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 19))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 32))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 33))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 34))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 35))];
  placeholder_shared_local[(0)] = placeholder_shared[((((int)threadIdx.z) * 9))];
  placeholder_shared_local[(1)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 1))];
  placeholder_shared_local[(2)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 2))];
  placeholder_shared_local[(3)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 3))];
  placeholder_shared_local[(4)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 4))];
  placeholder_shared_local[(5)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 5))];
  placeholder_shared_local[(6)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 6))];
  placeholder_shared_local[(7)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 7))];
  placeholder_shared_local[(8)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 8))];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(0)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(1)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(2)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(3)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(4)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(5)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(6)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(7)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(11)], placeholder_shared_local[(8)], DepthwiseConv2d[(1)]);
  T_relu[(((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((((int)blockIdx.z) * 2) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = max((DepthwiseConv2d[(1)] + placeholder2[(((((int)blockIdx.z) * 2) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

__device__ void fused_nn_conv2d_add_nn_relu_10_kernel0_device(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2){
  float compute[4];
  __shared__ float pad_temp_shared[1792];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 112) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = placeholder[((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 112) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[(((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 4)) < 32) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) < 512) {
        if (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 8) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 4) * 128)) + (rc_outer * 16)) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 560))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 560))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 561))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 561))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 672))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 672))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 673))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 673))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 785))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 785))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1008))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1008))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1009))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1009))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1120))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1120))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1121))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1121))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1232))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1232))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1233))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1233))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1344))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1344))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1345))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1345))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1456))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1456))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1457))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1457))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1568))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1568))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1569))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1569))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1680))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1680))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1681))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1681))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(3)]);
  }
  T_relu[((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12544))] = max((compute[(2)] + placeholder2[((((((int)blockIdx.z) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12545))] = max((compute[(3)] + placeholder2[((((((int)blockIdx.z) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
}


extern "C" __global__  __attribute__((amdgpu_num_vgpr(61))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_12_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 4 + threadIdx.z * 4 * 4 >= 4 * 4 * 16) return;
    // if (blockIdx.x + blockIdx.y * 7 + blockIdx.z * 14 * 7 >= 7 * 14 * 2) return;
    fused_nn_conv2d_add_nn_relu_12_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_7_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 14 + threadIdx.z * 14 * 14 >= 14 * 14 * 1) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 1 * 1 >= 1 * 1 * 256) return;
    fused_nn_conv2d_add_nn_relu_7_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_6_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 7 + threadIdx.z * 2 * 7 >= 7 * 2 * 32) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 7 * 1 >= 1 * 7 * 16) return;
    fused_nn_conv2d_add_nn_relu_6_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(37))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_14_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 4 + threadIdx.z * 2 * 4 >= 4 * 2 * 32) return;
    // if (blockIdx.x + blockIdx.y * 7 + blockIdx.z * 14 * 7 >= 7 * 14 * 2) return;
    fused_nn_conv2d_add_nn_relu_14_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(63))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_18_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 16 + threadIdx.z * 1 * 16 >= 16 * 1 * 32) return;
    // if (blockIdx.x + blockIdx.y * 7 + blockIdx.z * 16 * 7 >= 7 * 16 * 1) return;
    fused_nn_conv2d_add_nn_relu_18_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_17_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 28 + threadIdx.z * 14 * 28 >= 28 * 14 * 1) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 8 * 1 >= 1 * 8 * 32) return;
    fused_nn_conv2d_add_nn_relu_17_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_11_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 28 + threadIdx.z * 28 * 28 >= 28 * 28 * 1) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 1 * 1 >= 1 * 1 * 128) return;
    fused_nn_conv2d_add_nn_relu_11_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(63))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_16_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 4 + threadIdx.z * 4 * 4 >= 4 * 4 * 16) return;
    // if (blockIdx.x + blockIdx.y * 7 + blockIdx.z * 28 * 7 >= 7 * 28 * 1) return;
    fused_nn_conv2d_add_nn_relu_16_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_2_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 7 + threadIdx.z * 7 * 7 >= 7 * 7 * 16) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 1 * 1 >= 1 * 1 * 64) return;
    fused_nn_conv2d_add_nn_relu_2_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_batch_flatten_kernel0_device_wrapper(float* __restrict__ tensor, float* __restrict__ placeholder) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 256 + threadIdx.z * 1 * 256 >= 256 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 4 + blockIdx.z * 1 * 4 >= 4 * 1 * 1) return;
    fused_nn_batch_flatten_kernel0_device((float* __restrict__)tensor,(float* __restrict__)placeholder);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_13_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 28 + threadIdx.z * 28 * 28 >= 28 * 28 * 1) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 1 * 1 >= 1 * 1 * 128) return;
    fused_nn_conv2d_add_nn_relu_13_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(33))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_8_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 14 + threadIdx.z * 4 * 14 >= 14 * 4 * 16) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 7 * 1 >= 1 * 7 * 8) return;
    fused_nn_conv2d_add_nn_relu_8_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_global_avg_pool2d_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ tensor) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 8 + threadIdx.z * 8 * 8 >= 8 * 8 * 1) return;
    // if (blockIdx.x + blockIdx.y * 128 + blockIdx.z * 1 * 128 >= 128 * 1 * 1) return;
    fused_nn_global_avg_pool2d_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)tensor);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_9_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 7 + threadIdx.z * 14 * 7 >= 7 * 14 * 4) return;
    // if (blockIdx.x + blockIdx.y * 2 + blockIdx.z * 2 * 2 >= 2 * 2 * 64) return;
    fused_nn_conv2d_add_nn_relu_9_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(27))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_dense_add_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 64 + threadIdx.z * 1 * 64 >= 64 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 1000 + blockIdx.z * 1 * 1000 >= 1000 * 1 * 1) return;
    fused_nn_dense_add_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_add,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_15_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 56 + threadIdx.z * 14 * 56 >= 56 * 14 * 1) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 4 * 1 >= 1 * 4 * 64) return;
    fused_nn_conv2d_add_nn_relu_15_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(35))) __attribute__((amdgpu_num_sgpr(54))) void fused_nn_softmax_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ T_softmax_norm) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 64 + threadIdx.z * 1 * 64 >= 64 * 1 * 1) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 1 * 1 >= 1 * 1 * 1) return;
    fused_nn_softmax_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)T_softmax_norm);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_3_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 7 + threadIdx.z * 7 * 7 >= 7 * 7 * 8) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 1 * 1 >= 1 * 1 * 64) return;
    fused_nn_conv2d_add_nn_relu_3_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(29))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_4_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 7 + threadIdx.z * 7 * 7 >= 7 * 7 * 8) return;
    // if (blockIdx.x + blockIdx.y * 2 + blockIdx.z * 2 * 2 >= 2 * 2 * 32) return;
    fused_nn_conv2d_add_nn_relu_4_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 7 + threadIdx.z * 7 * 7 >= 7 * 7 * 16) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 1 * 1 >= 1 * 1 * 64) return;
    fused_nn_conv2d_add_nn_relu_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_1_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 7 + threadIdx.z * 7 * 7 >= 7 * 7 * 4) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 1 * 1 >= 1 * 1 * 256) return;
    fused_nn_conv2d_add_nn_relu_1_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(25))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_5_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 7 + threadIdx.z * 14 * 7 >= 7 * 14 * 2) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 1 * 1 >= 1 * 1 * 256) return;
    fused_nn_conv2d_add_nn_relu_5_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  __attribute__((amdgpu_num_vgpr(33))) __attribute__((amdgpu_num_sgpr(30))) void fused_nn_conv2d_add_nn_relu_10_kernel0_device_wrapper(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
    // Force the compiler to use all the index
    if (threadIdx.x + threadIdx.y * 14 + threadIdx.z * 4 * 14 >= 14 * 4 * 16) return;
    // if (blockIdx.x + blockIdx.y * 1 + blockIdx.z * 7 * 1 >= 1 * 7 * 8) return;
    fused_nn_conv2d_add_nn_relu_10_kernel0_device((float* __restrict__)placeholder,(float* __restrict__)placeholder1,(float* __restrict__)T_relu,(float* __restrict__)placeholder2);
    asm volatile(";; end_flag"); // jump back to the caller
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_12_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_7_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_6_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_14_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_18_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_17_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_11_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_16_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_2_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_batch_flatten_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_13_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_8_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_global_avg_pool2d_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_9_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_dense_add_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_15_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_softmax_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_3_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_4_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_1_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_5_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __global__  void fused_nn_conv2d_add_nn_relu_10_kernel0(
    void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,
    void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,
    int cu_partition) {
    asm volatile(";; caller_flag");
    return;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_128_1_1(int idx) {
  dim3 dim(128, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_256_1_1(int idx) {
  dim3 dim(256, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_7_28_1(int idx) {
  dim3 dim(7, 28, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_16_1_32(int idx) {
  dim3 dim(16, 1, 32);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_2_2_32(int idx) {
  dim3 dim(2, 2, 32);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_4_2_32(int idx) {
  dim3 dim(4, 2, 32);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_7_7_8(int idx) {
  dim3 dim(7, 7, 8);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_1_7_16(int idx) {
  dim3 dim(1, 7, 16);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_1_1_64(int idx) {
  dim3 dim(1, 1, 64);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_28_14_1(int idx) {
  dim3 dim(28, 14, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_2_2_64(int idx) {
  dim3 dim(2, 2, 64);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_1_8_32(int idx) {
  dim3 dim(1, 8, 32);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_7_14_2(int idx) {
  dim3 dim(7, 14, 2);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_64_1_1(int idx) {
  dim3 dim(64, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_28_28_1(int idx) {
  dim3 dim(28, 28, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_4_1_1(int idx) {
  dim3 dim(4, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_56_14_1(int idx) {
  dim3 dim(56, 14, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_1_1_1(int idx) {
  dim3 dim(1, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_7_14_4(int idx) {
  dim3 dim(7, 14, 4);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_7_16_1(int idx) {
  dim3 dim(7, 16, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_8_8_1(int idx) {
  dim3 dim(8, 8, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_1_1_128(int idx) {
  dim3 dim(1, 1, 128);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_1_1_256(int idx) {
  dim3 dim(1, 1, 256);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_7_7_16(int idx) {
  dim3 dim(7, 7, 16);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_1_4_64(int idx) {
  dim3 dim(1, 4, 64);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_14_14_1(int idx) {
  dim3 dim(14, 14, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_7_7_4(int idx) {
  dim3 dim(7, 7, 4);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_7_2_32(int idx) {
  dim3 dim(7, 2, 32);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_14_4_16(int idx) {
  dim3 dim(14, 4, 16);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_1_7_8(int idx) {
  dim3 dim(1, 7, 8);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_1000_1_1(int idx) {
  dim3 dim(1000, 1, 1);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

extern "C" __device__ __noinline__ dim3 get_3d_idx_4_4_16(int idx) {
  dim3 dim(4, 4, 16);
  dim3 result;
  result.x = idx % dim.x;
  result.y = idx / dim.x % dim.y;
  result.z = idx / (dim.x * dim.y);
  return result;
}

__global__ void get_3d_idx_caller(int* buf) {
    dim3 task_idx;

    task_idx = get_3d_idx_128_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_256_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_7_28_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_16_1_32(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_2_2_32(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_4_2_32(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_7_7_8(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_1_7_16(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_1_1_64(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_28_14_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_2_2_64(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_1_8_32(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_7_14_2(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_64_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_28_28_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_4_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_56_14_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_1_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_7_14_4(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_7_16_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_8_8_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_1_1_128(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_1_1_256(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_7_7_16(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_1_4_64(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_14_14_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_7_7_4(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_7_2_32(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_14_4_16(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_1_7_8(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_1000_1_1(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

    task_idx = get_3d_idx_4_4_16(threadIdx.x);
    buf[task_idx.x] = task_idx.x;
    buf[task_idx.y] = task_idx.y;
    buf[task_idx.z] = task_idx.z;

}

#define CALL_FRAMEWORK(idx) \
extern "C" __global__ void call_framework_##idx(\
  void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,\
  void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,\
  int cu_partition) \
{\
  asm volatile(\
    "  s_load_dwordx2 s[14:15], s[4:5], 0x0\n"\
    "  s_waitcnt lgkmcnt(0)\n"\
    "  s_setpc_b64 s[14:15]\n"\
    "  s_endpgm\n"\
  );\
}

CALL_FRAMEWORK(1)
CALL_FRAMEWORK(2)
CALL_FRAMEWORK(3)
CALL_FRAMEWORK(4)
CALL_FRAMEWORK(5)
CALL_FRAMEWORK(6)
CALL_FRAMEWORK(7)
CALL_FRAMEWORK(8)
CALL_FRAMEWORK(9)
CALL_FRAMEWORK(10)

#define MERGE_FRAMEWORK(idx) \
extern "C" __global__ void merge_framework_##idx(\
  void* func_l, int layers_l, int task_num_l, int task_offset_l, float** param_l,\
  void* func_r, int layers_r, int task_num_r, int task_offset_r, float** param_r,\
  int cu_partition) \
{\
  asm volatile(\
    "  s_load_dword s10, s[4:5], 0x40\n"\
    "  s_load_dwordx2 s[12:13], s[4:5], 0x0\n"\
    "  s_load_dwordx2 s[14:15], s[4:5], 0x20\n"\
    "  s_mul_hi_u32 s11, s6, 0x88888889\n"\
    "  s_lshr_b32 s11, s11, 5\n"\
    "  s_mul_i32 s11, s11, 60\n"\
    "  s_sub_i32 s11, s6, s11\n"\
    "  s_waitcnt lgkmcnt(0)\n"\
    "  s_cmp_ge_u32 s11, s10\n"\
    "  s_mov_b64 s[10:11], -1\n"\
    "  s_cbranch_scc1 MyBB"#idx"_3\n"\
    "; %bb.1:                                ; %Flow\n"\
    "  s_andn2_b64 vcc, exec, s[10:11]\n"\
    "  s_cbranch_vccz MyBB"#idx"_4\n"\
    "  s_endpgm\n"\
    "MyBB"#idx"_3:\n"\
    "  s_setpc_b64 s[14:15]\n"\
    "  s_endpgm\n"\
    "MyBB"#idx"_4:\n"\
    "  s_setpc_b64 s[12:13]\n"\
    "  s_endpgm\n"\
  );\
}
MERGE_FRAMEWORK(1)
MERGE_FRAMEWORK(2)
MERGE_FRAMEWORK(3)
MERGE_FRAMEWORK(4)
MERGE_FRAMEWORK(5)
MERGE_FRAMEWORK(6)
MERGE_FRAMEWORK(7)
MERGE_FRAMEWORK(8)
MERGE_FRAMEWORK(9)
MERGE_FRAMEWORK(10)
MERGE_FRAMEWORK(nostack_1)
MERGE_FRAMEWORK(nostack_2)
MERGE_FRAMEWORK(nostack_3)
MERGE_FRAMEWORK(nostack_4)
MERGE_FRAMEWORK(nostack_5)
MERGE_FRAMEWORK(nostack_6)
MERGE_FRAMEWORK(nostack_7)
MERGE_FRAMEWORK(nostack_8)
MERGE_FRAMEWORK(nostack_9)
MERGE_FRAMEWORK(nostack_10)
