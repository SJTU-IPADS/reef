#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_12_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[8];
  __shared__ float pad_temp_shared[512];
  __shared__ float placeholder_shared[1024];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder[(((((((rc_outer * 50176) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[((((((((rc_outer * 50176) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))];
    placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + (((int)threadIdx.y) * 128)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder1[(((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + (((int)threadIdx.y) * 128)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 1))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder1[(((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + (((int)threadIdx.y) * 128)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 2))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder1[(((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + (((int)threadIdx.y) * 128)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 3))];
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(2)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(4)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(3)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(5)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(7)]);
  }
  T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 50176))] = max((compute[(2)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100352))] = max((compute[(4)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 150528))] = max((compute[(6)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 50177))] = max((compute[(3)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100353))] = max((compute[(5)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 150529))] = max((compute[(7)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 48))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_7_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[841];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) < 841) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 14) + ((int)threadIdx.y)) < 61) {
        PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] = (((29 <= (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x))) && (1 <= ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 29))) ? placeholder[(((((((int)blockIdx.z) * 784) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) / 29) * 28)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)) % 29)) - 29))] : 0.000000e+00f);
      }
    }
  }
  if (((((int)threadIdx.y) * 14) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 14) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 14) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  for (int ax2 = 0; ax2 < 3; ++ax2) {
    for (int ax3 = 0; ax3 < 3; ++ax3) {
      PaddedInput_shared_local[(((ax2 * 3) + ax3))] = PaddedInput_shared[(((((((int)threadIdx.y) * 58) + (ax2 * 29)) + (((int)threadIdx.x) * 2)) + ax3))];
    }
  }
  for (int ax21 = 0; ax21 < 3; ++ax21) {
    for (int ax31 = 0; ax31 < 3; ++ax31) {
      placeholder_shared_local[(((ax21 * 3) + ax31))] = placeholder_shared[(((ax21 * 3) + ax31))];
    }
  }
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  for (int di = 0; di < 3; ++di) {
    for (int dj = 0; dj < 3; ++dj) {
      DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(((di * 3) + dj))], placeholder_shared_local[(((di * 3) + dj))], DepthwiseConv2d[(0)]);
    }
  }
  T_relu[((((((int)blockIdx.z) * 196) + (((int)threadIdx.y) * 14)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_6_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[2];
  __shared__ float pad_temp_shared[448];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 14) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = placeholder[(((((((rc_outer * 3136) + ((((int)threadIdx.z) >> 1) * 196)) + (((int)blockIdx.y) * 28)) + ((((int)threadIdx.z) & 1) * 14)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
    if (((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) >> 4) + ((int)threadIdx.z)) < 32) {
      if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) < 512) {
        if (((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) < 16) {
          if (((int)threadIdx.x) < 4) {
            placeholder_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 256)) + (rc_outer * 16)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))];
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1) >> 4) + ((int)threadIdx.z)) < 32) {
      if ((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) < 511) {
        if (((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) < 15) {
          if (((int)threadIdx.x) < 4) {
            placeholder_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder1[(((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 256)) + (rc_outer * 16)) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 28))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 29))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 56))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 57))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 84))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 85))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 140))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 141))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 168))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 169))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 197))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 252))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 253))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 280))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 281))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 308))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 309))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 364))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 365))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 393))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 420))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 14) + (((int)threadIdx.x) * 2)) + 421))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
  }
  T_relu[((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 6272) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_14_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[8];
  __shared__ float pad_temp_shared[512];
  __shared__ float placeholder_shared[1024];
  compute[(0)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)))] = placeholder[(((((((rc_outer * 50176) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 16) + (((int)threadIdx.y) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[((((((((rc_outer * 50176) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) >> 2) * 3136)) + (((int)blockIdx.y) * 224)) + ((((((int)threadIdx.z) * 2) + ((int)threadIdx.y)) & 3) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))];
    placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)))] = placeholder1[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 64)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)))];
    placeholder_shared[(((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder1[(((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 64)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 1))];
    placeholder_shared[(((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder1[(((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 64)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 2))];
    placeholder_shared[(((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder1[(((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 128)) + (((int)threadIdx.y) * 64)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 3))];
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 16))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 16))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 17))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 17))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 32))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 48))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 48))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 33))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 80))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 80))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 81))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 81))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 96))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 97))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 144))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 144))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 145))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 145))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 160))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 176))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 176))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 161))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 177))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 177))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 208))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 208))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 209))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 209))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 240))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 240))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 241))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 241))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 272))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 272))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 273))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 273))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 288))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 304))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 304))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 289))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 305))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 305))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 352))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 368))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 368))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 353))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 369))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 369))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 400))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 400))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 401))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 401))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 416))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 432))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 432))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 417))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 433))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 433))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 464))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 464))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 465))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 465))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(7)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 480))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(4)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 496))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 496))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(6)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 481))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(5)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 497))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 8) + (((int)threadIdx.x) * 2)) + 497))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(7)]);
  }
  T_relu[(((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100352))] = max((compute[(4)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 112))] = max((compute[(2)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100464))] = max((compute[(6)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100353))] = max((compute[(5)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 113))] = max((compute[(3)] + placeholder2[(((((int)blockIdx.z) * 64) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 200704) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 224)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 8)) + (((int)threadIdx.x) * 2)) + 100465))] = max((compute[(7)] + placeholder2[((((((int)blockIdx.z) * 64) + ((int)threadIdx.z)) + 32))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_18_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[7];
  __shared__ float pad_temp_shared[495];
  __shared__ float placeholder_shared[288];
  for (int yy_init = 0; yy_init < 7; ++yy_init) {
    compute[(yy_init)] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 3; ++rc_outer) {
    __syncthreads();
    if (((((int)threadIdx.z) * 16) + ((int)threadIdx.x)) < 495) {
      pad_temp_shared[(((((int)threadIdx.z) * 16) + ((int)threadIdx.x)))] = (((1 <= ((((int)blockIdx.y) * 14) + (((((int)threadIdx.z) * 16) + ((int)threadIdx.x)) / 33))) && (1 <= ((((int)blockIdx.x) * 32) + (((((int)threadIdx.z) * 16) + ((int)threadIdx.x)) % 33)))) ? placeholder[(((((((rc_outer * 50176) + (((int)blockIdx.y) * 3136)) + ((((((int)threadIdx.z) * 16) + ((int)threadIdx.x)) / 33) * 224)) + (((int)blockIdx.x) * 32)) + (((((int)threadIdx.z) * 16) + ((int)threadIdx.x)) % 33)) - 225))] : 0.000000e+00f);
    }
    if (((((int)threadIdx.x) / 9) + ((int)threadIdx.z)) < 32) {
      if (((((int)threadIdx.z) * 3) + (((int)threadIdx.x) / 3)) < 96) {
        if (((((int)threadIdx.z) * 9) + ((int)threadIdx.x)) < 288) {
          if (((int)threadIdx.x) < 9) {
            placeholder_shared[(((((int)threadIdx.z) * 9) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.z) * 27) + (rc_outer * 9)) + ((int)threadIdx.x)))];
          }
        }
      }
    }
    __syncthreads();
    for (int ry_inner = 0; ry_inner < 3; ++ry_inner) {
      for (int rx_inner = 0; rx_inner < 3; ++rx_inner) {
        for (int yy = 0; yy < 7; ++yy) {
          compute[(yy)] = __ocml_fma_f32(pad_temp_shared[(((((yy * 66) + (ry_inner * 33)) + (((int)threadIdx.x) * 2)) + rx_inner))], placeholder_shared[((((((int)threadIdx.z) * 9) + (ry_inner * 3)) + rx_inner))], compute[(yy)]);
        }
      }
    }
  }
  for (int ax2_inner_inner_inner = 0; ax2_inner_inner_inner < 7; ++ax2_inner_inner_inner) {
    T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 784)) + (ax2_inner_inner_inner * 112)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)))] = max((compute[(ax2_inner_inner_inner)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_17_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[1824];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[18];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[4];
  PaddedInput_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = ((((1 <= ((((int)blockIdx.y) * 14) + (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) / 114))) && (1 <= (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 114))) && ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) / 114) * 112)) + (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 392))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 50) % 114)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 50) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 392) / 114) * 112)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 50) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 100) % 114)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 100) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784) / 114) * 112)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 100) % 114)) - 113))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1176))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 36) % 114)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 36) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1176) / 114) * 112)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 36) % 114)) - 113))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 256) {
    if (((int)threadIdx.y) < 10) {
      PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568))] = ((((((((int)blockIdx.y) * 14) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568) / 114)) < 113) && (1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 86) % 114))) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 86) % 114) < 113)) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568) / 114) * 112)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 86) % 114)) - 113))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 28) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 3))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 4))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 5))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 114))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 115))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 116))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 117))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 118))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 119))];
  PaddedInput_shared_local[(12)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 228))];
  PaddedInput_shared_local[(13)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 229))];
  PaddedInput_shared_local[(14)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 230))];
  PaddedInput_shared_local[(15)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 231))];
  PaddedInput_shared_local[(16)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 232))];
  PaddedInput_shared_local[(17)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 4)) + 233))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(12)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(13)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(14)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(0)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(1)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(2)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(3)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(4)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(5)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(13)], placeholder_shared_local[(6)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(14)], placeholder_shared_local[(7)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(15)], placeholder_shared_local[(8)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(0)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(1)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(2)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(3)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(4)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(5)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(14)], placeholder_shared_local[(6)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(15)], placeholder_shared_local[(7)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(16)], placeholder_shared_local[(8)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(0)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(1)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(2)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(3)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(4)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(11)], placeholder_shared_local[(5)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(15)], placeholder_shared_local[(6)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(16)], placeholder_shared_local[(7)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(17)], placeholder_shared_local[(8)], DepthwiseConv2d[(3)]);
  T_relu[(((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 4)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 4)) + 1))] = max((DepthwiseConv2d[(1)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 4)) + 2))] = max((DepthwiseConv2d[(2)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 1568)) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 4)) + 3))] = max((DepthwiseConv2d[(3)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_11_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[3249];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  PaddedInput_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = (((57 <= ((((int)threadIdx.y) * 28) + ((int)threadIdx.x))) && (1 <= (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 57))) ? placeholder[(((((((int)blockIdx.z) * 3136) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) / 57) * 56)) + (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 43) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 43) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 29) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 29) % 57)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2352))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 15) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2352) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 15) % 57)) - 57))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 113) {
    if (((int)threadIdx.y) < 5) {
      PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 3136))] = ((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1) % 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 3136) / 57) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1) % 57)) - 57))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 28) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 57))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 58))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 59))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 114))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 115))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 114) + (((int)threadIdx.x) * 2)) + 116))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  T_relu[((((((int)blockIdx.z) * 784) + (((int)threadIdx.y) * 28)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_16_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[16];
  __shared__ float pad_temp_shared[1024];
  __shared__ float placeholder_shared[1024];
  compute[(0)] = 0.000000e+00f;
  compute[(4)] = 0.000000e+00f;
  compute[(8)] = 0.000000e+00f;
  compute[(12)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(5)] = 0.000000e+00f;
  compute[(9)] = 0.000000e+00f;
  compute[(13)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(6)] = 0.000000e+00f;
  compute[(10)] = 0.000000e+00f;
  compute[(14)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  compute[(7)] = 0.000000e+00f;
  compute[(11)] = 0.000000e+00f;
  compute[(15)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)))] = placeholder[(((((((rc_outer * 200704) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder[((((((((rc_outer * 200704) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 1))];
    pad_temp_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder[((((((((rc_outer * 200704) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 2))];
    pad_temp_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder[((((((((rc_outer * 200704) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 3))];
    placeholder_shared[((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)))] = placeholder1[(((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 1))] = placeholder1[((((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 1))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 2))] = placeholder1[((((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 2))];
    placeholder_shared[(((((((int)threadIdx.z) * 64) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 4)) + 3))] = placeholder1[((((((((int)threadIdx.z) * 128) + (((int)threadIdx.y) * 32)) + (rc_outer * 16)) + (((int)threadIdx.x) * 4)) + 3))];
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 2))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 2))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 2))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 2))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 3))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 3))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 3))], placeholder_shared[(((((int)threadIdx.z) * 16) + 512))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 3))], placeholder_shared[(((((int)threadIdx.z) * 16) + 768))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 64))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 65))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 66))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 66))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 66))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 66))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 67))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 67))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 67))], placeholder_shared[(((((int)threadIdx.z) * 16) + 513))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 67))], placeholder_shared[(((((int)threadIdx.z) * 16) + 769))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 128))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 129))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 130))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 130))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 130))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 130))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 131))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 131))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 131))], placeholder_shared[(((((int)threadIdx.z) * 16) + 514))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 131))], placeholder_shared[(((((int)threadIdx.z) * 16) + 770))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 192))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 193))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 194))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 194))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 194))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 194))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 195))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 195))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 195))], placeholder_shared[(((((int)threadIdx.z) * 16) + 515))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 195))], placeholder_shared[(((((int)threadIdx.z) * 16) + 771))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 256))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 257))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 258))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 258))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 258))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 258))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 259))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 259))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 259))], placeholder_shared[(((((int)threadIdx.z) * 16) + 516))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 259))], placeholder_shared[(((((int)threadIdx.z) * 16) + 772))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 320))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 321))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 322))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 322))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 322))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 322))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 323))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 323))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 323))], placeholder_shared[(((((int)threadIdx.z) * 16) + 517))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 323))], placeholder_shared[(((((int)threadIdx.z) * 16) + 773))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 384))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 385))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 386))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 386))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 386))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 386))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 387))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 387))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 387))], placeholder_shared[(((((int)threadIdx.z) * 16) + 518))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 387))], placeholder_shared[(((((int)threadIdx.z) * 16) + 774))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 450))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 450))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 450))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 450))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 451))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 451))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 451))], placeholder_shared[(((((int)threadIdx.z) * 16) + 519))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 451))], placeholder_shared[(((((int)threadIdx.z) * 16) + 775))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 512))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 512))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 512))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 512))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 513))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 513))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 513))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 513))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 514))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 514))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 514))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 514))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 515))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 515))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 515))], placeholder_shared[(((((int)threadIdx.z) * 16) + 520))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 515))], placeholder_shared[(((((int)threadIdx.z) * 16) + 776))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 576))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 576))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 576))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 576))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 577))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 577))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 577))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 577))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 578))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 578))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 578))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 578))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 579))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 579))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 579))], placeholder_shared[(((((int)threadIdx.z) * 16) + 521))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 579))], placeholder_shared[(((((int)threadIdx.z) * 16) + 777))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 640))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 640))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 640))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 640))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 641))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 641))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 641))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 641))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 642))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 642))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 642))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 642))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 643))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 643))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 643))], placeholder_shared[(((((int)threadIdx.z) * 16) + 522))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 643))], placeholder_shared[(((((int)threadIdx.z) * 16) + 778))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 704))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 704))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 704))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 704))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 705))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 705))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 705))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 705))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 706))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 706))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 706))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 706))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 707))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 707))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 707))], placeholder_shared[(((((int)threadIdx.z) * 16) + 523))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 707))], placeholder_shared[(((((int)threadIdx.z) * 16) + 779))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 768))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 768))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 768))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 768))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 769))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 769))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 769))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 769))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 770))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 770))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 770))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 770))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 771))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 771))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 771))], placeholder_shared[(((((int)threadIdx.z) * 16) + 524))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 771))], placeholder_shared[(((((int)threadIdx.z) * 16) + 780))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 832))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 832))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 832))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 832))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 834))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 834))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 834))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 834))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 835))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 835))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 835))], placeholder_shared[(((((int)threadIdx.z) * 16) + 525))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 835))], placeholder_shared[(((((int)threadIdx.z) * 16) + 781))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 898))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 898))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 898))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 898))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 899))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 899))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 899))], placeholder_shared[(((((int)threadIdx.z) * 16) + 526))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 899))], placeholder_shared[(((((int)threadIdx.z) * 16) + 782))], compute[(15)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 960))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(4)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 960))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(4)]);
    compute[(8)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 960))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(8)]);
    compute[(12)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 960))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(12)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 961))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
    compute[(5)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 961))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(5)]);
    compute[(9)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 961))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(9)]);
    compute[(13)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 961))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(13)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 962))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(2)]);
    compute[(6)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 962))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(6)]);
    compute[(10)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 962))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(10)]);
    compute[(14)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 962))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(14)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 963))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(3)]);
    compute[(7)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 963))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(7)]);
    compute[(11)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 963))], placeholder_shared[(((((int)threadIdx.z) * 16) + 527))], compute[(11)]);
    compute[(15)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 16) + (((int)threadIdx.x) * 4)) + 963))], placeholder_shared[(((((int)threadIdx.z) * 16) + 783))], compute[(15)]);
  }
  T_relu[((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)))] = max((compute[(0)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 200704))] = max((compute[(4)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 401408))] = max((compute[(8)] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 602112))] = max((compute[(12)] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 1))] = max((compute[(1)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 200705))] = max((compute[(5)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 401409))] = max((compute[(9)] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 602113))] = max((compute[(13)] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 2))] = max((compute[(2)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 200706))] = max((compute[(6)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 401410))] = max((compute[(10)] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 602114))] = max((compute[(14)] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 3))] = max((compute[(3)] + placeholder2[(((int)threadIdx.z))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 200707))] = max((compute[(7)] + placeholder2[((((int)threadIdx.z) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 401411))] = max((compute[(11)] + placeholder2[((((int)threadIdx.z) + 32))]), 0.000000e+00f);
  T_relu[(((((((((int)threadIdx.z) * 12544) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 602115))] = max((compute[(15)] + placeholder2[((((int)threadIdx.z) + 48))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_2_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[1];
  __shared__ float pad_temp_shared[1568];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))] = placeholder[(((((rc_outer * 1568) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[(((((rc_outer * 1568) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 14)) + ((((int)threadIdx.x) * 2) + 1)))];
    if (((((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 5) + ((int)threadIdx.z)) < 16) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)) < 512) {
        if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 5) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + (rc_outer * 32)) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[((((int)threadIdx.z) * 32))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 32) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))], placeholder_shared[(((((int)threadIdx.z) * 32) + 2))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))], placeholder_shared[(((((int)threadIdx.z) * 32) + 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 32) + 4))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 245))], placeholder_shared[(((((int)threadIdx.z) * 32) + 5))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 294))], placeholder_shared[(((((int)threadIdx.z) * 32) + 6))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 343))], placeholder_shared[(((((int)threadIdx.z) * 32) + 7))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 32) + 8))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))], placeholder_shared[(((((int)threadIdx.z) * 32) + 9))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 490))], placeholder_shared[(((((int)threadIdx.z) * 32) + 10))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 539))], placeholder_shared[(((((int)threadIdx.z) * 32) + 11))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 588))], placeholder_shared[(((((int)threadIdx.z) * 32) + 12))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))], placeholder_shared[(((((int)threadIdx.z) * 32) + 13))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 686))], placeholder_shared[(((((int)threadIdx.z) * 32) + 14))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 735))], placeholder_shared[(((((int)threadIdx.z) * 32) + 15))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 32) + 16))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 32) + 17))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 882))], placeholder_shared[(((((int)threadIdx.z) * 32) + 18))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 931))], placeholder_shared[(((((int)threadIdx.z) * 32) + 19))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 980))], placeholder_shared[(((((int)threadIdx.z) * 32) + 20))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1029))], placeholder_shared[(((((int)threadIdx.z) * 32) + 21))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1078))], placeholder_shared[(((((int)threadIdx.z) * 32) + 22))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1127))], placeholder_shared[(((((int)threadIdx.z) * 32) + 23))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1176))], placeholder_shared[(((((int)threadIdx.z) * 32) + 24))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1225))], placeholder_shared[(((((int)threadIdx.z) * 32) + 25))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1274))], placeholder_shared[(((((int)threadIdx.z) * 32) + 26))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1323))], placeholder_shared[(((((int)threadIdx.z) * 32) + 27))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1372))], placeholder_shared[(((((int)threadIdx.z) * 32) + 28))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1421))], placeholder_shared[(((((int)threadIdx.z) * 32) + 29))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1470))], placeholder_shared[(((((int)threadIdx.z) * 32) + 30))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1519))], placeholder_shared[(((((int)threadIdx.z) * 32) + 31))], compute[(0)]);
  }
  T_relu[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_batch_flatten_kernel0(float* __restrict__ tensor, float* __restrict__ placeholder) {
  tensor[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))] = placeholder[(((((int)blockIdx.x) * 256) + ((int)threadIdx.x)))];
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_13_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[3364];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[16];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[4];
  PaddedInput_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = ((((58 <= ((((int)threadIdx.y) * 28) + ((int)threadIdx.x))) && (1 <= (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 58))) && ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 58) < 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) / 58) * 56)) + (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 30) % 58)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 30) % 58) < 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 784) / 58) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 30) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2) % 58)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2) % 58) < 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1568) / 58) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2) % 58)) - 57))] : 0.000000e+00f);
  PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2352))] = (((1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 32) % 58)) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 32) % 58) < 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2352) / 58) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 32) % 58)) - 57))] : 0.000000e+00f);
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 228) {
    if (((int)threadIdx.y) < 9) {
      PaddedInput_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 3136))] = ((((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 170) && (1 <= ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 4) % 58))) && (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 4) % 58) < 57)) ? placeholder[(((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 3136) / 58) * 56)) + ((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 4) % 58)) - 57))] : 0.000000e+00f);
    }
  }
  if (((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 28) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[(((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 3))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 58))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 59))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 60))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 61))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 116))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 117))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 118))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 119))];
  PaddedInput_shared_local[(12)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 174))];
  PaddedInput_shared_local[(13)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 175))];
  PaddedInput_shared_local[(14)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 176))];
  PaddedInput_shared_local[(15)] = PaddedInput_shared[((((((int)threadIdx.y) * 116) + (((int)threadIdx.x) * 2)) + 177))];
  placeholder_shared_local[(0)] = placeholder_shared[(0)];
  placeholder_shared_local[(1)] = placeholder_shared[(1)];
  placeholder_shared_local[(2)] = placeholder_shared[(2)];
  placeholder_shared_local[(3)] = placeholder_shared[(3)];
  placeholder_shared_local[(4)] = placeholder_shared[(4)];
  placeholder_shared_local[(5)] = placeholder_shared[(5)];
  placeholder_shared_local[(6)] = placeholder_shared[(6)];
  placeholder_shared_local[(7)] = placeholder_shared[(7)];
  placeholder_shared_local[(8)] = placeholder_shared[(8)];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(0)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(1)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(2)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(3)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(4)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(5)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(6)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(7)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(11)], placeholder_shared_local[(8)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(2)] = 0.000000e+00f;
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(0)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(1)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(2)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(3)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(4)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(5)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(12)], placeholder_shared_local[(6)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(13)], placeholder_shared_local[(7)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(2)] = __ocml_fma_f32(PaddedInput_shared_local[(14)], placeholder_shared_local[(8)], DepthwiseConv2d[(2)]);
  DepthwiseConv2d[(3)] = 0.000000e+00f;
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(0)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(1)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(2)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(3)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(4)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(11)], placeholder_shared_local[(5)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(13)], placeholder_shared_local[(6)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(14)], placeholder_shared_local[(7)], DepthwiseConv2d[(3)]);
  DepthwiseConv2d[(3)] = __ocml_fma_f32(PaddedInput_shared_local[(15)], placeholder_shared_local[(8)], DepthwiseConv2d[(3)]);
  T_relu[((((((int)blockIdx.z) * 3136) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 2)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 2)) + 1))] = max((DepthwiseConv2d[(1)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 2)) + 56))] = max((DepthwiseConv2d[(2)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)threadIdx.y) * 112)) + (((int)threadIdx.x) * 2)) + 57))] = max((DepthwiseConv2d[(3)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_8_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[4];
  __shared__ float pad_temp_shared[1792];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 112) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = placeholder[((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 112) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[(((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 4)) < 32) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) < 512) {
        if (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 8) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 512)) + ((((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 4) * 256)) + (rc_outer * 16)) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 560))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 560))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 561))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 561))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 672))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 672))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 673))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 673))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 785))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 785))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1008))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1008))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1009))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1009))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1120))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1120))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1121))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1121))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1232))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1232))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1233))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1233))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1344))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1344))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1345))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1345))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1456))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1456))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1457))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1457))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1568))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1568))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1569))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1569))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1680))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1680))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1681))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1681))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(3)]);
  }
  T_relu[((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12544))] = max((compute[(2)] + placeholder2[((((((int)blockIdx.z) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12545))] = max((compute[(3)] + placeholder2[((((((int)blockIdx.z) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_global_avg_pool2d_kernel0(float* __restrict__ placeholder, float* __restrict__ tensor) {
  float tensor1[1];
  tensor1[(0)] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 7; ++rv0) {
    for (int rv1 = 0; rv1 < 7; ++rv1) {
      if (((int)threadIdx.y) < 1) {
        tensor1[(0)] = (tensor1[(0)] + placeholder[((((((((int)threadIdx.y) * 50176) + (((int)blockIdx.x) * 392)) + (((int)threadIdx.x) * 49)) + (rv0 * 7)) + rv1))]);
      }
    }
  }
  if (((int)threadIdx.y) < 1) {
    tensor[((((((int)threadIdx.y) * 1024) + (((int)blockIdx.x) * 8)) + ((int)threadIdx.x)))] = (tensor1[(0)] * 2.040816e-02f);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_9_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[1024];
  __shared__ float placeholder_shared[36];
  float PaddedInput_shared_local[12];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[2];
  PaddedInput_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = (((((1 <= ((((int)blockIdx.y) * 14) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)) < 29)) ? placeholder[((((((((((int)blockIdx.z) * 3136) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)) - 29))] : 0.000000e+00f);
  PaddedInput_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392))] = (((((1 <= ((((int)blockIdx.y) * 14) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 136) & 255) >> 4))) && (((((int)blockIdx.y) * 14) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 136) & 255) >> 4)) < 29)) && (1 <= ((((int)blockIdx.x) * 14) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15)))) && (((((int)blockIdx.x) * 14) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15)) < 29)) ? placeholder[((((((((((int)blockIdx.z) * 3136) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + (((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 136) & 255) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15)) - 29))] : 0.000000e+00f);
  if ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 240) {
    if (((((int)threadIdx.z) * 14) + ((int)threadIdx.y)) < 35) {
      if (((int)threadIdx.z) < 3) {
        PaddedInput_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 784))] = ((((((((int)blockIdx.y) * 14) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 16) >> 4)) < 29) && (1 <= ((((int)blockIdx.x) * 14) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)))) && (((((int)blockIdx.x) * 14) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)) < 29)) ? placeholder[((((((((((int)blockIdx.z) * 3136) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 784) >> 8) * 784)) + (((int)blockIdx.y) * 392)) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 16) >> 4) * 28)) + (((int)blockIdx.x) * 14)) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)) - 29))] : 0.000000e+00f);
      }
    }
  }
  if ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 36) {
    if (((((int)threadIdx.z) * 14) + ((int)threadIdx.y)) < 6) {
      if (((int)threadIdx.z) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = placeholder1[(((((((int)threadIdx.z) * 98) + (((int)blockIdx.z) * 36)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
      }
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 16))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 17))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 18))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 19))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 32))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 33))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 34))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 35))];
  placeholder_shared_local[(0)] = placeholder_shared[((((int)threadIdx.z) * 9))];
  placeholder_shared_local[(1)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 1))];
  placeholder_shared_local[(2)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 2))];
  placeholder_shared_local[(3)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 3))];
  placeholder_shared_local[(4)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 4))];
  placeholder_shared_local[(5)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 5))];
  placeholder_shared_local[(6)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 6))];
  placeholder_shared_local[(7)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 7))];
  placeholder_shared_local[(8)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 8))];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(0)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(1)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(2)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(3)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(4)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(5)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(6)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(7)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(11)], placeholder_shared_local[(8)], DepthwiseConv2d[(1)]);
  T_relu[(((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((((int)blockIdx.z) * 4) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 392)) + (((int)threadIdx.y) * 28)) + (((int)blockIdx.x) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = max((DepthwiseConv2d[(1)] + placeholder2[(((((int)blockIdx.z) * 4) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_dense_add_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_add, float* __restrict__ placeholder2) {
  float T_dense_rf[1];
  float red_buf0[1];
  __shared__ float T_dense[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 16; ++k_outer) {
    T_dense_rf[(0)] = __ocml_fma_f32(placeholder[(((k_outer * 64) + ((int)threadIdx.x)))], placeholder1[((((((int)blockIdx.x) * 1024) + (k_outer * 64)) + ((int)threadIdx.x)))], T_dense_rf[(0)]);
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = T_dense_rf[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  if (((int)threadIdx.x) == 0) {
    T_dense[(0)] = red_buf0[(0)];
  }
  if (((int)threadIdx.x) == 0) {
    T_add[(((int)blockIdx.x))] = (T_dense[(0)] + placeholder2[(((int)blockIdx.x))]);
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_15_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[3277];
  __shared__ float placeholder_shared[9];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  for (int ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer = 0; ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer) {
    if ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) < 3277) {
      if (((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 14) + ((int)threadIdx.y)) < 59) {
        PaddedInput_shared[((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)))] = (((1 <= ((((int)blockIdx.y) * 28) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) / 113))) && (1 <= ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) % 113))) ? placeholder[((((((((int)blockIdx.z) * 12544) + (((int)blockIdx.y) * 3136)) + (((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) / 113) * 112)) + ((((ax0_ax1_fused_ax2_fused_ax3_fused_outer_outer_outer * 784) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)) % 113)) - 113))] : 0.000000e+00f);
      }
    }
  }
  if (((((int)threadIdx.y) * 56) + ((int)threadIdx.x)) < 9) {
    if (((int)threadIdx.y) < 1) {
      placeholder_shared[(((((int)threadIdx.y) * 56) + ((int)threadIdx.x)))] = placeholder1[((((((int)threadIdx.y) * 56) + (((int)blockIdx.z) * 9)) + ((int)threadIdx.x)))];
    }
  }
  __syncthreads();
  for (int ax2 = 0; ax2 < 3; ++ax2) {
    for (int ax3 = 0; ax3 < 3; ++ax3) {
      PaddedInput_shared_local[(((ax2 * 3) + ax3))] = PaddedInput_shared[(((((((int)threadIdx.y) * 226) + (ax2 * 113)) + (((int)threadIdx.x) * 2)) + ax3))];
    }
  }
  for (int ax21 = 0; ax21 < 3; ++ax21) {
    for (int ax31 = 0; ax31 < 3; ++ax31) {
      placeholder_shared_local[(((ax21 * 3) + ax31))] = placeholder_shared[(((ax21 * 3) + ax31))];
    }
  }
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  for (int di = 0; di < 3; ++di) {
    for (int dj = 0; dj < 3; ++dj) {
      DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(((di * 3) + dj))], placeholder_shared_local[(((di * 3) + dj))], DepthwiseConv2d[(0)]);
    }
  }
  T_relu[(((((((int)blockIdx.z) * 3136) + (((int)blockIdx.y) * 784)) + (((int)threadIdx.y) * 56)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((int)blockIdx.z))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_softmax_kernel0(float* __restrict__ placeholder, float* __restrict__ T_softmax_norm) {
  float normal_reduce_temp0[1];
  float red_buf0[1];
  float T_softmax_exp[16];
  float normal_reduce_temp01[1];
  float red_buf01[1];
  normal_reduce_temp0[(0)] = -3.402823e+38f;
  for (int k_inner = 0; k_inner < 16; ++k_inner) {
    if (((((int)threadIdx.x) * 16) + k_inner) < 1000) {
      normal_reduce_temp0[(0)] = max(normal_reduce_temp0[(0)], placeholder[(((((int)threadIdx.x) * 16) + k_inner))]);
    }
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = normal_reduce_temp0[(0)];
  ((int*)mask)[(0)] = 0;
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  t0[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf0[(0)]);
  red_buf0[(0)] = max(red_buf0[(0)], t0[(0)]);
  red_buf0[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf0[(0)]);
  for (int i1_inner_outer = 0; i1_inner_outer < 4; ++i1_inner_outer) {
    for (int i1_inner_inner_s = 0; i1_inner_inner_s < 4; ++i1_inner_inner_s) {
      if ((((((int)threadIdx.x) * 16) + (i1_inner_outer * 4)) + i1_inner_inner_s) < 1000) {
        T_softmax_exp[(((i1_inner_outer * 4) + i1_inner_inner_s))] = __ocml_exp_f32((placeholder[((((((int)threadIdx.x) * 16) + (i1_inner_outer * 4)) + i1_inner_inner_s))] - red_buf0[(0)]));
      }
    }
  }
  normal_reduce_temp01[(0)] = 0.000000e+00f;
  for (int k_inner1 = 0; k_inner1 < 16; ++k_inner1) {
    if (((((int)threadIdx.x) * 16) + k_inner1) < 1000) {
      normal_reduce_temp01[(0)] = (normal_reduce_temp01[(0)] + __hip_ds_bpermute(((((int)threadIdx.x) + (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63))) << 2), T_softmax_exp[(k_inner1)]));
    }
  }
  unsigned int mask1[1];
  float t01[1];
  red_buf01[(0)] = normal_reduce_temp01[(0)];
  ((int*)mask1)[(0)] = 0;
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 32) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 32)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 16) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 16)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 8) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 8)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 4) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 4)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 2) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 2)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  t01[(0)] = __hip_ds_bpermute((((((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & 63) + 1) >= 64) ? __mbcnt_hi(-1, __mbcnt_lo(-1, 0)) : (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) + 1)) << 2), red_buf01[(0)]);
  red_buf01[(0)] = (red_buf01[(0)] + t01[(0)]);
  red_buf01[(0)] = __hip_ds_bpermute(((__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63)) << 2), red_buf01[(0)]);
  for (int i1_inner_outer1 = 0; i1_inner_outer1 < 4; ++i1_inner_outer1) {
    for (int i1_inner_inner_s1 = 0; i1_inner_inner_s1 < 4; ++i1_inner_inner_s1) {
      if ((((((int)threadIdx.x) * 16) + (i1_inner_outer1 * 4)) + i1_inner_inner_s1) < 1000) {
        T_softmax_norm[((((((int)threadIdx.x) * 16) + (i1_inner_outer1 * 4)) + i1_inner_inner_s1))] = (__hip_ds_bpermute(((((int)threadIdx.x) + (__mbcnt_hi(-1, __mbcnt_lo(-1, 0)) & (~63))) << 2), T_softmax_exp[(((i1_inner_outer1 * 4) + i1_inner_inner_s1))]) / red_buf01[(0)]);
      }
    }
  }
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_3_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[1800];
  __shared__ float placeholder_shared[72];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  PaddedInput_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = (((15 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 225)) && (1 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 15))) ? placeholder[((((((((int)blockIdx.z) * 1568) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 225) * 196)) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 225) / 15) * 14)) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[(((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392))] = (((15 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 167) % 225)) && (1 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 2) % 15))) ? placeholder[((((((((int)blockIdx.z) * 1568) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392) / 225) * 196)) + (((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 167) % 225) / 15) * 14)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 2) % 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[(((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 784))] = (((15 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 109) % 225)) && (1 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 4) % 15))) ? placeholder[((((((((int)blockIdx.z) * 1568) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 784) / 225) * 196)) + (((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 109) % 225) / 15) * 14)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 4) % 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[(((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 1176))] = (((15 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 51) % 225)) && (1 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 6) % 15))) ? placeholder[((((((((int)blockIdx.z) * 1568) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 1176) / 225) * 196)) + (((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 51) % 225) / 15) * 14)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 6) % 15)) - 15))] : 0.000000e+00f);
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 232) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 34) {
      if (((int)threadIdx.z) < 5) {
        PaddedInput_shared[(((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 1568))] = (((15 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 218) % 225)) && (1 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) % 15))) ? placeholder[((((((((int)blockIdx.z) * 1568) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 1568) / 225) * 196)) + (((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 218) % 225) / 15) * 14)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) % 15)) - 15))] : 0.000000e+00f);
      }
    }
  }
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 72) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 11) {
      if (((int)threadIdx.z) < 2) {
        placeholder_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = placeholder1[(((((((int)blockIdx.z) * 72) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
      }
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 15))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 16))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 17))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 30))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 31))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[(((((((int)threadIdx.z) * 225) + (((int)threadIdx.y) * 30)) + (((int)threadIdx.x) * 2)) + 32))];
  placeholder_shared_local[(0)] = placeholder_shared[((((int)threadIdx.z) * 9))];
  placeholder_shared_local[(1)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 1))];
  placeholder_shared_local[(2)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 2))];
  placeholder_shared_local[(3)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 3))];
  placeholder_shared_local[(4)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 4))];
  placeholder_shared_local[(5)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 5))];
  placeholder_shared_local[(6)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 6))];
  placeholder_shared_local[(7)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 7))];
  placeholder_shared_local[(8)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 8))];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  T_relu[(((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((((int)blockIdx.z) * 8) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_4_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[2];
  __shared__ float pad_temp_shared[784];
  __shared__ float placeholder_shared[256];
  compute[(0)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))] = placeholder[((((((((rc_outer * 3136) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.y) * 2) + ((((int)threadIdx.x) * 2) / 7)) / 7) * 196)) + (((int)blockIdx.y) * 98)) + ((((((int)threadIdx.y) * 2) + ((((int)threadIdx.x) * 2) / 7)) % 7) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 2) % 7)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[((((((((rc_outer * 3136) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) / 7)) / 7) * 196)) + (((int)blockIdx.y) * 98)) + ((((((int)threadIdx.y) * 2) + (((((int)threadIdx.x) * 2) + 1) / 7)) % 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 2) + 1) % 7)))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 4)) < 16) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)) < 256) {
        if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 5) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 8192) + (((int)threadIdx.z) * 1024)) + ((((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 4) * 512)) + (rc_outer * 16)) + (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 128))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 16) + 129))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))], placeholder_shared[(((((int)threadIdx.z) * 16) + 130))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))], placeholder_shared[(((((int)threadIdx.z) * 16) + 131))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 16) + 132))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 245))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 245))], placeholder_shared[(((((int)threadIdx.z) * 16) + 133))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 294))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 294))], placeholder_shared[(((((int)threadIdx.z) * 16) + 134))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 343))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 343))], placeholder_shared[(((((int)threadIdx.z) * 16) + 135))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 16) + 136))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))], placeholder_shared[(((((int)threadIdx.z) * 16) + 137))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 490))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 490))], placeholder_shared[(((((int)threadIdx.z) * 16) + 138))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 539))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 539))], placeholder_shared[(((((int)threadIdx.z) * 16) + 139))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 588))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 588))], placeholder_shared[(((((int)threadIdx.z) * 16) + 140))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))], placeholder_shared[(((((int)threadIdx.z) * 16) + 141))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 686))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 686))], placeholder_shared[(((((int)threadIdx.z) * 16) + 142))], compute[(1)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 735))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 735))], placeholder_shared[(((((int)threadIdx.z) * 16) + 143))], compute[(1)]);
  }
  T_relu[(((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((((int)blockIdx.z) * 3136) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 98)) + (((int)threadIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 1568))] = max((compute[(1)] + placeholder2[((((((int)blockIdx.z) * 16) + ((int)threadIdx.z)) + 8))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[1];
  __shared__ float pad_temp_shared[1568];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))] = placeholder[(((((rc_outer * 1568) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[(((((rc_outer * 1568) + (((int)threadIdx.z) * 98)) + (((int)threadIdx.y) * 14)) + ((((int)threadIdx.x) * 2) + 1)))];
    if (((((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) >> 5) + ((int)threadIdx.z)) < 16) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)) < 512) {
        if (((((int)threadIdx.y) * 5) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 5) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 16384) + (((int)threadIdx.z) * 1024)) + (rc_outer * 32)) + (((int)threadIdx.y) * 5)) + ((int)threadIdx.x)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))], placeholder_shared[((((int)threadIdx.z) * 32))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 49))], placeholder_shared[(((((int)threadIdx.z) * 32) + 1))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 98))], placeholder_shared[(((((int)threadIdx.z) * 32) + 2))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 147))], placeholder_shared[(((((int)threadIdx.z) * 32) + 3))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))], placeholder_shared[(((((int)threadIdx.z) * 32) + 4))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 245))], placeholder_shared[(((((int)threadIdx.z) * 32) + 5))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 294))], placeholder_shared[(((((int)threadIdx.z) * 32) + 6))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 343))], placeholder_shared[(((((int)threadIdx.z) * 32) + 7))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))], placeholder_shared[(((((int)threadIdx.z) * 32) + 8))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))], placeholder_shared[(((((int)threadIdx.z) * 32) + 9))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 490))], placeholder_shared[(((((int)threadIdx.z) * 32) + 10))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 539))], placeholder_shared[(((((int)threadIdx.z) * 32) + 11))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 588))], placeholder_shared[(((((int)threadIdx.z) * 32) + 12))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))], placeholder_shared[(((((int)threadIdx.z) * 32) + 13))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 686))], placeholder_shared[(((((int)threadIdx.z) * 32) + 14))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 735))], placeholder_shared[(((((int)threadIdx.z) * 32) + 15))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 32) + 16))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 833))], placeholder_shared[(((((int)threadIdx.z) * 32) + 17))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 882))], placeholder_shared[(((((int)threadIdx.z) * 32) + 18))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 931))], placeholder_shared[(((((int)threadIdx.z) * 32) + 19))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 980))], placeholder_shared[(((((int)threadIdx.z) * 32) + 20))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1029))], placeholder_shared[(((((int)threadIdx.z) * 32) + 21))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1078))], placeholder_shared[(((((int)threadIdx.z) * 32) + 22))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1127))], placeholder_shared[(((((int)threadIdx.z) * 32) + 23))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1176))], placeholder_shared[(((((int)threadIdx.z) * 32) + 24))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1225))], placeholder_shared[(((((int)threadIdx.z) * 32) + 25))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1274))], placeholder_shared[(((((int)threadIdx.z) * 32) + 26))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1323))], placeholder_shared[(((((int)threadIdx.z) * 32) + 27))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1372))], placeholder_shared[(((((int)threadIdx.z) * 32) + 28))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1421))], placeholder_shared[(((((int)threadIdx.z) * 32) + 29))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1470))], placeholder_shared[(((((int)threadIdx.z) * 32) + 30))], compute[(0)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1519))], placeholder_shared[(((((int)threadIdx.z) * 32) + 31))], compute[(0)]);
  }
  T_relu[(((((((int)blockIdx.z) * 784) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 16) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_1_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[324];
  __shared__ float placeholder_shared[36];
  float PaddedInput_shared_local[9];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[1];
  PaddedInput_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = (((((9 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81)) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) < 72)) && (1 <= ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9))) && (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9) < 8)) ? placeholder[((((((((int)blockIdx.z) * 196) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) / 81) * 49)) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 81) / 9) * 7)) + ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) % 9)) - 8))] : 0.000000e+00f);
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 128) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 19) {
      if (((int)threadIdx.z) < 3) {
        PaddedInput_shared[(((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196))] = (((((9 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 34) % 81)) && ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 34) % 81) < 72)) && (1 <= (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 7) % 9))) && ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 7) % 9) < 8)) ? placeholder[((((((((int)blockIdx.z) * 196) + ((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196) / 81) * 49)) + (((((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 34) % 81) / 9) * 7)) + (((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 7) % 9)) - 8))] : 0.000000e+00f);
      }
    }
  }
  if ((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 36) {
    if (((((int)threadIdx.z) * 7) + ((int)threadIdx.y)) < 6) {
      if (((int)threadIdx.z) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 49) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = placeholder1[(((((((int)threadIdx.z) * 49) + (((int)blockIdx.z) * 36)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
      }
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 9))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 10))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 11))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 18))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 19))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[(((((((int)threadIdx.z) * 81) + (((int)threadIdx.y) * 9)) + ((int)threadIdx.x)) + 20))];
  placeholder_shared_local[(0)] = placeholder_shared[((((int)threadIdx.z) * 9))];
  placeholder_shared_local[(1)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 1))];
  placeholder_shared_local[(2)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 2))];
  placeholder_shared_local[(3)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 3))];
  placeholder_shared_local[(4)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 4))];
  placeholder_shared_local[(5)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 5))];
  placeholder_shared_local[(6)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 6))];
  placeholder_shared_local[(7)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 7))];
  placeholder_shared_local[(8)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 8))];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  T_relu[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((((int)blockIdx.z) * 4) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_5_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  __shared__ float PaddedInput_shared[512];
  __shared__ float placeholder_shared[18];
  float PaddedInput_shared_local[12];
  float placeholder_shared_local[9];
  float DepthwiseConv2d[2];
  PaddedInput_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = ((((16 <= (((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x))) && (1 <= ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15))) && (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15) < 15)) ? placeholder[(((((((int)blockIdx.z) * 392) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) >> 4) * 14)) + ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) & 15)) - 15))] : 0.000000e+00f);
  PaddedInput_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196))] = (((((16 <= (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196) & 255)) && ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196) & 255) < 240)) && (1 <= (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 4) & 15))) && ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 4) & 15) < 15)) ? placeholder[((((((((int)blockIdx.z) * 392) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196) >> 8) * 196)) + (((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 196) & 255) >> 4) * 14)) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 4) & 15)) - 15))] : 0.000000e+00f);
  if ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 120) {
    if (((((int)threadIdx.z) * 14) + ((int)threadIdx.y)) < 18) {
      PaddedInput_shared[(((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392))] = (((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 104) && (1 <= (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15))) && ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15) < 15)) ? placeholder[((((((((int)blockIdx.z) * 392) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 392) >> 8) * 196)) + ((((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 136) >> 4) * 14)) + (((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) + 8) & 15)) - 15))] : 0.000000e+00f);
    }
  }
  if ((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)) < 18) {
    if (((((int)threadIdx.z) * 14) + ((int)threadIdx.y)) < 3) {
      if (((int)threadIdx.z) < 1) {
        placeholder_shared[((((((int)threadIdx.z) * 98) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = placeholder1[(((((((int)threadIdx.z) * 98) + (((int)blockIdx.z) * 18)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))];
      }
    }
  }
  __syncthreads();
  PaddedInput_shared_local[(0)] = PaddedInput_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)))];
  PaddedInput_shared_local[(1)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 1))];
  PaddedInput_shared_local[(2)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 2))];
  PaddedInput_shared_local[(3)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 3))];
  PaddedInput_shared_local[(4)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 16))];
  PaddedInput_shared_local[(5)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 17))];
  PaddedInput_shared_local[(6)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 18))];
  PaddedInput_shared_local[(7)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 19))];
  PaddedInput_shared_local[(8)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 32))];
  PaddedInput_shared_local[(9)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 33))];
  PaddedInput_shared_local[(10)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 34))];
  PaddedInput_shared_local[(11)] = PaddedInput_shared[(((((((int)threadIdx.z) * 256) + (((int)threadIdx.y) * 16)) + (((int)threadIdx.x) * 2)) + 35))];
  placeholder_shared_local[(0)] = placeholder_shared[((((int)threadIdx.z) * 9))];
  placeholder_shared_local[(1)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 1))];
  placeholder_shared_local[(2)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 2))];
  placeholder_shared_local[(3)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 3))];
  placeholder_shared_local[(4)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 4))];
  placeholder_shared_local[(5)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 5))];
  placeholder_shared_local[(6)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 6))];
  placeholder_shared_local[(7)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 7))];
  placeholder_shared_local[(8)] = placeholder_shared[(((((int)threadIdx.z) * 9) + 8))];
  DepthwiseConv2d[(0)] = 0.000000e+00f;
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(0)], placeholder_shared_local[(0)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(1)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(2)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(4)], placeholder_shared_local[(3)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(4)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(5)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(8)], placeholder_shared_local[(6)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(7)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(0)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(8)], DepthwiseConv2d[(0)]);
  DepthwiseConv2d[(1)] = 0.000000e+00f;
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(1)], placeholder_shared_local[(0)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(2)], placeholder_shared_local[(1)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(3)], placeholder_shared_local[(2)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(5)], placeholder_shared_local[(3)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(6)], placeholder_shared_local[(4)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(7)], placeholder_shared_local[(5)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(9)], placeholder_shared_local[(6)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(10)], placeholder_shared_local[(7)], DepthwiseConv2d[(1)]);
  DepthwiseConv2d[(1)] = __ocml_fma_f32(PaddedInput_shared_local[(11)], placeholder_shared_local[(8)], DepthwiseConv2d[(1)]);
  T_relu[(((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)))] = max((DepthwiseConv2d[(0)] + placeholder2[(((((int)blockIdx.z) * 2) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[((((((((int)blockIdx.z) * 392) + (((int)threadIdx.z) * 196)) + (((int)threadIdx.y) * 14)) + (((int)threadIdx.x) * 2)) + 1))] = max((DepthwiseConv2d[(1)] + placeholder2[(((((int)blockIdx.z) * 2) + ((int)threadIdx.z)))]), 0.000000e+00f);
}

extern "C" __global__ void fused_nn_conv2d_add_nn_relu_10_kernel0(float* __restrict__ placeholder, float* __restrict__ placeholder1, float* __restrict__ T_relu, float* __restrict__ placeholder2) {
  float compute[4];
  __shared__ float pad_temp_shared[1792];
  __shared__ float placeholder_shared[512];
  compute[(0)] = 0.000000e+00f;
  compute[(2)] = 0.000000e+00f;
  compute[(1)] = 0.000000e+00f;
  compute[(3)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 112) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = placeholder[((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))];
    pad_temp_shared[(((((((int)threadIdx.z) * 112) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = placeholder[(((((((rc_outer * 12544) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 4)) < 32) {
      if ((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)) < 512) {
        if (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) < 32) {
          if (((int)threadIdx.x) < 8) {
            placeholder_shared[((((((int)threadIdx.z) * 32) + (((int)threadIdx.y) * 8)) + ((int)threadIdx.x)))] = placeholder1[((((((((int)blockIdx.z) * 4096) + (((int)threadIdx.z) * 256)) + ((((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) >> 4) * 128)) + (rc_outer * 16)) + (((((int)threadIdx.y) * 8) + ((int)threadIdx.x)) & 15)))];
          }
        }
      }
    }
    __syncthreads();
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[(((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[((((int)threadIdx.z) * 16))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1))], placeholder_shared[(((((int)threadIdx.z) * 16) + 256))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 112))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 1))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 113))], placeholder_shared[(((((int)threadIdx.z) * 16) + 257))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 224))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 2))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 225))], placeholder_shared[(((((int)threadIdx.z) * 16) + 258))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 336))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 3))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 337))], placeholder_shared[(((((int)threadIdx.z) * 16) + 259))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 448))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 4))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 449))], placeholder_shared[(((((int)threadIdx.z) * 16) + 260))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 560))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 560))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 561))], placeholder_shared[(((((int)threadIdx.z) * 16) + 5))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 561))], placeholder_shared[(((((int)threadIdx.z) * 16) + 261))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 672))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 672))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 673))], placeholder_shared[(((((int)threadIdx.z) * 16) + 6))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 673))], placeholder_shared[(((((int)threadIdx.z) * 16) + 262))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 784))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 785))], placeholder_shared[(((((int)threadIdx.z) * 16) + 7))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 785))], placeholder_shared[(((((int)threadIdx.z) * 16) + 263))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 896))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 8))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 897))], placeholder_shared[(((((int)threadIdx.z) * 16) + 264))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1008))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1008))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1009))], placeholder_shared[(((((int)threadIdx.z) * 16) + 9))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1009))], placeholder_shared[(((((int)threadIdx.z) * 16) + 265))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1120))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1120))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1121))], placeholder_shared[(((((int)threadIdx.z) * 16) + 10))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1121))], placeholder_shared[(((((int)threadIdx.z) * 16) + 266))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1232))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1232))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1233))], placeholder_shared[(((((int)threadIdx.z) * 16) + 11))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1233))], placeholder_shared[(((((int)threadIdx.z) * 16) + 267))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1344))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1344))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1345))], placeholder_shared[(((((int)threadIdx.z) * 16) + 12))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1345))], placeholder_shared[(((((int)threadIdx.z) * 16) + 268))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1456))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1456))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1457))], placeholder_shared[(((((int)threadIdx.z) * 16) + 13))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1457))], placeholder_shared[(((((int)threadIdx.z) * 16) + 269))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1568))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1568))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1569))], placeholder_shared[(((((int)threadIdx.z) * 16) + 14))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1569))], placeholder_shared[(((((int)threadIdx.z) * 16) + 270))], compute[(3)]);
    compute[(0)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1680))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(0)]);
    compute[(2)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1680))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(2)]);
    compute[(1)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1681))], placeholder_shared[(((((int)threadIdx.z) * 16) + 15))], compute[(1)]);
    compute[(3)] = __ocml_fma_f32(pad_temp_shared[((((((int)threadIdx.y) * 28) + (((int)threadIdx.x) * 2)) + 1681))], placeholder_shared[(((((int)threadIdx.z) * 16) + 271))], compute[(3)]);
  }
  T_relu[((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)))] = max((compute[(0)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12544))] = max((compute[(2)] + placeholder2[((((((int)blockIdx.z) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 1))] = max((compute[(1)] + placeholder2[(((((int)blockIdx.z) * 32) + ((int)threadIdx.z)))]), 0.000000e+00f);
  T_relu[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 784)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 28)) + (((int)threadIdx.x) * 2)) + 12545))] = max((compute[(3)] + placeholder2[((((((int)blockIdx.z) * 32) + ((int)threadIdx.z)) + 16))]), 0.000000e+00f);
}

